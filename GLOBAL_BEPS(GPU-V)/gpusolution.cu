
#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 

__device__ void gpusolution11(float xx[],float x1[],float x2[],float x3[],float Y[])
{
int i,j;
float alpha[10],gamma[10],a[10],b[10],c[10];
int num;
num=layer;

float X[10][10];

for(i=0;i<10;i++){
	X[1][i]=x1[i];
	X[2][i]=x2[i];
	X[3][i]=x3[i];
}





for(i=1;i<=num;i++){
a[i]=X[i][1];b[i]=X[i][2];c[i]=X[i][3];
}
 
alpha[1]=1.0/b[1];
gamma[1]=c[1]*alpha[1];

for(i=2;i<num;i++){
 
alpha[i]=1/(b[i]-a[i]*gamma[i-1]);
gamma[i]=c[i]*alpha[i];
}


xx[1]=Y[1]*alpha[1];

for(i=2;i<num;i++) xx[i]=(Y[i]-a[i]*xx[i-1])*alpha[i];

xx[num]=(Y[num]-a[num]*xx[num-1])/(b[num]-a[num]*gamma[num-1]);
 
for(i=1;i<num;i++){
j=num-i;
xx[j]=xx[j]-gamma[j]*xx[j+1];
}

}
