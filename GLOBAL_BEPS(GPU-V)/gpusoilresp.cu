#include "hip/hip_runtime.h"
/*************************************************************************
  Program:     soilresp.c
  --------
  Description:
  ----------- 
	Output soil respiration.
***************************************************************************
  CCRS (EMS/Applications Division)
  Written by:   J. Liu      
  Last update:	May 1998

*****************************************************************************/	
#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 
/* void soilresp(coef,lambda,Cstem1,Cleaf1,Ccroot1,Cfroot1,Ccd1,Csmd1,Cssd1,
	 Cfmd1 ,Cfsd1 ,Csm1 ,Cm1 ,Cs1 ,Cp1 ,initialValues) 

	 float coef[];
 float  lambda,Cstem1,Cleaf1,Ccroot1,Cfroot1,Ccd1,Csmd1,Cssd1,Cfmd1 ,Cfsd1,Csm1 ,Cm1 ,Cs1 ,Cp1;
float  initialValues[];*/




__device__ void gpusoilresp(int long pix,int short lc,float coef[],long jday,float lat,float dTmean,float dNPP,float Ndep, float Ndep0,
struct carbonpool Cpoolold[], struct CNratio CNRold[], struct carbonpool Cpoolnew[], struct CNratio CNRnew[],float lambda, float lambda2) 

{  
   	float fw, fcr, fl, ffr; 
	float kw_cd, kcr_cd;  //Stem ��Croot����ת����
	float kl_sl, kfr_fl,kl_sl1;  //Leaf ��froot����ת����
	float km_p, ks_p;
   	float kssd_a, kssd_sm, kssd_s, ksmd_a, ksmd_sm,kfsd_a, kfsd_m, kfsd_s, kfmd_a, kfmd_m;
	float kcd_a, kcd_m;
   	float kcd_s,ksm_a,ksm_s, km_a, km_s, ks_a, ks_m,kp_a, kp_m; 	
	float dCcd,dCssd,dCsmd,dCfsd,dCfmd,dCsm,dCm,dCs, dCp;

	float Cw0, Ccr0, Cl0, Cfr0, Ccd0,Cssd0,Csmd0,Cfsd0,Cfmd0, Cm0, Csm0, Cs0, Cp0;
    float Cw1, Ccr1, Cl1, Cfr1, Ccd1,Cssd1,Csmd1,Cfsd1,Cfmd1, Cm1, Csm1, Cs1, Cp1;
	float Nav,NT;
   	float CNssd, CNsmd, CNfsd,CNfmd,CNs, CNw, CNl,CNcd,CNsm,CNm,CNp;

	float Sc, Sn, NP, Nt,u1,u2,u3;
    float Nmin;
    float Nleaf1,Nfroot1,Nwood1, Ncroot1, Ncd1,Nssd1,Nsmd1,Nfsd1,Nfmd1,Ns1,Nsm1,Nm1,Np1;
   float Nleaf0,Nfroot0,Nwood0, Ncroot0, Ncd0,Nssd0,Nsmd0,Nfsd0,Nfmd0,Ns0,Nsm0,Nm0,Np0;
	float Rnpp_leaf,Rnpp_wood,Rnpp_froot,Rnpp_croot;
   // float A1,A3;
    float part1,part2;	
	float Fm;	
	float lam_u,lam_d;
    short  num=0;
    float a1,a2,a3,a4,a5,a6,aa;
	float b1,b2,b3,b4,b5,b6,b7,Tk,Kt,Nav1;
   // float  totalN2;
     int j;


	lam_u= lambda;                      //for surface pool
    lam_d= lambda2;  // for soil pool     
  
    fw = coef[0];  	   fcr = coef[1]; 	fl = coef[2]; 	ffr = coef[3];   //NPP�������������ı���ϵ��  
	  a1=1-coef[4]; a2=1-coef[5]; a3=1-coef[6]; a4=1-coef[7];
     if(a1<0.0001) a1=0.0001;
	 if(a2<0.0001) a2=0.0001;
	 if(a3<0.0001) a3=0.0001;
	 if(a4<0.0001) a4=0.0001;

     



	if(lc==1 || lc==7 || lc==8 || lc==4 || lc==10){      //������ 
  
	a5=1.0/90.0; a6=1.0/365.0;	
	
	if(lat>0){  //==========================================������==================================== 
	if(jday>=60 && jday<150){   //����ҶƬ��245��365����� 
			kw_cd   =1.0-pow(a1,a6);    
			kcr_cd  =1.0-pow(a2,a6);

			kl_sl1  =1.0-pow(a3,a5);  //	kw_cd =coef[4]*1.0/365.0;  //=========//
			kfr_fl  =1.0-pow(a4,a5);  //fl represents fine root litter 
		}
		else {
			kw_cd   =1.0-pow(a1,a6); 
			kl_sl1  =0;   	
			kcr_cd  =1.0-pow(a2,a6);
			kfr_fl  =0;
		}
	}

	else
	{//====================================================================�ϰ���================================
		if(jday>=240 && jday<330){
			kw_cd   =1.0-pow(a1,a6);   
			kcr_cd  =1.0-pow(a2,a6);
			kl_sl1  =1.0-pow(a3,a5);     //kw_cd =coef[4]*1.0/365.0;  //=========//
			kfr_fl  =1.0-pow(a4,a5);     //fl represents fine root litter 
		}
		else {
			kw_cd   =1.0-pow(a1,a6);
			kl_sl1   =0;   
			kcr_cd  =1.0-pow(a2,a6); 
			kfr_fl  =0;
		}

	} //�ϰ���
	
		} 
	else{  //===================================================�ǳ���ֲ��==========================================================================

	 a5=1.0/65.0; a6=1.0/365.0;	
	
		if(lat>0){  //==========================================������==================================== 
	
	if(jday>=300&& jday<365){   //����ҶƬ��245��365����� 
    kw_cd   =1.0-pow(a1,a6);    
	kcr_cd  =1.0-pow(a2,a6);

	kl_sl1  =1.0-pow(a3,a5);  //	kw_cd =coef[4]*1.0/365.0;  //=========//
	kfr_fl  =1.0-pow(a4,a5);  //fl represents fine root litter 
	}
   else {
   kw_cd   =1.0-pow(a1,a6); 
   kl_sl1  =0;   	
   kcr_cd  =1.0-pow(a2,a6);
   kfr_fl  =0;
	}
  
  	
	}

	else
    {//====================================================================�ϰ���================================
	if(jday>=120 && jday<180){
     kw_cd   =1.0-pow(a1,a6);   
	 kcr_cd  =1.0-pow(a2,a6);
     kl_sl1  =1.0-pow(a3,a5);     //kw_cd =coef[4]*1.0/365.0;  //=========//
	 kfr_fl  =1.0-pow(a4,a5);     //fl represents fine root litter 
	}
   else {
   kw_cd   =1.0-pow(a1,a6);
   kl_sl1   =0;   
   kcr_cd  =1.0-pow(a2,a6); 
   kfr_fl  =0;
	}
	 
	} //�ϰ���
	
	}

 
  
	if(lc==16 || lc==17 || lc==18) kl_sl=kl_sl1;    //����ũ���������������0.6���ջ�
	else                           kl_sl=kl_sl1;


	kssd_a  = coef[8] ;       kssd_sm = coef[9];   kssd_s  = coef[10];
    
	ksmd_a  = coef[11];       ksmd_sm = coef[12];
    
    kfsd_a  = coef[13];       kfsd_m  = coef[14];  kfsd_s  = coef[15];
	
	kfmd_a  = coef[16];    	  kfmd_m  = coef[17];	

	kcd_a   = coef[18];       kcd_m   = coef[19];  kcd_s   = coef[20];

    km_a    = coef[21];       km_p    = coef[22];  km_s    = coef[23];

    ksm_a   = coef[24];       ksm_s   = coef[25]; 
	
	ks_a    = coef[26];      ks_p    = coef[27];   ks_m    = coef[28];

	kp_a    = coef[29];       kp_m    = coef[30];
    
	//ֲ��̼�� 
	Cw0   = Cpoolold[pix].Cstem;  Ccr0 =Cpoolold[pix].Ccroot; Cl0   =Cpoolold[pix].Cleaf; Cfr0  =Cpoolold[pix].Cfroot;
    
	//����̼��
	Ccd0  = Cpoolold[pix].Ccd; 
	Cssd0= Cpoolold[pix].Cssd;
	Csmd0= Cpoolold[pix].Csmd; 
	Cfsd0 = Cpoolold[pix].Cfsd; 
	Cfmd0 = Cpoolold[pix].Cfmd;
	Csm0 =  Cpoolold[pix].Csm; 
	Cm0   = Cpoolold[pix]. Cm; 
	Cs0  =  Cpoolold[pix].Cs;
	Cp0   = Cpoolold[pix].Cp;

CNcd=CNRold[pix].CNcd;
CNssd=CNRold[pix].CNssd; 
CNsmd= CNRold[pix].CNsmd;
CNfsd= CNRold[pix].CNfsd;
CNfmd= CNRold[pix].CNfmd;
CNsm = CNRold[pix].CNsm;
CNm  = CNRold[pix].CNm;
CNs  = CNRold[pix].CNs;
CNp  = CNRold[pix].CNp;

CNw  = CNRold[pix].CNstem;
CNl  = CNRold[pix].CNleaf;
Nav  =CNRold[pix].Nav;
NT  = CNRold[pix].Nt; 

if(lc==13 || lc==16 || lc==17 || lc==18)   Fm=0.6;//0.85-0.018*0.6*CNl;   //2014��8��20����0.2 ��Ϊ0.3 
else                                       Fm=0.3;


 //a1=Cw0+Ccr0+Cl0+Cfr0+Ccd0+	Cssd0+Csmd0+Cfsd0+Cfmd0+Csm0+Cm0+Cs0+Cp0;

 
           part1   =(kw_cd * Cw0+kcr_cd * Ccr0);
           part2   =Ccd0 * lam_d* (kcd_a + kcd_m + kcd_s);
           dCcd =part1-part2;
           Ccd1 =Ccd0+ dCcd;   // Coarse detritus from woody and coarse root;
  
           Ncd0=Ccd0/CNcd;   //ǰһʱ�̸ÿ�ĵ���
           //��ʱ�̸ÿ�ĵ���
		   Ncd1=Ccd0/CNcd+(kw_cd * Cw0/CNw+kcr_cd * Ccr0/CNw) 
	                      -Ccd0/CNcd * lam_d* (kcd_a + kcd_m + kcd_s);
              
            part1   =(1 - Fm)* kl_sl*Cl0;
            part2   =Cssd0* lam_u * (kssd_a + kssd_sm + kssd_s);
            dCssd   =part1-part2;
            Cssd1   = Cssd0+dCssd;                 // for surface structural litter
           
			Nssd0   = Cssd0/CNssd;                 //ǰһʱ�̸ÿ�ĵ���  
             //��ʱ�̸ÿ�ĵ���
			Nssd1   =Cssd0/CNssd + (1 - Fm)* kl_sl1*Cl0/CNl
                     -Cssd0/CNssd* lam_u * (kssd_a + kssd_sm + kssd_s);

             part1   =Fm* kl_sl * Cl0;
             part2   = Csmd0* lam_u * (ksmd_a + ksmd_sm);
             dCsmd   =part1-part2;
             Csmd1   = Csmd0+dCsmd;                 // for surface metabolic litter
            
			 Nsmd0   =Csmd0/CNsmd;    //ǰһʱ�̸ÿ�ĵ���
             //��ʱ�̸ÿ�ĵ���
			 Nsmd1   =Csmd0/CNsmd+ Fm* kl_sl1 * Cl0/CNl
                     -Csmd0/CNsmd* lam_u * (ksmd_a + ksmd_sm);

               
             part1=(1 - Fm)* kfr_fl* Cfr0;
             part2=Cfsd0* lam_d * (kfsd_a + kfsd_m + kfsd_s);
             dCfsd =part1-part2;
             Cfsd1= Cfsd0+dCfsd;                  //for soil strutural litter pool
  
             Nfsd0=Cfsd0/CNfsd; //ǰһʱ�̸ÿ�ĵ���
             //��ʱ�̸ÿ�ĵ���
			 Nfsd1=Cfsd0/CNfsd+ (1 - Fm)* kfr_fl* Cfr0/CNl
                     - Cfsd0/CNfsd* lam_d * (kfsd_a + kfsd_m + kfsd_s);

  
             part1=Fm * kfr_fl * Cfr0; 
             part2=lam_d * (kfmd_a + kfmd_m)* Cfmd0;
             dCfmd=part1-part2;
             Cfmd1= Cfmd0+dCfmd;                   // for soil metabolic pool
             Nfmd0= Cfmd0/CNfmd;  //ǰһʱ�̸ÿ�ĵ���
             //��ʱ�̸ÿ�ĵ���
			 Nfmd1= Cfmd0/CNfmd+Fm * kfr_fl * Cfr0/CNl
                    -lam_d * (kfmd_a + kfmd_m)* Cfmd0/CNfmd;

             part1=lam_u*(Cssd0*kssd_sm+Csmd0*ksmd_sm);
             part2=lam_u*Csm0*(ksm_a+ksm_s);
             dCsm=part1-part2;
             Csm1=Csm0+dCsm;                       // for surface microbe pool
            
			 Nsm0=Csm0/CNsm ;     //ǰһʱ�̸ÿ�ĵ���
             //��ʱ�̸ÿ�ĵ���
			 Nsm1=Csm0/CNsm + lam_u*(Cssd0/CNssd*kssd_sm+Csmd0/CNsmd*ksmd_sm)
                    - lam_u*Csm0/CNsm*(ksm_a+ksm_s);


             part1=(lam_d * (kfsd_m * Cfsd0+kfmd_m*Cfmd0 + Ccd0* kcd_m) +lam_d*(Cs0*ks_m+Cp0 * kp_m));				
             part2=Cm0 * lam_d*(km_a +  km_s +km_p);
             dCm=part1-part2;
             Cm1=Cm0+dCm;                          // for soil microbe pool
            
			 Nm0=Cm0/CNm;                          //ǰһʱ�̸ÿ�ĵ���
             //��ʱ�̸ÿ�ĵ���
             Nm1=Cm0/CNm +(lam_d * (kfsd_m * Cfsd0/CNfsd+kfmd_m*Cfmd0/CNfmd+Ccd0/CNcd* kcd_m)+
				     lam_d*(Cs0/CNs*ks_m+Cp0/CNp* kp_m))
	                      -Cm0/CNm * lam_d*(km_a +  km_s +km_p);
     
         
			 part1=(lam_d*(Cm0*km_s + Ccd0 * kcd_s +Cfsd0*kfsd_s )+ lam_u* (Csm0*ksm_s + Cssd0*kssd_s));
             part2=Cs0* lam_d *( ks_a + ks_p+ks_m)/(1+lam_d *( ks_a + ks_p+ks_m));
             dCs=part1-part2;
             Cs1=Cs0+dCs;                          // for slow carbon pool
            
			 Ns0=Cs0/CNs;        //ǰһʱ�̸ÿ�ĵ���
              //��ʱ�̸ÿ�ĵ���
			 Ns1=Cs0/CNs+(lam_d*(Cm0/CNm*km_s + Ccd0/CNcd * kcd_s +Cfsd0/CNfsd*kfsd_s )+ 
				 lam_u* (Csm0/CNsm*ksm_s+Cssd0/CNssd*kssd_s))
	             -Cs0/CNs* lam_d *( ks_a + ks_p+ks_m);
   				
             dCp =(lam_d *( km_p * Cm0 + ks_p * Cs0))
		                 - Cp0*lam_d * (kp_m + kp_a );
             Cp1=Cp0+dCp;                         // for passive carbon pool.
             Np0=Cp0/CNp;  // //ǰһʱ�̸ÿ�ĵ���		
             // //��ʱ�̸ÿ�ĵ���
			 Np1=Cp0/CNp+lam_d *( km_p * Cm0/CNm + ks_p * Cs0/CNs) 
                        - Cp0/CNp*lam_d * (kp_m + kp_a );

			 
			// A1=Ncd0+Nssd0+Nsmd0+Nfsd0+Nfmd0+Nsm0+Nm0+Ns0+Np0;  //ǰһ��������N��
  
           //  Nwood0   = Cw0/CNw;   Ncroot0  =  Ccr0/CNw;  Nleaf0   =Cl0/CNl;       Nfroot0  = Cfr0/CNl; 
            b6=(Cw0+Ccr0)/CNw+(Cl0+Cfr0)/CNl; 
            b7=(Cw0+Ccr0)/coef[47]+(Cl0+Cfr0)/coef[46]; 
            
           // A2=Ncd1+Nssd1+Nsmd1+Nfsd1+Nfmd1+Nsm1+Nm1+Ns1+Np1;  //��ǰ��������N�� 

         	 Nmin =(float)(lam_u * (Cssd0* kssd_a/CNssd+Csmd0* ksmd_a/CNsmd+Csm0*ksm_a/CNsm) 
                      +lam_d*(Ccd0 *kcd_a/CNcd+Cfsd0* kfsd_a/CNfsd+Cfmd0 * kfmd_a/CNfmd 
                      +Cm0*km_a/CNm+Cs0*ks_a/CNs+Cp0*kp_a/CNp));    //�󻯵���N��
                  

	
  //===========================================================================================================================================//
	  if(Ndep>Ndep0)   Nav=Nav+Nmin+(Ndep-Ndep0)*0.8/365.0;   //���赪������������Ϊ60%
	  else             Nav=Nav+Nmin;
	  
	
		 Tk=dTmean+273.15;
		
       	Sc=Ccd1+Cssd1+Csmd1+Cfsd1+Cfmd1+Csm1+Cm1+Cs1;//+Cp1);//������̼�� 
   	 	Sn=Nav;                                            //�����ܵ���    
        a3=Sn/600.0; 
        
        aa=b7/b6;     
        if(aa>2.0) aa=2.0;
        if(aa<1)   aa=1.0; 


		if(a3>aa) a2=aa;//+0.25*log(a3);   //2017-10-12 ��Ϊ��ģ� ������Ҫ���飬 Ŀ����������N ���ӣ� N����Ҳ����
		else {

			a2=a3;
		}   

		NP=120*a2*exp(-8.0/100000*Sc);

		u1=40.8+0.01*dTmean-0.002*dTmean*dTmean;  //dTmean Ϊƽ���¶�

		u2=0.738-0.002*dTmean;

		u3=97.42-2.504*log(NP);


        a1=exp(u1-u3/(0.00831*Tk)); //NT ������Ŀ

		a2=1+exp((u2*Tk-205.9)/(0.00831*Tk));  //NT��ĸ��Ŀ


if(dTmean<15 && Sc>13000) Kt=(1+(15-dTmean)/30.0)*(1+(Sc-13000)/10000);
                     else Kt=1.0;

 
			 //   if(dNPP>0) Nt=a1/a2*Kt*14.0/1000.0;//ֲ��ÿ�����յ�N������λg N/m-2/d
	//  else        Nt=0;



Nt=a1/a2*Kt*14.0/1000.0;//ֲ��ÿ�����յ�N������λg N/m-2/d


		if(Nt>Nav) Nt=Nav;       //ֲ�����������ȫ����N
		
		Nav=Nav -Nt;     
       
		NT=NT+Nt; //����

		  Nav1=Nav;
	   
       Rnpp_wood=coef[0]; Rnpp_croot=coef[1]; //NPP ����ϵ��
       Rnpp_leaf=coef[2]; Rnpp_froot=coef[3]; 
 
       Cw1   = Cw0 +  (dNPP*Rnpp_wood- Cw0*kw_cd);///(1+kw_cd); 
       Nwood0   = Cw0/CNw;       //+Nt*Rnpp_wood/1000.0  ;///(1+kw_cd); //+
       Nwood1   = Cw0/CNw-Cw0*kw_cd/CNw;///(1+kw_cd) ;
       
	   Ccr1 =  Ccr0+  (dNPP*Rnpp_croot- Ccr0*kcr_cd);///(1+kcr_cd);  
       Ncroot0  =  Ccr0/CNw;   //+Nt*Rnpp_croot/1000.0; //+
       Ncroot1  =  Ccr0/CNw- Ccr0*kcr_cd/CNw;///(1+kcr_cd);
	   
       a1=Rnpp_wood*(CNw/coef[47])*(CNw/coef[47])*(CNw/coef[47])*(CNw/coef[47]);
       a2=Rnpp_croot*(CNw/coef[47])*(CNw/coef[47])*(CNw/coef[47])*(CNw/coef[47]); 
       
	   Cl1 = Cl0  +  (dNPP*Rnpp_leaf-Cl0*kl_sl1);///(1+kl_sl);
       Nleaf0   =Cl0/CNl;        // +Nt*Rnpp_leaf/1000.0;///(1+kl_sl);    //+
       Nleaf1   =Cl0/CNl- Cl0*kl_sl1/CNl;///(1+kl_sl);        // +Nt*Rnpp_leaf/1000.0;///(1+kl_sl);    //+

	   Cfr1     = Cfr0+  (dNPP*Rnpp_froot-Cfr0*kfr_fl);///(1+kfr_fl);        
       Nfroot0  = Cfr0/CNl;  //+Nt*Rnpp_froot/1000.0;///(1+kfr_fl);   //+
       Nfroot1  = Cfr0/CNl- Cfr0*kfr_fl/CNl;///(1+kfr_fl);  
      
	  a3=Rnpp_leaf*(CNl/coef[46])*(CNl/coef[46])*(CNl/coef[46])*(CNl/coef[46]);
      a4=Rnpp_froot*(CNl/coef[46])*(CNl/coef[46])*(CNl/coef[46])*(CNl/coef[46]);
       
	   b1=Nt*a1/(a1+a2+a3+a4);
	   b2=Nt*a2/(a1+a2+a3+a4);


	   b3=Nt*a3/(a1+a2+a3+a4);
	   b4=Nt*a4/(a1+a2+a3+a4);



	   b5=Nwood1+Ncroot1+Nleaf1+ Nfroot1;
	   
	   Nwood1 =Nwood1+b1;
	   Ncroot1=Ncroot1+b2;
	   Nleaf1 =Nleaf1+b3;
	   Nfroot1=Nfroot1+b4;
	
       //b6=Nwood1+Ncroot1+Nleaf1+ Nfroot1;


	 //  A3=A1+Nwood0+Ncroot0+Nleaf0+Nfroot0;  //ǰһ��ʱ�䲽����ֲ����������
	  // A4=A2+Nwood1+Ncroot1+Nleaf1+Nfroot1;  //��ǰһ��ʱ�䲽����ֲ����������
      // A5=Nwood1+Ncroot1+Nleaf1+Nfroot1; 
	   //A6=Nwood1+Ncroot1+Nleaf1+Nfroot1-(Nwood0+Ncroot0+Nleaf0+Nfroot0);
 
	if((Cw1+Ccr1)>0 &&( Nwood1 +Ncroot1)>0 )   CNw=(Cw1+Ccr1)/( Nwood1 +Ncroot1);
	else CNw  =CNRold[pix].CNstem;
	
	if((Cl1+Cfr1)>0 && ( Nleaf1 +Nfroot1)>0  )   CNl=(Cl1+Cfr1)/( Nleaf1 +Nfroot1);
	else CNl  =CNRold[pix].CNleaf;

  
if(CNw>(coef[47]*2)) {
Nav=Nav-(Cw1+Ccr1)/(coef[47]*2)+(Cw1+Ccr1)/CNw;

CNw=coef[47]*2;

}
if(Nav<0) Nav=0;


if(CNw<(coef[47]*0.5)) {
Nav=Nav-(Cw1+Ccr1)/(coef[47]*0.5)+(Cw1+Ccr1)/CNw;
CNw=coef[47]*0.5;
}

if(Nav<0) Nav=0;


if(CNl>(coef[46]*2)) {
		 CNl=coef[46]*2;
Nav=Nav-(Cl1+Cfr1)/(coef[46]*2)+(Cl1+Cfr1)/CNl;
CNl=coef[46]*2.0;

}
if(Nav<0) Nav=0;

if(CNl<(coef[46]*0.5)) { 
Nav=Nav-(Cl1+Cfr1)/(coef[46]*0.5)+(Cl1+Cfr1)/CNl;
CNl=coef[46]*0.5;

}

if(Nav<0) Nav=0;



   Nwood1  = Cw1/CNw;   Ncroot1  =  Ccr1/CNw;  Nleaf1  =Cl1/CNl;       Nfroot1 = Cfr1/CNl; 

   
   if(Ccd1<0.001)  Ccd1=0.001;
   if(Cssd1<0.001) Cssd1=0.001;
   if(Csmd1<0.001) Csmd1=0.001; 
   if(Cfsd1<0.001) Cfsd1=0.001;
   if(Cfmd1<0.001) Cfmd1=0.001;
   if(Csm1<0.001)  Csm1=0.001;
   if(Cm1<0.001)    Cm1=0.001; 
   if(Cs1<0.001)    Cs1=0.001;
   if(Cp1<0.001)    Cp1=0.001; 
   if(Cw1<0.001)    Cw1=0.001;
   if(Ccr1<0.001)   Ccr1=0.001; 
   if(Cfr1<0.001)  Cfr1=0.001;  
   if(Cl1<0.001)  Cl1=0.001;

    if(Ccd1>0)  CNcd=Ccd1/Ncd1;   
	else  CNcd=CNRold[pix].CNcd;
	 
	 if(Cssd1>0) CNssd=Cssd1/Nssd1;
	 else CNssd=CNRold[pix].CNssd;
	 
	 if(Csmd1>0) CNsmd=Csmd1/Nsmd1; 
	 else CNsmd=CNRold[pix].CNsmd;
	 
	 if(Cfsd1>0) CNfsd=Cfsd1/Nfsd1;
     else CNfsd=CNRold[pix].CNfsd;
	 
	 if(Cfmd1>0) CNfmd=Cfmd1/Nfmd1;
	 else CNfmd=CNRold[pix].CNfmd;
	 
	 if(Csm1>0) CNsm=Csm1/Nsm1;
	 else CNsm =CNRold[pix].CNsm;
	 
	 if(Cm1>0) CNm=Cm1/Nm1; 
	 else CNm  =CNRold[pix].CNm;
	 
	 if(Cs1>0) CNs=Cs1/Ns1;
     else CNs  =CNRold[pix].CNs;
	 
	 if(Cp1>0) CNp=Cp1/Np1; 
     else CNp  =CNRold[pix].CNp;


    if(CNcd>2*coef[35])   CNcd=2*coef[35]; 
    if(CNcd<0.5*coef[35]) CNcd=0.5*coef[35]; 

	if(CNssd>2*coef[36])   CNssd=2*coef[36]; 
    if(CNssd<0.5*coef[36]) CNssd=0.5*coef[36]; 

    if(CNsmd>2*coef[37])   CNsmd=2*coef[37]; 
    if(CNsmd<0.5*coef[37]) CNsmd=0.5*coef[37]; 

    if(CNfsd>2*coef[38])   CNfsd=2*coef[38]; 
    if(CNfsd<0.5*coef[38]) CNfsd=0.5*coef[38]; 

    if(CNfmd>2*coef[39])   CNfmd=2*coef[39]; 
    if(CNfmd<0.5*coef[39]) CNfmd=0.5*coef[39];

    if(CNs>2*coef[40])   CNs=2*coef[40]; 
    if(CNs<0.5*coef[40]) CNs=0.5*coef[40];

    if(CNsm>2*coef[41])   CNsm=2*coef[41]; 
    if(CNsm<0.5*coef[41]) CNsm=0.5*coef[41];
    
	if(CNm>2*coef[42])   CNm=2*coef[42]; 
    if(CNm<0.5*coef[42]) CNm=0.5*coef[42];

    if(CNp>2*coef[43])   CNp=2*coef[43]; 
    if(CNp<0.5*coef[43]) CNp=0.5*coef[43];

   // initialValues[27]=(float) (dCsmd+dCssd+dCfsd+dCfmd+dCcd+dCm+dCsm+dCs+dCp);
   
	Cpoolnew[pix].Hr=(float)(lam_u * (Cssd0* kssd_a+Csmd0* ksmd_a+Csm0*ksm_a) 
                      +lam_d*(Ccd0 *kcd_a+Cfsd0* kfsd_a+Cfmd0 * kfmd_a 
                      +Cm0*km_a+Cs0*ks_a+Cp0*kp_a))+(kl_sl1-kl_sl)*Cl0;    //heterotrophic respiration+ũ�����ջ�  
 

	 
Cpoolnew[pix].Cstem=Cw1; 
Cpoolnew[pix].Ccroot=Ccr1;
Cpoolnew[pix].Cleaf=Cl1;
Cpoolnew[pix].Cfroot=Cfr1;

Cpoolnew[pix].Ccd=Ccd1; 
Cpoolnew[pix].Cssd=Cssd1;
Cpoolnew[pix].Csmd=Csmd1; 
Cpoolnew[pix].Cfsd=Cfsd1; 
Cpoolnew[pix].Cfmd=Cfmd1;
Cpoolnew[pix].Csm=Csm1; 
Cpoolnew[pix].Cm=Cm1; 
Cpoolnew[pix].Cs=Cs1;
Cpoolnew[pix].Cp=Cp1;

CNRnew[pix].CNcd=CNcd;
CNRnew[pix].CNssd=CNssd; 
CNRnew[pix].CNsmd=CNsmd;
CNRnew[pix].CNfsd=CNfsd;
CNRnew[pix].CNfmd=CNfmd;
CNRnew[pix].CNsm=CNsm;
CNRnew[pix].CNm=CNm;
CNRnew[pix].CNs=CNs;
CNRnew[pix].CNp=CNp;

CNRnew[pix].CNstem=CNw;
CNRnew[pix].CNcroot=CNw;


CNRnew[pix].CNleaf=CNl;
CNRnew[pix].CNfroot=CNl;


CNRnew[pix].Nav=Nav;
CNRnew[pix].Nt=(float)(Cssd1+Csmd1+Csm1+Ccd1+Cfsd1+Cfmd1+Cm1+Cs1+Cp1)/1000.0; 


return;
   // return;
} 
