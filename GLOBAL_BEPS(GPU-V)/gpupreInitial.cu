#include "hip/hip_runtime.h"


#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 

//#include "beps.h" 
/****************************************************************
																
	   Function:	preInitials_J(NPP0, initValues[])								
	   Description: Calculate initial values of Cx, 
					and CN ratios for a specific NPP0							
																
*****************************************************************/;
__device__ void gpupreInitials(float  NPP, float lambd,float *lambddd, float * Tmean_m, float * NPPd_m,float * coef1, float * initialValues)

{
 	

   float Cw, Ccr, Cl, Cfr;
   float Ccd, Cfsd,Cfmd,Cssd,Csmd, Csm,Cm, Cs, Cp;
   float Fm;
   float CNssd, CNsmd, CNfsd,CNfmd,CNs, CNw, CNfr,CNl,CNcd,CNsm,CNm,CNp;
	/*The ratios of carbon to nitrogen of different carbon pools*/
	
	float fw, fcr, fl, ffr; 
	float kw_cd, kcr_cd;
	float kl_sl, kfr_fl;/* sl: surface litter;fl:fine root litter*/ 
	float kssd_a, kssd_sm, kssd_s, ksmd_a, ksmd_sm;
   	float kfsd_a, kfsd_m, kfsd_s, kfmd_a, kfmd_m;
    float kcd_s, kcd_a, kcd_m;
	float km_p, km_a,km_s;
	float ksm_a, ksm_s, ks_p,ks_a,ks_m, kp_a, kp_m; 	

	float a1,a2,a3,a4,a5,a6,a7,a8;
    float lam1,lam;
    //short interval=1; 
	// Calculate the initial values of C components	
   short year,day;

    float Cw0, Ccr0, Cl0, Cfr0, Ccd0,Cssd0,Csmd0,Cfsd0,Cfmd0, Cm0, Csm0, Cs0, Cp0;
    float Cw1, Ccr1, Cl1, Cfr1, Ccd1,Cssd1,Csmd1,Cfsd1,Cfmd1, Cm1, Csm1, Cs1, Cp1;
	
	float lam_u,lam_d,part1,part2;
     
	float dCcd,dCssd,dCsmd,dCfsd,dCfmd,dCsm,dCm,dCs, dCp;
	//float ratio;
   
    float Nav;
    float Sc, Sn, NP, Nt,u1,u2,u3;
  
    float Nleaf1,Nfroot1,Nwood1, NCcroot1, Ncd1,Nssd1,Nsmd1,Nfsd1,Nfmd1,Ns1,Nsm1,Nm1,Np1;
    float Nleaf0,Nfroot0,Nwood0, NCcroot0, Ncd0,Nssd0,Nsmd0,Nfsd0,Nfmd0,Ns0,Nsm0,Nm0,Np0;
	
	float Rnpp_leaf,Rnpp_wood,Rnpp_froot,Rnpp_Ccroot;
    float totalN,totalNup,totalNmin,A1,A2;//,A3,A4;
    
 
    float  b2;
	//float TN0,TN1;
	float Nmin;//,Nav0;
   // float  B5,B6,A6;
    float CNl_av;

	//======================================================================//
 	
  
	Nav=0; 
     
    fw     = coef1[0];     //The ratio of NPP allocated to stem
	fcr    = coef1[1];     //The ratio of NPP allocated to coarse roots
	fl     = coef1[2];     //The ratio of NPP allocated to leaves
	ffr    = coef1[3];     //The ratio of NPP allocated to fine roots
	kw_cd  = coef1[4];     //The turn over rate of stem pool
	kcr_cd = coef1[5];     //The turn over rate of coarse root pool
	kl_sl  = coef1[6];     //The turn over rate of leaf pool
	kfr_fl = coef1[7];     //The turn over rate of fine root pool

    kssd_a = coef1[8];     //surface structural litter pool
    kssd_sm =coef1[9];      
    kssd_s = coef1[10];
	
	ksmd_a = coef1[11];   //surface metabolic litter  pool
	ksmd_sm =coef1[12];
    
    kfsd_a = coef1[13];   //soil structural litter pool
    kfsd_m = coef1[14];      
    kfsd_s = coef1[15];
	
	kfmd_a = coef1[16];  //soil metabolic litter  pool
	kfmd_m = coef1[17];	

	kcd_a =  coef1[18];   //coarse detritus litter  pool
    kcd_m =  coef1[19];
    kcd_s =  coef1[20];

	km_a =   coef1[21];   //soil microbial C pool
	km_p =   coef1[22];
    km_s =   coef1[23] ;

    ksm_a=   coef1[24];   //surface microbial C pool
	ksm_s=   coef1[25];

	ks_a =   coef1[26];  //slow C  pool
	ks_p =   coef1[27];
    ks_m =   coef1[28] ;

	kp_a =   coef1[29];   //passive C pool
    kp_m =   coef1[30];

	CNcd  =  coef1[35]; 
    CNssd =  coef1[36];
	CNsmd =  coef1[37]; 
    CNfsd =  coef1[38];
	CNfmd =  coef1[39];
	CNs   =  coef1[40];
	CNsm  =  coef1[41];
	CNm   =  coef1[42];
	CNp   =  coef1[43];
     
    
	CNl   =  coef1[46];  
	CNw   =  coef1[47];

   	lam1  =lambd;                                      //for surface pools
    lam   =lambd;  // for soil pools      
  
 	
	totalNup=0;
    Fm=0.2;//0.85-0.018*0.6*CNl;
    	 
	Cw =  (fw/kw_cd)* NPP ;
	Ccr=  (fcr/kcr_cd)* NPP;
	Cl =  (fl/kl_sl) * NPP;
	Cfr=  (ffr/kfr_fl)* NPP;
 
    Cssd=(1-Fm)*kl_sl*Cl/(kssd_a+kssd_sm+kssd_s);
    Csmd=Fm*kl_sl*Cl/(ksmd_a+ksmd_sm);
    Cfsd=(1-Fm)*kfr_fl*Cfr/(kfsd_a+kfsd_m+kfsd_s);
    Cfmd=Fm*kfr_fl*Cfr/(kfmd_a+kfmd_m);
    Ccd =((kw_cd*Cw+kcr_cd*Ccr)/(kcd_a+kcd_m+kcd_s));
    Cssd= Cssd/lam1; 
    Csmd= Csmd/lam1;
    Cfsd= Cfsd/lam; 
    Cfmd= Cfmd/lam;
    Ccd = Ccd/lam; 

   Csm = (Cssd*kssd_sm+Csmd*ksmd_sm)/(ksm_a+ksm_s);

   a1=Cfsd*(kfsd_m*lam*(ks_a*lam+ks_p*lam+ks_m*lam)*(kp_a*lam+kp_m*lam)+(kp_a*lam+kp_m*lam)*ks_m*lam*kfsd_s*lam+ks_p*lam*kfsd_s*lam*kp_m*lam);
   
   a2=Cfmd*kfmd_m*lam*(kp_a*lam+kp_m*lam)*(ks_a*lam+ks_p*lam+ks_m*lam);
   
   a3=Ccd*(kcd_m*lam*(ks_a*lam+ks_p*lam+ks_m*lam)*(kp_a*lam+kp_m*lam)+(kp_a*lam+kp_m*lam)*ks_m*lam*kcd_s*lam +kp_m*lam*ks_p*lam*kcd_s*lam);
   
   a4=Csm*(ksm_s*lam1*ks_m*lam*(kp_a*lam+kp_m*lam)+ks_p*lam*kp_m*lam*ksm_s*lam1);
   
   a5=Cssd*(ks_m*lam*kssd_s*lam1*(kp_a*lam+kp_m*lam)+ks_p*lam*kp_m*lam*kssd_s*lam1);
   
   a6=(km_a*lam+km_p*lam+km_s*lam)*(ks_a*lam+ks_p*lam+ks_m*lam)*(kp_a*lam+kp_m*lam);
   
   a7=km_s*lam*ks_m*lam*(kp_a*lam+kp_m*lam);
      
   a8=kp_m*lam*(km_s*lam*ks_p*lam+km_p*lam*(ks_a*lam+ks_p*lam+ks_m*lam)); 
  
   Cm=(a1+a2+a3+a4+a5)/(a6-a7-a8);
   Cs=(Csm*ksm_s*lam1+Cssd*kssd_s*lam1+Cfsd*kfsd_s*lam+Cm*km_s*lam+Ccd*kcd_s*lam)/
	   (ks_a*lam+ks_p*lam+ks_m*lam);    
   Cp=(ks_p*Cs+km_p*Cm)/(kp_a+kp_m);


    //��ֲ��̼������ת���ʺ�����̼����ֽ�����ת��Ϊÿ���ֵ
	kw_cd   =kw_cd/365.0; 
    kl_sl   =kl_sl/365.0;  
	kcr_cd  =kcr_cd /365.0;
 	kfr_fl  =kfr_fl/365.0;  
	
	kssd_a  = kssd_a /365.0;
    kssd_sm = kssd_sm/365.0;      
    kssd_s  = kssd_s/365.0;
    
	ksmd_a  = ksmd_a/365.0;
	ksmd_sm = ksmd_sm/365.0;
    
    kfsd_a  = kfsd_a/365.0;
    kfsd_m  = kfsd_m/365.0;      
    kfsd_s  = kfsd_s/365.0;
	
	kfmd_a  = kfmd_a/365.0;
	kfmd_m  = kfmd_m/365.0;	

	kcd_a   = kcd_a/365.0;
    kcd_m   = kcd_m/365.0;
    kcd_s   = kcd_s/365.0;

    km_a    = km_a/365.0;
    km_p    = km_p/365.0;
	km_s    = km_s/365.0;

    ksm_a   = ksm_a/365.0;
	ksm_s   = ksm_s/365.0;

	ks_a    = ks_a/365.0;
	ks_p    = ks_p/365.0;
    ks_m    = ks_m/365.0;

	kp_a    = kp_a/365.0;
    kp_m    = kp_m/365.0;

Nwood0=Cw/CNw;     NCcroot0=Ccr/CNw;    Nleaf0=Cl/CNl;     Nfroot0=Cfr/CNl;
Ncd0 =Ccd/CNcd;    Nssd0=Cssd/CNssd;   Nsmd0=Csmd/CNsmd;  Nfsd0=Cfsd/CNfsd;
Nfmd0=Cfmd/CNfmd;  Ns0  =Cs/CNs;       Nsm0 =Csm/CNsm;    Nm0  =Cm/CNm;
Np0  =Cp/CNp;      

Cw1 = Cw  ;   Ccr1= Ccr ;    Cl1=  Cl;    Cfr1= Cfr ;
Ccd1 = Ccd;   Cssd1=Cssd;    Csmd1=Csmd;  Cfsd1=Cfsd;
Cfmd1=Cfmd;   Csm1 =Csm;     Cm1  =Cm;    Cs1  =Cs;
Cp1  =Cp;
  
NPPd_m[0]=0;

     	 lam_u= lam1;   //for surface pool
         lam_d= lam1;   // for soil pool     
         Nmin=Cssd1 *lam_u *(kssd_a + kssd_sm + kssd_s)/CNssd+ 
	     Cfsd1 *lam_d *(kfsd_a + kfsd_m  + kfsd_s)/CNfsd+ 
		 Csmd1*(ksmd_a+ksmd_sm) * lam_u/CNsmd+
	     Cfmd1*(kfmd_a+kfmd_m) * lam_d/CNfmd+
		 Ccd1 * lam_d * (kcd_a +kcd_m +kcd_s)/CNcd+
		 Csm1 *  lam_u*( ksm_a+  ksm_s )/CNsm+ 
     	 Cm1 *lam_d*(km_a+km_s+km_p)/CNm +
		 Cs1*lam_d*(ks_a +ks_p+ks_m)/CNs+ 
		 Cp1 * lam_d * (kp_a+kp_m)/CNp 
	
		 -(Cssd1* kssd_sm +Csmd1 *ksmd_sm)*lam_u / CNsm 
		 -((Cfsd1* kfsd_m +Cfmd1 *kfmd_m+ Ccd1 * kcd_m)*lam_d+ (kp_m * Cp1+ks_m*Cs1)*lam_d)/ CNm
		 -(lam_u*(Csm1*ksm_s+ Cssd1*kssd_s)+lam_d*(km_s * Cm1+Cfsd1* kfsd_s+Ccd1 * kcd_s)) / CNs 
		 -lam_d *(km_p * Cm1+ ks_p * Cs1) / CNp;
 
		 //���������л����� 
	     Nav=(coef1[0]*NPP/CNw+coef1[1]*NPP/CNw+coef1[2]*NPP/CNl+coef1[3]*NPP/CNl)*1; 
		 //��λΪ��g N m-2
		 //Nav0=Nav;//(coef1[0]*NPP/CNw+coef1[1]*NPP/CNw+coef1[2]*NPP/CNl+coef1[3]*NPP/CNl)*1000; 
 
	     Nt=0;//; TN0=Nav; Nav0=Nav;
	

 
	     for(year=0;year<20;year++){  //=============================================================/
           A1=0;A2=0;
         totalNup=0; totalN=0; totalNmin=0;
           CNl_av=0;
          for(day=1;day<365;day++){  //==========================================day loop==================================/
		 
			  if(day==364){
	       kw_cd  = coef1[4];     //The turn over rate of stem pool
	       kcr_cd = coef1[5];     //The turn over rate of coarse root pool
	       kl_sl  = coef1[6];     //The turn over rate of leaf pool
	       kfr_fl = coef1[7];     //The turn over rate of fine root pool
			  }
			  else{
            kw_cd   =0; 
            kl_sl   =0;  
	        kcr_cd  =0;
 	        kfr_fl  =0;  
			  }

         //  Fm=0.85-0.018*0.6*CNl;
         //  if(Fm<0.1) Fm=0.1;
         
		  Fm=0.2;
			
		   lam_u= lambddd[day];   //for surface pool
           lam_d= lambddd[day];   // for soil pool     

	       Cw0 = Cw1;    Ccr0 = Ccr1;   Cl0  = Cl1;   Cfr0 = Cfr1;
           Ccd0 =Ccd1;   Cssd0=Cssd1;   Csmd0=Csmd1;  Cfsd0=Cfsd1;   Cfmd0=Cfmd1;
           Csm0 =Csm1;   Cm0  =Cm1;     Cs0  =Cs1;    Cp0  =Cp1;
  
           part1   =(kw_cd * Cw0+kcr_cd * Ccr0);//(1+lam_d* (kcd_a + kcd_m + kcd_s));
           part2   =Ccd0 * lam_d* (kcd_a + kcd_m + kcd_s);//(1+lam_d* (kcd_a + kcd_m + kcd_s));
           dCcd =part1-part2;
           Ccd1 =Ccd0+ dCcd;   // Coarse detritus from woody and coarse root;
  
           Ncd0=Ccd0/CNcd;   //ǰһʱ�̸ÿ�ĵ���
           //��ʱ�̸ÿ�ĵ���
		   Ncd1=Ccd0/CNcd+(kw_cd * Cw0/CNw+kcr_cd * Ccr0/CNw)        //(1+lam_d* (kcd_a + kcd_m + kcd_s))
	                      -Ccd0/CNcd * lam_d* (kcd_a + kcd_m + kcd_s);//(1+lam_d* (kcd_a + kcd_m + kcd_s));
              
            part1   =(1 - Fm)* kl_sl*Cl0;//(1+lam_u*(kssd_a + kssd_sm + kssd_s));
            part2   =Cssd0* lam_u * (kssd_a + kssd_sm + kssd_s);//(1+lam_u*(kssd_a + kssd_sm + kssd_s));
            dCssd   =part1-part2;
            Cssd1   = Cssd0+dCssd;                 // for surface structural litter
            Nssd0   = Cssd0/CNssd;                 //ǰһʱ�̸ÿ�ĵ���  
             //��ʱ�̸ÿ�ĵ���
			Nssd1   =Cssd0/CNssd + (1 - Fm)* kl_sl*Cl0/CNl///(1+lam_u*(kssd_a + kssd_sm + kssd_s))
                     -Cssd0/CNssd* lam_u * (kssd_a + kssd_sm + kssd_s);///(1+lam_u*(kssd_a + kssd_sm + kssd_s));

             part1   =Fm* kl_sl * Cl0;///(1+lam_u*(ksmd_a + ksmd_sm));
             part2   = Csmd0* lam_u * (ksmd_a + ksmd_sm);///(1+lam_u*(ksmd_a + ksmd_sm));
             dCsmd   =part1-part2;
             Csmd1   = Csmd0+dCsmd;                 // for surface metabolic litter
             Nsmd0   =Csmd0/CNsmd;    //ǰһʱ�̸ÿ�ĵ���
             //��ʱ�̸ÿ�ĵ���
			 Nsmd1   =Csmd0/CNsmd+ Fm* kl_sl * Cl0/CNl//(1+lam_u*(ksmd_a + ksmd_sm))
                     -Csmd0/CNsmd* lam_u * (ksmd_a + ksmd_sm);//(1+lam_u*(ksmd_a + ksmd_sm));

               
             part1=(1 - Fm)* kfr_fl* Cfr0;//(1+lam_d*(kfsd_a + kfsd_m + kfsd_s));
             part2=Cfsd0* lam_d * (kfsd_a + kfsd_m + kfsd_s);//(1+lam_d*(kfsd_a + kfsd_m + kfsd_s));
             dCfsd =part1-part2;
             Cfsd1= Cfsd0+dCfsd;                  //for soil strutural litter pool
  
             Nfsd0=Cfsd0/CNfsd; //ǰһʱ�̸ÿ�ĵ���
             //��ʱ�̸ÿ�ĵ���
			 Nfsd1=Cfsd0/CNfsd+ (1 - Fm)* kfr_fl* Cfr0/CNfr//(1+lam_d*(kfsd_a + kfsd_m + kfsd_s))
                     - Cfsd0/CNfsd* lam_d * (kfsd_a + kfsd_m + kfsd_s);//(1+lam_d*(kfsd_a + kfsd_m + kfsd_s));

  
             part1=Fm * kfr_fl * Cfr0;//(1+lam_d * (kfmd_a + kfmd_m)); 
             part2=lam_d * (kfmd_a + kfmd_m)* Cfmd0;//(1+lam_d * (kfmd_a + kfmd_m));
             dCfmd=part1-part2;
             Cfmd1= Cfmd0+dCfmd;                   // for soil metabolic pool
             Nfmd0= Cfmd0/CNfmd;  //ǰһʱ�̸ÿ�ĵ���
             //��ʱ�̸ÿ�ĵ���
			 Nfmd1= Cfmd0/CNfmd+Fm * kfr_fl * Cfr0/CNfr//(1+lam_d * (kfmd_a + kfmd_m))
                    -lam_d * (kfmd_a + kfmd_m)* Cfmd0/CNfmd;//(1+lam_d * (kfmd_a + kfmd_m));
			
             part1=lam_u*(Cssd0*kssd_sm+Csmd0*ksmd_sm);//(1+lam_u*(ksm_a+ksm_s));
             part2=lam_u*Csm0*(ksm_a+ksm_s);//(1+lam_u*(ksm_a+ksm_s));
             dCsm=part1-part2;
             Csm1=Csm0+dCsm;                       // for surface microbe pool
             Nsm0=Csm0/CNsm ;     //ǰһʱ�̸ÿ�ĵ���
             //��ʱ�̸ÿ�ĵ���
			 Nsm1=Csm0/CNsm + lam_u*(Cssd0/CNssd*kssd_sm+Csmd0/CNsmd*ksmd_sm)//(1+lam_u*(ksm_a+ksm_s))
                    - lam_u*Csm0/CNsm*(ksm_a+ksm_s);//(1+lam_u*(ksm_a+ksm_s));


             part1=(lam_d * (kfsd_m * Cfsd0+kfmd_m*Cfmd0 + Ccd0 * kcd_m) +lam_d*(Cs0*ks_m+Cp0 * kp_m));//(1+lam_d*(km_a +  km_s +km_p));				
             part2=Cm0 * lam_d*(km_a +  km_s +km_p);//(1+lam_d*(km_a +  km_s +km_p));
             dCm=part1-part2;
             Cm1=Cm0+dCm;                          // for soil microbe pool
             Nm0=Cm0/CNm;                          //ǰһʱ�̸ÿ�ĵ���
             //��ʱ�̸ÿ�ĵ���
             Nm1=Cm0/CNm +(lam_d * (kfsd_m * Cfsd0/CNfsd+kfmd_m*Cfmd0/CNfmd+Ccd0/CNcd* kcd_m)+lam_d*(Cs0/CNs*ks_m+Cp0/CNp* kp_m))
	                     //(1+lam_d*(km_a +  km_s +km_p))
                         -Cm0/CNm * lam_d*(km_a +  km_s +km_p);//(1+lam_d*(km_a +  km_s +km_p));
     
             part1=(lam_d*(Cm0*km_s + Ccd0 * kcd_s +Cfsd0*kfsd_s )+ lam_u* (Csm0*ksm_s + Cssd0*kssd_s))
	               ;//(1+lam_d *( ks_a + ks_p+ks_m));
             part2=Cs0* lam_d *( ks_a + ks_p+ks_m);//(1+lam_d *( ks_a + ks_p+ks_m));
             dCs=part1-part2;
             Cs1=Cs0+dCs;                          // for slow carbon pool
             Ns0=Cs0/CNs;        //ǰһʱ�̸ÿ�ĵ���
              //��ʱ�̸ÿ�ĵ���
			 Ns1=Cs0/CNs+(lam_d*(Cm0/CNm*km_s + Ccd0/CNcd * kcd_s +Cfsd0/CNfsd*kfsd_s )+ lam_u* (Csm0/CNsm*ksm_s+Cssd0/CNssd*kssd_s))
	           //(1+lam_d *( ks_a + ks_p+ks_m))
                -Cs0/CNs* lam_d *( ks_a + ks_p+ks_m);//(1+lam_d *( ks_a + ks_p+ks_m));
   				
             dCp =(lam_d *( km_p * Cm0 + ks_p * Cs0))//(1+lam_d * (kp_m + kp_a ))
		                 - Cp0*lam_d * (kp_m + kp_a );//(1+lam_d * (kp_m + kp_a ));
             Cp1=Cp0+dCp;                         // for passive carbon pool.
             Np0=Cp0/CNp;  // //ǰһʱ�̸ÿ�ĵ���		
             // //��ʱ�̸ÿ�ĵ���
			 Np1=Cp0/CNp+lam_d *( km_p * Cm0/CNm + ks_p * Cs0/CNs)//(1+lam_d * (kp_m + kp_a )) 
                        - Cp0/CNp*lam_d * (kp_m + kp_a );//(1+lam_d * (kp_m + kp_a ));

            A1=Ncd0+Nssd0+Nsmd0+Nfsd0+Nfmd0+Nsm0+Nm0+Ns0+Np0;  
            A2=Ncd1+Nssd1+Nsmd1+Nfsd1+Nfmd1+Nsm1+Nm1+Ns1+Np1;

  
       Nmin  =(float)(lam_u * (Cssd0* kssd_a/CNssd+Csmd0* ksmd_a/CNsmd+Csm0*ksm_a/CNsm) 
                      +lam_d*(Ccd0 *kcd_a/CNcd+Cfsd0* kfsd_a/CNfsd+Cfmd0 * kfmd_a/CNfmd 
                      +Cm0*km_a/CNm+Cs0*ks_a/CNs+Cp0*kp_a/CNp));       
   
       totalNmin=totalNmin+ Nmin;  
    	  
	 	Nav=Nav+Nmin;//+Ndep/365.0;                                  //����ÿ��ĵ���������Ϊ2.0 g C N/a
        if(Nav<0) Nav=0;  
       
		Sc=(Ccd1+Cssd1+Csmd1+Cfsd1+Cfmd1+Csm1+Cm1+Cs1+Cp1);
   	 		 
		Sn=Nav;
        b2=Sn/600; 
		if(b2>1.0) b2=1.0;
	 
 	NP=120*b2*exp(-8.0/100000*Sc);
        
		u1=40.8+0.01*Tmean_m[day]-0.002*Tmean_m[day]*Tmean_m[day];
		u2=0.738-0.002*Tmean_m[day];
        u3=97.42-2.504*log(NP);
        a3=Tmean_m[day]+273.37;
        a1=exp(u1-u3/(0.00831*a3));
		a2=1+exp((u2*a3-205.9)/(0.00831*a3));
        Nt=a1/a2*14.0/1000.0;//+2.0/365.0;   //g N/m-2/d
		totalNup=totalNup+Nt;
     
		Nav=(Nav-Nt);
        if(Nav<0)  Nav=0;
	    totalN=totalN+Nt; 

totalNup=totalNup+Nt;

       Rnpp_wood=coef1[0]; Rnpp_Ccroot=coef1[1];
       Rnpp_leaf=coef1[2]; Rnpp_froot=coef1[3]; 
 
       Cw1   = Cw0 +  ((NPPd_m[day]-NPPd_m[day-1])*Rnpp_wood- Cw0*kw_cd);//(1+kw_cd); 
       //Nwood0   = Cw0/CNw;       //+Nt*Rnpp_wood/1000.0  ;///(1+kw_cd); //+
       Nwood1   = Nwood0-Cw0*kw_cd/CNw;//(1+kw_cd) ;
       
	   Ccr1 =  Ccr0+  ((NPPd_m[day]-NPPd_m[day-1])*Rnpp_Ccroot- Ccr0*kcr_cd);//(1+kcr_cd);  
       //NCcroot0  =  Ccr0/CNw;   //+Nt*Rnpp_Ccroot/1000.0; //+
       NCcroot1  =  NCcroot0- Ccr0*kcr_cd/CNw;//(1+kcr_cd);
	   
	  //a1=(NPPd_m[day]-NPPd_m[day-1])*Rnpp_wood/CNw*(CNw/125.0);
      // a2=(NPPd_m[day]-NPPd_m[day-1])*Rnpp_Ccroot/CNw*(CNw/125.0); 
       
       a1=Rnpp_wood*(CNw/coef1[47]);
       a2=Rnpp_Ccroot*(CNw/coef1[47]); 


	   Cl1 = Cl0  +  ((NPPd_m[day]-NPPd_m[day-1])*Rnpp_leaf-Cl0*kl_sl);//(1+kl_sl);
      // Nleaf0   =Cl0/CNl;        // +Nt*Rnpp_leaf/1000.0;///(1+kl_sl);    //+
       Nleaf1   =Nleaf0- Cl0*kl_sl/CNl;//(1+kl_sl);        // +Nt*Rnpp_leaf/1000.0;///(1+kl_sl);    //+

	   Cfr1     = Cfr0+  ((NPPd_m[day]-NPPd_m[day-1])*Rnpp_froot-Cfr0*kfr_fl);//(1+kfr_fl);        
      // Nfroot0  = Cfr0/CNfr;  //+Nt*Rnpp_froot/1000.0;///(1+kfr_fl);   //+
       Nfroot1  = Nfroot0- Cfr0*kfr_fl/CNfr;//(1+kfr_fl);  
      
	  // a4=(NPPd_m[day]-NPPd_m[day-1])*Rnpp_leaf/CNl*(CNl/56);
       //a5=(NPPd_m[day]-NPPd_m[day-1])*Rnpp_froot/CNl*(CNl/56);
       
        a3=Rnpp_leaf*(CNl/coef1[46]);
        a4=Rnpp_froot*(CNl/coef1[46]);
       


	   Nwood1 =Nwood1 +Nt*a1/(a1+a2+a3+a4);
	   NCcroot1=NCcroot1+Nt*a2/(a1+a2+a3+a4);
	   Nleaf1 =Nleaf1 +Nt*a3/(a1+a2+a3+a4);
	   Nfroot1=Nfroot1+Nt*a4/(a1+a2+a3+a4);
	     
	  // A3=A1+Nwood0+NCcroot0+Nleaf0+Nfroot0;  //ǰһ��ʱ�䲽����ֲ����������
	  // A4=A2+Nwood1+NCcroot1+Nleaf1+Nfroot1;  //��ǰһ��ʱ�䲽����ֲ����������
     //  A5=Nwood1+NCcroot1+Nleaf1+Nfroot1; 
	   //TN1=A4+Nav;
	  // TN0=A3+Nav0;
	 
	   //Nav0=Nav;
	   CNw=(Cw1+Ccr1)/( Nwood1 +NCcroot1);
       CNl=(Cl1+Cfr1)/( Nleaf1 +Nfroot1);
       CNfr=CNl;
	   
     if(CNw>(coef1[47]*2)) CNw=coef1[47]*2;
     if(CNw<(coef1[47]*0.5)) CNw=coef1[47]*0.5;
    
	 if(CNl>(coef1[46]*2)) CNl=coef1[46]*2;
     if(CNl<(coef1[46]*0.5)) CNl=coef1[46]*0.5;


       Nwood0 =Nwood1;
	   NCcroot0=NCcroot1;
	   Nleaf0 =Nleaf1;
	   Nfroot0=Nfroot1;

       CNl_av=CNl_av+CNl;

    if(Ncd1<0.00001) Ncd1=0.00001; 
 	if(Nssd1<0.00001) Nssd1=0.00001;   
    if(Nsmd1<0.00001) Nsmd1=0.00001;   

    if(Nfsd1<0.00001) Nfsd1=0.00001;   
    if(Nfmd1<0.00001) Nfmd1=0.00001;   
	if(Nsm1<0.00001) Nsm1=0.00001;   
	if(Nm1<0.00001) Nm1=0.00001;   
	if(Ns1<0.00001) Ns1=0.00001;   
	if(Np1<0.00001) Np1=0.00001;    


	 CNcd=Ccd1/Ncd1;    CNssd=Cssd1/Nssd1;  CNsmd=Csmd1/Nsmd1;  CNfsd=Cfsd1/Nfsd1;
     CNfmd=Cfmd1/Nfmd1; CNsm=Csm1/Nsm1;     CNm=Cm1/Nm1;        CNs=Cs1/Ns1;
     CNp=Cp1/Np1; 

 
  //fprintf(f1,"%f, %f,%8.5f,%8.5f,%8.5f,%8.5f, %8.5f, %8.5f,%8.5f,%8.5f,%8.5f,%8.5f,%8.5f, %8.5f,%8.5f, %8.5f,%8.5f\n",
  // a1,a2,Nav+A4,totalNup,totalNmin*1000,CNw,CNl ,a4,a5, CNssd,CNsmd,CNfsd, CNfmd,CNsm,CNm, CNs,  CNp); 
 
} // =======================================the end of day loop===============================

} // the end of year loop


 

	initialValues[0] =(float) NPP;
	initialValues[1] =(float) Cw1;
	initialValues[2] =(float) Ccr1;
	initialValues[3] =(float) Cl1;
	initialValues[4] =(float) Cfr1; /* 1 to 4 is for vegetation*/
	
	initialValues[5] =(float) (Ccd1);  /* cd means coarse structural detritus*/
	initialValues[6] =(float) (Cssd1); /*ssd means surface structural detritus*/
	initialValues[7] =(float) (Csmd1); /*smd means surface metabolic  detritus*/
	initialValues[8] =(float) (Cfsd1); /*fsd means fine root strutual detritus*/
	initialValues[9] =(float) (Cfmd1); /*fmd means fine root metabolic detritus*/
	
	initialValues[10] =(float) (Csm1); /* sm is surface microbe*/
	initialValues[11] =(float) (Cm1);   /* soil microbe*/
    initialValues[12] =(float) (Cs1);   /* s represents slow carbon*/ 
 	initialValues[13] =(float) (Cp1);   /* p represents passive carbon*/

	
	initialValues[14] =(float) Fm;
    initialValues[15] =(float) 0;
	initialValues[16] =(float) CNcd;
	initialValues[17] =(float) CNssd;
	initialValues[18] =(float) CNsmd;
	initialValues[19] =(float) CNfsd;
	initialValues[20] =(float) CNfmd;
	initialValues[21] =(float) CNsm;
    initialValues[22] =(float) CNm;
    initialValues[23] =(float) CNs;
    initialValues[24] =(float) CNp;

	initialValues[25] =(float) CNw;
	initialValues[26] =(float) CNl;
	initialValues[27] =(float) Nav;
	initialValues[28] =(float) CNl_av;
 
return;    
}

