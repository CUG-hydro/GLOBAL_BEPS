#include "hip/hip_runtime.h"
/*************************************************************************
  Program:     melt.c
  --------
  Description:
  ----------- 
	Compute snowmelt.
  Details:
  -------
	input:
	b[6]: snow melt temp. coefficient (m/C/day)(=0.001) 
	b[8]: snow aborptivity (=0.12)
	x[1]: snowpack (m)
	z[8]: shortwave radiation at forest floor (kj/m2/d)
	z[14]: daytime average temperature (C deg)		
	output:
	g[3]: total melt (m)
	g[8]: temperature melt (m)
	g[9]: radiation melt (m)
	g[13]: sublimation from snow (m)
***************************************************************************
  CCRS (EMS/Applications Division)
  Written by:   J. Liu     
  Modified by: X.F. Feng
  Last update:  July 2003
*****************************************************************************/
#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 

/*float min(float a,float b)
{
	if (a<b)
		return a;
	return b;
}
float max(float a,float b)
{
	if (a>b)
		return a;
	return b;
}*/
__device__ void gpumelt (
    float b[],float g[],float x[],float z[])
    {
	float lh_fus=3.5E+5;
//	float lh_evp=2.5E+6; 
	float lh_sub=2.8E+6;   ///?????????3.25
	short int lc_p;
/* Snowmelt Temp Coeff is different for different land cover type  X.F. July */

	lc_p=(int short )z[23];
    switch(lc_p)
	{

	/* for LCTM, reference about Snowmelt Temp Coffe. is  G.W. Kite et al. 1988 X.F. July */

	case 4: case 10: case 5:              /* conifer X.F. Aug. */ // conifer ;zhoulei 4.1
		b[6]=0.001/RTIMES;		           /* unit m/degree C/d  */     //2018-03-05: 0.0022 to 0.0010
		break;

	case 1: case 7:case 8: case 2:case 3:            /* braodleaf forest,for LCTM */ // braodleaf forest zhoulei 4.1
        b[6]=0.0010/RTIMES;		           /* unit m/degree C/d  */ 
		break;

	case 6: case 9:                                /*mixe ,for LCTM */ //mixed zhoulei 4.1
		b[6]=0.0015/RTIMES;		           /* unit m/degree C/d  */ 
		break;
		
	case 11:case 12:case 14:  /*closedshrub, pasture and crop for LCTM */ //closed shrub &grassland &cropland ;zhoulei 4.1
		b[6]=0.0020/RTIMES;		           /* unit m/degree C/d  */ 
		break;

	case 13:  case 16:case 17: case 18: //zfm3.25  10,12,14
        b[6]=0.001/RTIMES;
		break;

	default:                             /* other land cover */
	b[6]=0.006/RTIMES;
	}



/*	b[8]: snow absorption */
	b[8]=0.3/3.0*0.1;


    if (z[14] > 0.0) { 
	    g[8] = z[8]*b[8]/lh_fus;	/* radiation  melt */ 
		g[9] = b[6] * z[14];		/* temperature melt */ 
		g[3] = __min (x[1], (g[8]+g[9]));
		g[13] =0.0;
	}
    else { 
		g[8]=0.0;
		g[9]=0.0;
		g[3]=0.0;
 	
		g[13] = (z[8]-z[19])*b[8]/lh_sub*0.001;  //2012-12-04
       	//	g[13]=z[19]*b[8]/lh_sub;
        g[13]=__max(0,g[13]);        
		g[13] = __min(x[1],g[13] );
		

	}


	return;

	}


