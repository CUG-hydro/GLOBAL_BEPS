#include "hip/hip_runtime.h"
/*************************************************************************
  Program:     model.c
  --------
  Description:
  ----------- 
	Simulate carbon and water cycles for a single pixel on a day.
***************************************************************************
  CCRS (EMS/Applications Division)
  Written by:   J. Liu      
  Modified by:  X.F. Feng
  Last update:  July 2003
*****************************************************************************/
   

#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 

    
__device__  void gpumodel(float CNl,long jday,int long pix,long index,float lat_p,float lai_p,int short  lc_p,float CI,float TI,float sw_o1,float sw_o2,float sw_o3,
                           struct hy_c1 HY1[],struct hy_c2 HY2[],struct hy_c3 HY3[],  float  x[],  float  b[], short climatedata[], 
						   struct     xvalue xx[], float co,float*soilw1_new,float* soilw2_new,float* soilw3_new,float *tmean,float *t_d,float *t_n,float *d_l)
    
	{
    
	//float *sw_n1,*sw_n2,*sw_n3;	
	float z[SIZEZ];
    float g[SIZEG];
    float  swn1,swn2,swn3;
    float  awc_p=0.5;
    float Tmean;
 //lixuansong
    if(jday==jday_start)
    {
    gpureadxx(pix,x,xx);	                	/* read snowpack and others */
	gpusetx(lc_p,awc_p,x,b);	                /* set x[2]-x[10] */ 		
    }

    /* read & compute forest-bgc daily climate data */ 
    else
    {
		gpureadxx(pix,x,xx);
    }

	gpuzcomp(jday,pix,lat_p,lc_p,CI,TI,awc_p,lai_p,b,x,z, climatedata,&Tmean); 

    *tmean=(z[4]+z[5])/2.0;

       *t_d=z[14];
	   *t_n=z[15];
	   *d_l=z[18];



	/* daily water and carbon dynamics */ 

	gpudoflux(CNl,pix,index,b,g,x,z,co,TI,sw_o1,sw_o2,sw_o3,HY1,HY2,HY3,&swn1,&swn2,&swn3); 


    (*soilw1_new)=swn1;(*soilw2_new)=swn2;(*soilw3_new)=swn3;


	gpuwritexx(pix,x,xx); 


/***********************************************************************/
//	free(g);
   	return;		
} 
 

  __device__  void gpusetx( int short lc_p, float awc_p, float x[], float b[])
    {  
    //x[2]=awc_p*1.1;  

/* Set x[3] to x[7], x[11]-x[16] to zero */ 

    x[3]=0.0; 
    x[4]=0.0; 
    x[5]=0.0; 
    x[6]=0.0; 
    x[7]=0.0; 
  
    x[11]=0.0; 
    x[12]=0.0;
	x[13]=0.0;
	x[14]=0.0;
	x[15]=0.0;
	x[16]=0.0;
	x[17]=0.0;
	x[18]=0.0;
    x[19]=0.0;

	 switch(lc_p)
    {
		 

//******* for MODIS, July ********* 
	 case 4: case 10:  //ENF
	x[9]=x[9]/b[1]*0.081;// 	x[9]=x[9]/b[1]*0.081;annavg livewood:annmax leaf(kgC/kgC) .0.081
        //if (x[9]==0) x[9]=30;	
   	x[10]=0.2317*x[9];					// root biomass in t/h b 

	break;
 case 1: case 7: case 8://EBF

		x[9]=x[9]/b[1]*0.162;// *0.162;annavg livewood:annmax leaf(kgC/kgC) 0.162  
		x[10]=exp(0.359)*pow(x[9],(float)0.639);

	break;
case 5: //DNF

		x[9]=x[9]/b[1]*0.152;// annavg livewood:annmax leaf(kgC/kgC) 0.152
		x[10]=0.2317*x[9];	
	
	break;
case 2: case 3://DBF
	
		x[9]=x[9]/b[1]*0.203;// annavg livewood:annmax leaf(kgC/kgC) 0.203
		x[10]=exp(0.359)*pow(x[9],(float)0.639);	
	
	break;

case 6: case 9: //MF
	
		x[9]=x[9]/b[1]*0.132;// annavg livewood:annmax leaf(kgC/kgC) 0.132
		x[10]=0.5*(0.2317*x[9]+exp(0.359)*pow(x[9],(float)0.639));	
	
	break;
 
case 11:case 12:case 14:  // 8:closed shrub;9:open shrub; 7: woody savanna;6:savannas;11: permanent wetlands
	
		x[9]=x[9]/b[1]*0.040;// annavg livewood:annmax leaf(kgC/kgC) 0.040
		x[10]=0.2317*x[9];
	
	break;

default: 
	
		x[9]=1.0;
		x[10]=0.2; // defult root biomass //zhoulei 4.1

    }
} 



