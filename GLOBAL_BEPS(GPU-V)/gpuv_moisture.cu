#include "hip/hip_runtime.h"

#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 


/* This subroutine is to estimate the vertical redistribution of soil water*/
__device__ void gpuv_moisture(int long pix, float depth3, float B[], float K[], float Psi[],
                              struct hy_c1 HY1[], struct hy_c2 HY2[], struct hy_c3 HY3[], float *theta_o, float *theta_n1,float *theta_n2, float *theta_n3)
{

float X[10][10],Y[10];
float x1[10],x2[10],x3[10];
//float Flux[5];// downward water across layer boundaries 
float m[5];
float n[5];

float bperm;         // =0 for impermeable (no drainage)  lower bc; =1 for free drainage lower bc
float hsoim[5]; //vertical distance between cernters of layers 
float wsoim[5]; // interpolated moisture values at layer boundaries 
float wsoia[5]; // interpolated moisture values at layer boundaries  
float wsoib[5]; // interpolated moisture values at layer boundaries  

float weim[5]; // interpolated moisture values at layer boundaries  
float weip[5]; // interpolated moisture values at layer boundaries  

int i;  //intermideate variable
float e[5],f[5],g[5];
float a[5],b[5],zbex;
float bwn1[5],bwn[5];
float rimp ;// the implicit f
//float rhs[10];
float dt;

//float rhow=1000.0; //the density of water
int km1;

int layer1;
float surface_plus;
float kk=1;
float d_n,d_o;
float SW[5], SS[5],depth[5],SF[5];//
float interval;
float thetan[5];


bperm=0;
interval=1.0;
layer1=layer;
rimp=1.0;//1: complete implicit solution, 0 explicit solution

SW[1]=HY1[pix].WP; SF[1]=HY1[pix].FC;  SS[1]=HY1[pix].PR;
SW[2]=HY2[pix].WP; SF[2]=HY2[pix].FC;  SS[2]=HY2[pix].PR;
SW[3]=HY3[pix].WP; SF[3]=HY3[pix].FC;  SS[3]=HY3[pix].PR;
depth[1]=depth1;depth[2]=depth2; depth[3]=depth3;


  hsoim[1]=0.5*depth[1];   weim[1]=0;   weip[1]=1.0;  wsoim[1]=theta_o[1];
  wsoia[1]=__min(wsoim[1],1.0); 
  wsoib[1]=__min(wsoim[1],1.0); 

  for(i=2;i<=layer1;i++){
  hsoim[i]=0.5*(depth[i-1]+depth[i]);
  //weim[i]=0.5*depth[i-1]/hsoim[i];    //ORGIS:   weim[i]=0.5*depth[i]/hsoim[i];  
 
  weim[i]=0.5*depth[i]/hsoim[i];    //2018-03-02�ָ� 
  weip[i]=1-weim[i];




  wsoim[i]=weim[i]*theta_o[i-1]+weip[i]* theta_o[i];
  wsoia[i]=__min(wsoim[i],1.0);
  wsoib[i]=__min(wsoim[i],1.0);
  }  // the end of i loop
  
  i=layer1+1;
  
  hsoim[i]=0.5 *depth[i-1];
  weim[i]=1.0;
  weip[1]=0;
   
  wsoim[i]=theta_o[i-1];
  wsoia[i]=__min(wsoim[i],1.0);
  wsoib[i]=__min(wsoim[i],1.0);

   
 e[1]=0; f[1]=0; g[1]=0;
 	  
 for (i=2;i<=layer1;i++) {
 a[i]=weim[i]*K[i-1]+weip[i]*K[i];
 b[i]=weim[i]*K[i-1]*Psi[i-1]*B[i-1]+weip[i]*K[i]*Psi[i]*B[i];
 zbex=weim[i]*B[i-1]+weip[i]*B[i];   // B 
 m[i]=2*zbex+3;  
 n[i]=zbex+2; 

 bwn1[i]=b[i]*pow(wsoib[i],(n[i]-1));

 bwn[i]=bwn1[i]*wsoib[i];



 e[i]=a[i]*(-1+rimp*m[i])*pow(wsoia[i],m[i])
	 +((1-rimp)*bwn[i]-rimp*n[i]*bwn1[i]*wsoib[i])*(theta_o[i]-theta_o[i-1])/hsoim[i];
 
 f[i]=-rimp*a[i]*m[i]*pow(wsoia[i],(m[i]-1))
	  +rimp*n[i]*bwn1[i]*(theta_o[i]-theta_o[i-1])/hsoim[i];
 



 /*
 e[i]=a[i]*(1-rimp*m[i])*pow(wsoia[i],m[i])
	 +((1-rimp)*bwn[i]-rimp*n[i]*bwn1[i]*wsoib[i])*(theta_o[i]-theta_o[i-1])/hsoim[i];

 f[i]=rimp*a[i]*m[i]*pow(wsoia[i],(m[i]-1))
	 +rimp*n[i]*bwn1[i]*(theta_o[i]-theta_o[i-1])/hsoim[i];


*/

 g[i]= rimp*bwn[i]; 
	
 } // the end of i loop

i=layer1+1;   // for the deepest layer
a[i]= K[i-1];
b[i]= K[i-1]*Psi[i-1]*B[i-1];
m[i]= 2*B[i-1]+3;
n[i]= B[i-1]+2;
d_o=rimp*a[i]*m[i]*pow(wsoia[i],m[i]-1)*theta_o[i-1]*bperm;
d_n=rimp*a[i]*m[i]*pow(wsoia[i],m[i]-1)*bperm;

e[i]=0;//-(a[i]*pow(wsoia[i],m[i])*bperm-d_o*kk); 
 //the considertion of vertical drainage from the bottom of soil profile
f[i]=0;
g[i]=0;



/*********************************************************************************************************/
for (i=1;i<=layer1;i++){
dt=interval/(SS[i]*depth[i]);
X[i][1]=dt*(f[i]*0.5*depth[i]/hsoim[i]-g[i]/hsoim[i]);
Y[i]=theta_o[i]+dt*(e[i+1]-e[i]); 



if(i<layer1){
km1=__max(i-1,1);
X[i][2]=1+dt*(-f[i+1]*0.5*depth[i+1]/hsoim[i+1]+f[i]*0.5*depth[km1]/hsoim[i]+g[i+1]/hsoim[i+1]+g[i]/hsoim[i]);
X[i][3]=  dt*(-f[i+1]*0.5*depth[i]/hsoim[i+1]-g[i+1]/hsoim[i+1]);
}  // end of i<layer

else if (i==layer){     //for the deepest layer 
dt=interval/(SS[i]*depth[i]);
X[i][2]=1.0+dt*(-f[i+1]+f[i]*0.5*depth[i-1]/hsoim[i]+g[i]/hsoim[i]);
X[i][2]=X[i][2]+dt*d_n*kk;           //*rimp*m[i]*a[i+1]*pow(wsoia[i+1],m[i]-1)*bperm;
//

X[i][3]=0;
}  // the end of else
}  //the end  of i loop

for(i=0;i<10;i++){
x1[i]=X[1][i];
x2[i]=X[2][i];
x3[i]=X[3][i];
}



 
 gpusolution11(thetan,x1,x2,x3,Y);

/*
for(i=layer1;i>=1;i--){
if(thetan[i]>1.0) {
if(i>1){
thetan[i-1]=thetan[i-1]+(thetan[i]-1)*SS[i]*depth[i]/(SS[i-1]*depth[i-1]);
thetan[i]=1.0;
	}
else {
//surface_plus=(theta_n[i]-1.0)*SS[i]*depth[i];	
//theta_n[i]=1.0;	
}

}   // the end of theta_n[i]>1.0

//if(theta_n[i]<(SW[i]/SS[i]*0.75))theta_n[i]=SW[i]/SS[i]*0.75;  
//ϵ��0.5Ϊ2012-10-15�ռ�    //2017-06-02��������ĸ����������⣬�ᵼ��ģ�������ˮ��ƫ�ߣ� ȡ��������  

}  // the end of i loop




if(thetan[1]>1.0){
surface_plus=(thetan[1]-1.0)*SS[1]*depth[1];
thetan[1]=1.0;
}
else  surface_plus=0;

*/


*theta_n1=thetan[1];

*theta_n2=thetan[2];

*theta_n3=thetan[3];

}  // the end of subroutine