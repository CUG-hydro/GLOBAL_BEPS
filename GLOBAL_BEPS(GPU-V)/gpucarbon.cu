#include "hip/hip_runtime.h"
/*************************************************************************
  Program:     carbon.c
  --------
  Description:
  ----------- 
	Estimate photosythesis and plant respiration.
  Details:
  -------	
 	Use Farquhar's model for photosynthesis calculation
	Input:
	b[25]: Q10 for leaf
	b[28]: Q10 for stem	
	b[29]: Q10 for root	

	b[19]: leaf maintenance res. coe 
	b[20]: stem maintenance res. coe  	
	b[21]: coarse root maintenance res. coe
	b[24]: fine root maintenance res. coe 
	x[8]: leafC
	x[9]: stemC
	x[10]: rootC
	z[4]:Tmax
	z[5]:Tmin
	z[10]: LAI	
	z[14]: daylight average temperature
	z[18]: daylength
	Output
	g[25]: leaf Rm(forest) or total Rm (other land cover) in kg C/m2/day
	g[30]stem Rm     //zfm
***************************************************************************
  CCRS (EMS/Applications Division)
  Wrintten by: 	J. Liu
  Modified by:  X.F. Feng
  Last update:  July 2003
*****************************************************************************/
#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 

//#include "beps.h" 

/*float max(float a,float b)
{
	if (a>b)
		return a;
	return b;
}*/
__device__ void gpucarbon(float CNl,long index,float b[],float g[],float x[],float z[],float co,float TI)
{
  float exponent,exponent_n,ratio_d;    //ratio_d_n; ���쳤�ȱ�����
//  float sapw_max=0.1;	
  float rf25, pr;		              /* for Rm with Bonan (1995) */
  int short lc_p;
//  float ratio_froot=1;
 
  float tb=15;                       //��������Ĳο��¶� 
//  float a=0.0;
  float a1,a2;
   float Gsunlit_o, Gshaded_o, Gsunlit_n, Gshaded_n, slope, D0;
int jj;

float cs1,cs2,cbl,cc;

  //JUW  2017-10-15  ��  
//b[25]=b[25];
//b[28]=b[28];
//b[29]=b[29];
a2=z[45];
if(a2<0) a2=0;
/*
 b[25]=2.62-0.023*a2;
 b[28]=2.52-0.046*a2;
 b[29]=2.52-0.046*a2;
*/
/*
b[25]=2.42-0.03*a2;

if(b[25]<1.9) b[25] =1.9;



b[28]=2.32-0.03*a2;
if(b[28]<1.8)b[28]=1.8;

b[29]=2.32-0.03*a2;

if(b[29]<1.8) b[29]=1.8;
*/

 
b[25]=2.42-0.02*a2;

if(b[25]<1.9) b[25] =1.9;


b[28]=2.32-0.02*a2;
if(b[28]<1.80)b[28]=1.80;

b[29]=2.32-0.02*a2;

if(b[29]<1.80) b[29]=1.80;
 

if (index==1) {
 
// RESPIRATION	����

	//===========================================================����һ���ƽ���������=======================================================

	if(z[23]==13) slope=8; 
	else          slope=10; 
    D0=0.005;


/*
slope=12.7-0.207*z[14];
if(slope>10) slope=10;
if(slope<6.9) slope=6.9;
 

slope=10;

D0=(142.4-4.8*z[14]);
if(D0>80) D0=80;
if(D0<8)  D0=8;
*/

    for(jj=1;jj<=5;jj++){
	gpufarq_psn(CNl,b,z,g,co);    

  //���׵���:umol/ m2/s  //mol m2/s
	//Gsunlit_o=slope*g[28]*g[80]/co/(1.0+z[16]/D0)+0.001; //z[16]:VPD(0.1kPa"     
   // Gshaded_o=slope*g[29]*g[80]/co/(1.0+z[16]/D0)+0.001; //z[16]:VPD(0.1kPa"



	Gsunlit_o=slope*g[28]*g[80]*z[46]/co+D0; //z[16]:VPD(0.1kPa"     
	Gshaded_o=slope*g[29]*g[80]*z[46]/co+D0; //z[16]:VPD(0.1kPa"



 //���׵���:m/s
	cs1=Gsunlit_o*8.314*(z[14]+273.14)/101350;   
			
	cs2=Gshaded_o*8.314*(z[14]+273.14)/101350;
	

	cbl=0.08;
	// leaf cuticular conductance 
	cc=0.00005;

	g[20] =cbl*(cc+cs1)/(cbl+cc+cs1);   //��Ҷ                   //2013��9��11�ոģ� ԭ������ѹ�Ͳ���

	g[21] =cbl*(cc+cs2)/(cbl+cc+cs2);   //��Ҷ     //�øù�ʽ�� �� 

}

 


    //һ���GPP  //convert umol C/m2/s --> kg C/m2/period ��λת����
    //z[32]: Sunlit LAI; z[33]: Shaded LAI, z[18]: һ����ճ���
	
    g[24] = (g[28]*z[32]+g[29]*z[33])*z[18]*12.0/1000000000.0;
   //========================================================================================================================================

	//leaf day time maintenance respiration //a equation below can overwrite this one  
	x[8]= z[10]/b[1];
	
	
	exponent=(z[14]-tb)/10.0;         //z[14] �����¶�
    exponent_n=(z[15]-tb)/10.0;       //z[15] ҹ���¶�   
    ratio_d=z[18]/86400;              //�����ճ���ռһ��ı���


	//g[23] =x[8]*b[19]*pow(b[25],exponent);//*z[18]/86400;   //Ҷ����ά���Ժ���

//g[23] =x[8]/1000.0*b[19]*pow(b[25],exponent);//*z[18]/86400;   //Ҷ����ά���Ժ���

g[23]=0.0;

	if(g[23]<0.0)  g[23]=0.0;
//Rm calculation, Bonan (1995), z[10]: LAI; z[4]: Tmax; z[5]:Tmin 

	lc_p=( int short)z[23];    //�ر�������
	switch(lc_p)
	{
	case 4: case 10://Evergreen needle forest, ������Ҷ��
	{
		//x[8]= z[10]/b[1];      //����LAI��SLA�õ��ڲ�������     //2013-08-13
		
	     //leaf night time maintenance respiration, ����ҶƬ��ά���Ժ���
	 
		//b[19],�ο��������ʣ���Readb();b[25]: Q10: ��read_init�и���
		
		g[25] =0.45*x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));  //2018-04-25: 0.4 to 0.45
		
      //  g[25] = x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 

		//g[25] = __max( g[25], 0.0);
       if(g[25]<0.0) g[25]=0.0; 

		//stem maintenance respiration�����ɺ���
    
		//x[9],�߲�ľ����������model(); b[20]���ο���������,��readb()��b[28],Q10,��read_init();
		//exponent=(z[15]-tb)/10.0;
	 	//g[30] = x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���
		
		g[30] =0.45*x[42]/1000.0/(0.4+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d)); //2018-04-25: 0.4 to 0.45
		
	//	g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
			
        // g[30]=__max(g[30],0.0);
        if(g[30]<0.0) g[30]=0.0;

		// root maintenance respiration  
		//exponent=(z[15]-tb)/10.0;
		
		//ratio_froot=exp(1.007)*pow(x[10],(float)-(0.841));
		//ratio_froot=__min(0.9, ratio_froot);
				
			 
	  //g[31]=0.05*x[10]*(1-ratio_froot)*b[21]*pow(b[29],exponent);		//coarse root  
      // g[31]=x[10]*(1-ratio_froot)*b[21]*pow(b[29],exponent);		//coarse root  
	
		g[31]=0.04*x[44]/1000.0 *b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));		// //2018-04-25: 0.04 to 0.045
       g[31]=g[31] +0.04*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	       //  //2018-04-25: 0.04 to 0.045
	 
	   //g[31]=g[31]+x[10]*ratio_froot*b[24]*pow(b[29],exponent);	// coarse + fine root */
       	  
	//����Ҷ������������ϸ����������b[24], �ο��������ʣ�b[29]: Q10	
	  //  g[31]=x[8]*0.8*b[24]*pow(b[29],exponent);//fine root respiration, ϸ���������� //2011��8��30�ս�����1.3 ��Ϊ1.1, ԭ����������̫��
	
		//g[31] = __max  (g[31], 0.0);  
		if(g[31]<0.0)  g[31]=0.0;
	}
	break;

	case 1:case 7: case 8: //Evergreen broadleaved forest, ������Ҷ��
	{
	//x[8]= z[10]/b[1];
		/* leaf night time maintenance respiration */
		
		
		g[25] =x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;   //2014-09-12 0.6 to 0.8
		//  g[25] = 0.5*x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 
		
		//g[25] = __max( g[25], 0.0);
           if(g[25]<0.0) g[25]=0.0;

		/* stem maintenance respiration */

    	//exponent=(z[15]-tb)/10.0;
	
		//g[30] =x[9]*b[20]*pow(b[28],exponent);//test2
		
	   	g[30] =0.4*x[42]/1000.0/(0.4+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�    //2014-09-12 0.75 to 0.9
		
      //  g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
		
        if(g[30]<0.0) g[30]=0.0;

        /* root maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
		/*g[31]=x[8]*1.0*b[24]*pow(b[29],exponent);	// fine root 
		g[31] = __max  (g[31], 0.0); 
          */
 
      g[31]= 0.1*x[44]/1000.0 *b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));		//*coarse root   //2013-08-13                                              //2014-09-12 0.05 to 0.1                              
   	  g[31]=g[31] +0.1*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));// coarse + fine root  ///2013-08-13
      if(g[31]<0.0)  g[31]=0.0;

	}
	break;

	case 5: //Deciduous needleleaved forest,   
	{
		//x[8]= z[10]/b[1];
		/* leaf night time maintenance respiration */
		
		
		g[25] =x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

  //g[25] = x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 
		//g[25] = __max( g[25], 0.0);
      if(g[25]<0.0) g[25]=0.0;

		/* stem maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
		//g[30] = x[9]*b[20]*pow(b[28],exponent); //test2
		
		g[30] =0.5*x[42]/1000.0/(0.5+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
		
		//g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
		
	//	g[30] = __max( g[30], 0.0);

if(g[30]<0.0) g[30]=0.0;

		/* root maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
	 	/*g[31]=x[8]*1.0*b[24]*pow(b[29],exponent);	// fine root 
		g[31] = __max  (g[31], 0.0); 
          */

      g[31]=  0.05*x[44]/1000.0 *b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));		//*coarse root   //2013-08-13
      g[31]=g[31] +0.05*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	    // coarse + fine root  ///2013-08-13
       if(g[31]<0.0)  g[31]=0.0;
	
	}
   	break;

	case 2: case 3://DBF����Ҷ��Ҷ��
	{
		//x[8]= z[10]/b[1];
		/* leaf night time maintenance respiration */
		
		
		g[25] =x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

  //g[25] = x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 

		//g[25] = __max( g[25], 0.0);

if(g[25]<0.0) g[25]=0.0;

	/* stem maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
		//g[30] = x[9]*b[20]*pow(b[28],exponent); //test2
		
		g[30] =0.45*x[42]/1000.0/(0.45+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
	
        //g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
		//g[30] = __max( g[30], 0.0);

if(g[30]<0.0) g[30]=0.0;

		/* root maintenance respiration */

		//exponent=(z[15]-tb)/10.0;
		/*
		g[31]=x[8]*1.0*b[24]*pow(b[29],exponent);	//1 fine root  
		g[31] = __max  (g[31], 0.0); 
         */
     
	  g[31]=  0.05*x[44]/1000.0 *b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));		//*coarse root   //2013-08-13
      g[31]=g[31] +0.05*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	    // coarse + fine root  ///2013-08-13
     if(g[31]<0.0)  g[31]=0.0;
	}
	break;

	case 6: case 9: //MF mixed forest���콻��
	{
		//x[8]= z[10]/b[1];
		/* leaf night time maintenance respiration */
		
		
		g[25] =x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

  //g[25] = x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 

		//g[25] = __max( g[25], 0.0);
if(g[25]<0.0) g[25]=0.0;
		/* stem maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
		//g[30] = x[9]*b[20]*pow(b[28],exponent);
		
		
	    g[30] =0.70*x[42]/1000.0/(0.70+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
			
		//g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
	//	g[30] = __max( g[30], 0.0);
       if(g[30]<0.0) g[30]=0.0;

		/* root maintenance respiration */

		//exponent=(z[15]-tb)/10.0;
		/*
		g[31]=x[8]*1.0*b[24]*pow(b[29],exponent);	// fine root  
		g[31] = __max  (g[31], 0.0); 
           */
      g[31]=  0.04*x[44]/1000.0 *b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));		//*coarse root   //2013-08-13
      g[31]=g[31] +0.04*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	    // coarse + fine root  ///2013-08-13
     if(g[31]<0.0)  g[31]=0.0;
	}
	break;
 
case 11:case 12:case 14: //  shrubland�� ��ľ
	{
		/* leaf carbon = LAI/SLA */ 

		//x[8]= z[10]/b[1];

		/* leaf night time maintenance respiration */
		
		g[25] =2.0*x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

  //g[25] = x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 
		//g[25] = __max( g[25], 0.0);

    if(g[25]<0.0) g[25]=0.0;

/* stem maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
	   // g[30] = x[9]*b[20]*pow(b[28],exponent);
		
		 g[30] =2.0*0.5*x[42]/1000.0/(0.5+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
		
	     //g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
		
		//g[30] = __max( g[30], 0.0);
       if(g[30]<0.0) g[30]=0.0;
		/* root maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
		/*
		g[31]=x[8]*1.0*b[24]*pow(b[29],exponent);	//fine root  
		g[31] = __max  (g[31], 0.0);  
            */

      g[31]=2.0*  0.04*x[44] /1000.0*b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	        //*coarse root   //2013-08-13
      g[31]=g[31] +2.0*0.04*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	    // coarse + fine root  ///2013-08-13
     if(g[31]<0.0)  g[31]=0.0;
    	}
	break;

case 13 ://grassland�� �ݵ�
	{
      		
		g[25] =1.6*x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

        g[30] =1.6*x[44]/1000.0*b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;



	}
	break;

case 16: case 17: case 18: //Zfm3.25 12,14://cropland
	{

		g[25] =2.0*x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;   //Leaf 

        g[30] =2.0*x[42]/1000.0*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;   //steam
	
		g[31] =2.0*x[44]/1000.0*b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;


		//g[30] =1.4*0.35*x[42]/1000.0/(0.5+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
		//if(g[30]<0.0) g[30]=0.0;


	}
    break;

	default:                              	/* other land cover types */
	{
	

         exponent=(z[15]-tb)/10.0;
		g[25] =0.45*x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

		g[30] =0.35*x[42]/1000.0/(0.5+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
		if(g[30]<0.0) g[30]=0.0;

	}

	}
	
  //a1 =g[24]*0.75-1.0*(g[25]*0.85+g[30] +g[31] );  //g[26] �� NPP ԭ����   //g[25] Ҷ������ g[30] �� ����   //g[31]������    //2018-05-17

 
   a1 =g[24]*0.75-1.0*(g[25]+g[30] +g[31] );  //g[26] �� NPP ԭ����   //g[25] Ҷ������ g[30] �� ����   //g[31]������    //201308011



  a2= a1/g[24];

  if(a2<0.35)      a2=0.35;
  else if(a2>0.65) a2=0.65;


  g[26]=g[24]*a2;
	
 
}  //==========================================================================the end of index==1===========================================

else { //----------------------------------------------------------------------------index=0;

/*
gpufarq_psn(CNl,b,z,g,co);    

g[24] = (g[28]*z[32]+g[29]*z[33])*z[18]*12.0/1000000000.0;

a2=0.5;
  g[26]=g[24]*a2;

*/



// RESPIRATION	����

	//===========================================================����һ���ƽ���������=======================================================
/*
	if(z[23]==13) slope=6; 
	else          slope=8; 


	 D0=1.35+z[45]*0.007;                                  // D0=1.25+z[45]*0.025;
	if(D0<1.35) D0=1.35; 
     D0=D0*10.0;


	slope=12.7-0.207*z[14];
	if(slope>10) slope=10;
	if(slope<6.9) slope=6.9;

slope=10;

	D0=(142.4-4.8*z[14]);
	if(D0>80) D0=80;
	if(D0<8)  D0=8;
*/

	if(z[23]==13) slope=8; 
	else          slope=10; 
	D0=0.005;



	 for(jj=1;jj<=5;jj++){

	gpufarq_psn(CNl,b,z,g,co);    

  //���׵���:mol/ m2/s
	//Gsunlit_o=slope*g[28]*g[80]/co/(1.0+z[16]/D0)+0.001; //z[16]:VPD(0.1kPa"     
   // Gshaded_o=slope*g[29]*g[80]/co/(1.0+z[16]/D0)+0.001; //z[16]:VPD(0.1kPa"



	Gsunlit_o=slope*g[28]*g[80]*z[46]/co+D0; //z[16]:VPD(0.1kPa"     
	Gshaded_o=slope*g[29]*g[80]*z[46]/co+D0; //z[16]:VPD(0.1kPa"


	//Rg=8.314 UNIT: m3 Pa/mol/k.

 //���׵���:m/s/PA
	cs1=Gsunlit_o*8.314*(z[14]+273.14)/101350;   
			
	cs2=Gshaded_o*8.314*(z[14]+273.14)/101350;
	

	cbl=0.08;
	// leaf cuticular conductance 
	cc=0.00005;

	g[20] =cbl*(cc+cs1)/(cbl+cc+cs1);   //��Ҷ                   //2013��9��11�ոģ� ԭ������ѹ�Ͳ���

	g[21] =cbl*(cc+cs2)/(cbl+cc+cs2);   //��Ҷ     //�øù�ʽ�� �� 

}

 


    //һ���GPP  //convert umol C/m2/s --> kg C/m2/period ��λת����
    //z[32]: Sunlit LAI; z[33]: Shaded LAI, z[18]: һ����ճ���
	
    g[24] = (g[28]*z[32]+g[29]*z[33])*z[18]*12.0/1000000000.0;
   //========================================================================================================================================

	//leaf day time maintenance respiration //a equation below can overwrite this one  
	x[8]= z[10]/b[1];
	
	
	exponent=(z[14]-tb)/10.0;         //z[14] �����¶�
    exponent_n=(z[15]-tb)/10.0;       //z[15] ҹ���¶�   
    ratio_d=z[18]/86400;              //�����ճ���ռһ��ı���


	//g[23] =x[8]*b[19]*pow(b[25],exponent);//*z[18]/86400;   //Ҷ����ά���Ժ���

//g[23] =x[8]/1000.0*b[19]*pow(b[25],exponent);//*z[18]/86400;   //Ҷ����ά���Ժ���

g[23]=0.0;

	if(g[23]<0.0)  g[23]=0.0;
//Rm calculation, Bonan (1995), z[10]: LAI; z[4]: Tmax; z[5]:Tmin 

	lc_p=(int short )z[23];    //�ر�������
	switch(lc_p)
	{
	case 4: case 10://Evergreen needle forest, ������Ҷ��
	{
		//x[8]= z[10]/b[1];      //����LAI��SLA�õ��ڲ�������     //2013-08-13
		
	     //leaf night time maintenance respiration, ����ҶƬ��ά���Ժ���
	 
		//b[19],�ο��������ʣ���Readb();b[25]: Q10: ��read_init�и���
		
		g[25] =0.45*x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));  //2018-04-25: 0.4 to 0.45
		
      //  g[25] = x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 

		//g[25] = __max( g[25], 0.0);
       if(g[25]<0.0) g[25]=0.0; 

		//stem maintenance respiration�����ɺ���
    
		//x[9],�߲�ľ����������model(); b[20]���ο���������,��readb()��b[28],Q10,��read_init();
		//exponent=(z[15]-tb)/10.0;
	 	//g[30] = x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���
		
		g[30] =0.45*x[42]/1000.0/(0.4+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d)); //2018-04-25: 0.4 to 0.45
		
	//	g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
			
        // g[30]=__max(g[30],0.0);
        if(g[30]<0.0) g[30]=0.0;

		// root maintenance respiration  
		//exponent=(z[15]-tb)/10.0;
		
		//ratio_froot=exp(1.007)*pow(x[10],(float)-(0.841));
		//ratio_froot=__min(0.9, ratio_froot);
				
			 
	  //g[31]=0.05*x[10]*(1-ratio_froot)*b[21]*pow(b[29],exponent);		//coarse root  
      // g[31]=x[10]*(1-ratio_froot)*b[21]*pow(b[29],exponent);		//coarse root  
	   g[31]=  0.04*x[44]/1000.0 *b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));		// //2018-04-25: 0.04 to 0.045
       g[31]=g[31] +0.04*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	       //  //2018-04-25: 0.04 to 0.045
	 
	   //g[31]=g[31]+x[10]*ratio_froot*b[24]*pow(b[29],exponent);	// coarse + fine root */
       	  
	//����Ҷ������������ϸ����������b[24], �ο��������ʣ�b[29]: Q10	
	  //  g[31]=x[8]*0.8*b[24]*pow(b[29],exponent);//fine root respiration, ϸ���������� //2011��8��30�ս�����1.3 ��Ϊ1.1, ԭ����������̫��
	
		//g[31] = __max  (g[31], 0.0);  
		if(g[31]<0.0)  g[31]=0.0;
	}
	break;

	case 1:case 7: case 8: //Evergreen broadleaved forest, ������Ҷ��
	{
	//x[8]= z[10]/b[1];
		/* leaf night time maintenance respiration */
		
		exponent=(z[15]-tb)/10.0;
		g[25] =x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;   //2014-09-12 0.6 to 0.8
		//  g[25] = 0.5*x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 
		
		//g[25] = __max( g[25], 0.0);
           if(g[25]<0.0) g[25]=0.0;

		/* stem maintenance respiration */

    	//exponent=(z[15]-tb)/10.0;
	
		//g[30] =x[9]*b[20]*pow(b[28],exponent);//test2
		
	   	g[30] =0.4*x[42]/1000.0/(0.4+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�    //2014-09-12 0.75 to 0.9
		
      //  g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
		
        if(g[30]<0.0) g[30]=0.0;

        /* root maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
		/*g[31]=x[8]*1.0*b[24]*pow(b[29],exponent);	// fine root 
		g[31] = __max  (g[31], 0.0); 
          */
 
      g[31]= 0.1*x[44]/1000.0 *b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));		//*coarse root   //2013-08-13                                              //2014-09-12 0.05 to 0.1                              
   	  g[31]=g[31] +0.1*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	       // coarse + fine root  ///2013-08-13
      if(g[31]<0.0)  g[31]=0.0;

	}
	break;

	case 5: //Deciduous needleleaved forest,   
	{
		//x[8]= z[10]/b[1];
		/* leaf night time maintenance respiration */
		
		exponent=(z[15]-tb)/10.0;
		g[25] =x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

  //g[25] = x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 
		//g[25] = __max( g[25], 0.0);
      if(g[25]<0.0) g[25]=0.0;

		/* stem maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
		//g[30] = x[9]*b[20]*pow(b[28],exponent); //test2
		
		g[30] =0.75*x[42]/1000.0/(0.75+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
		
		//g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
		
	//	g[30] = __max( g[30], 0.0);

if(g[30]<0.0) g[30]=0.0;

		/* root maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
	 	/*g[31]=x[8]*1.0*b[24]*pow(b[29],exponent);	// fine root 
		g[31] = __max  (g[31], 0.0); 
          */

      g[31]=  0.05*x[44]/1000.0 *b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));		//*coarse root   //2013-08-13
      g[31]=g[31] +0.05*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	    // coarse + fine root  ///2013-08-13
       if(g[31]<0.0)  g[31]=0.0;
	
	}
   	break;

	case 2: case 3://DBF����Ҷ��Ҷ��
	{
		//x[8]= z[10]/b[1];
		/* leaf night time maintenance respiration */
		
		exponent=(z[15]-tb)/10.0;
		g[25] =x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

  //g[25] = x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 

		//g[25] = __max( g[25], 0.0);

if(g[25]<0.0) g[25]=0.0;

	/* stem maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
		//g[30] = x[9]*b[20]*pow(b[28],exponent); //test2
		
		g[30] =0.75*x[42]/1000.0/(0.75+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
	
        //g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
		//g[30] = __max( g[30], 0.0);

if(g[30]<0.0) g[30]=0.0;

		/* root maintenance respiration */

		//exponent=(z[15]-tb)/10.0;
		/*
		g[31]=x[8]*1.0*b[24]*pow(b[29],exponent);	//1 fine root  
		g[31] = __max  (g[31], 0.0); 
         */
     
	  g[31]=  0.05*x[44]/1000.0 *b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));		//*coarse root   //2013-08-13
      g[31]=g[31] +0.05*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	    // coarse + fine root  ///2013-08-13
     if(g[31]<0.0)  g[31]=0.0;
	}
	break;

	case 6: case 9: //MF mixed forest���콻��
	{
		//x[8]= z[10]/b[1];
		/* leaf night time maintenance respiration */
		
		exponent=(z[15]-tb)/10.0;
		g[25] =x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

  //g[25] = x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 

		//g[25] = __max( g[25], 0.0);
if(g[25]<0.0) g[25]=0.0;
		/* stem maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
		//g[30] = x[9]*b[20]*pow(b[28],exponent);
		
		
	    g[30] =0.70*x[42]/1000.0/(0.70+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
			
		//g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
	//	g[30] = __max( g[30], 0.0);
       if(g[30]<0.0) g[30]=0.0;

		/* root maintenance respiration */

		//exponent=(z[15]-tb)/10.0;
		/*
		g[31]=x[8]*1.0*b[24]*pow(b[29],exponent);	// fine root  
		g[31] = __max  (g[31], 0.0); 
           */
      g[31]=  0.04*x[44]/1000.0 *b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));		//*coarse root   //2013-08-13
      g[31]=g[31] +0.04*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	    // coarse + fine root  ///2013-08-13
     if(g[31]<0.0)  g[31]=0.0;
	}
	break;
 
case 11:case 12:case 14: //  shrubland�� ��ľ
	{
		/* leaf carbon = LAI/SLA */ 

		//x[8]= z[10]/b[1];

		/* leaf night time maintenance respiration */
		
		exponent=(z[15]-tb)/10.0;
		g[25] =x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

  //g[25] = x[8]*b[19]*pow(b[25],exponent)*(86400.0-z[18])/86400; 
		//g[25] = __max( g[25], 0.0);

    if(g[25]<0.0) g[25]=0.0;

/* stem maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
	   // g[30] = x[9]*b[20]*pow(b[28],exponent);
		
		 g[30] =0.5*x[42]/1000.0/(0.5+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
		
	     //g[30] =x[9]*b[20]*pow(b[28],exponent);//test2    //��ά���Ժ���  //2011��7��25�ո�     //20130623��
		
		//g[30] = __max( g[30], 0.0);
       if(g[30]<0.0) g[30]=0.0;
		/* root maintenance respiration */
		//exponent=(z[15]-tb)/10.0;
		/*
		g[31]=x[8]*1.0*b[24]*pow(b[29],exponent);	//fine root  
		g[31] = __max  (g[31], 0.0);  
            */

      g[31]=  0.04*x[44] /1000.0*b[21]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	        //*coarse root   //2013-08-13
      g[31]=g[31] +0.04*x[43]/1000.0 *b[24]*(pow(b[29],exponent)*ratio_d+pow(b[29],exponent_n)*(1-ratio_d));	    // coarse + fine root  ///2013-08-13
     if(g[31]<0.0)  g[31]=0.0;
    	}
	break;

case 13 ://grassland�� �ݵ�
	{
    	
		/*
		rf25=0.4;      //0.5, 0.35 //5.18
		pr=0;      //  0.0   zfm5.14

		/// g[25] is total rm in kg C/m2/day
		//g[25] = 1.0368*(2*z[10])*rf25*(1+pr)*pow(2, 0.1*(0.5*(z[4]+z[5])-25.0))/1000;
	    // g[25] =0.30* 1.0368*(4*z[10])*rf25*(1+pr)*pow(b[28],exponent)/1000;  //2012-10-14��,����0.25    //20130803,ϵ����0.2��Ϊ0.3
		 g[25] =0.25* 1.0368*(4*z[10])*rf25*(1+pr)*pow(b[28],exponent)/1000;  
		g[30]=0.0;
		g[31]=0.0;
    */


		exponent=(z[15]-tb)/10.0;
		g[25] =0.45*x[41]/1000.0*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

		g[30] =0.35*x[42]/1000.0/(0.5+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
		if(g[30]<0.0) g[30]=0.0;



	}
	break;

case 16: case 17: case 18: //Zfm3.25 12,14://cropland
	{
       exponent=(z[15]-tb)/10.0;
		g[25] =0.5*x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

		g[30] =0.4*x[42]/1000.0/(0.5+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
		if(g[30]<0.0) g[30]=0.0;


	}
    break;

	default:                              	/* other land cover types */
	{
	

         exponent=(z[15]-tb)/10.0;
		g[25] =0.45*x[41]/1000.0*b[19]*(pow(b[25],exponent)*ratio_d+pow(b[25],exponent_n)*(1-ratio_d));//*(86400.0-z[18])/86400;

		g[30] =0.35*x[42]/1000.0/(0.5+x[42]/1000.0)*b[20]*(pow(b[28],exponent)*ratio_d+pow(b[28],exponent_n)*(1-ratio_d));//test2    //��ά���Ժ���  //2011��7��25�ո�
		if(g[30]<0.0) g[30]=0.0;

	}

	}
	
  a1 =g[24]*0.75-1.0*(g[25]*0.75+g[30] +g[31] );  //g[26] �� NPP ԭ����   //g[25] Ҷ������ g[30] �� ����   //g[31]������    //201308011

 
  a2= 0.5;

  g[26]=g[24]*a2;


}
	
/* 24HR. NET C/m2  */
   //	g[26] = 0.75*g[24]-g[25]-g[30]-g[31];  //g[26] �� NPP          //2012-10-14�Ļ�
	
	// g[26] = 0.7*g[24]-(g[25]*1.5+g[30]*4.0+g[31]*1.0)*1.0;  //g[26] �� NPP ԭ����   //g[25] Ҷ������ g[30] �� ����   //g[31]������ 

  //g[26] = 0.7*(g[24]-(g[25]+g[30]+g[31])*0.8);  //g[26] �� NPP ԭ����

    //g[26] = 0.46*g[24];
	
	
	//g[26] = 0.7*g[24]-(g[25]*1.5+g[30]*4.0+g[31]*1.0)*1.0;  //g[26] �� NPP ԭ����   //g[25] Ҷ������ g[30] �� ����   //g[31]������ 
	
	//g[26] = 0.75*g[24]-(g[25]*1.8+g[30]*15.0+g[31]*1.0);  //g[26] �� NPP ԭ����   //g[25] Ҷ������ g[30] �� ����   //g[31]������    //20130803
	
	
    return;

   }

__device__ int gpufarq_psn(float CNl, float b[],float z[],float g[],float co)
{
	float pa; 		/* (Pa) atmospheric pressure */
//	float co2;		/* (ppm) atmospheric [CO2] */
	float t,tt;		/* (deg C) air temperature */
	float irad;	/* (umol/m2/s) PAR photon flux density */
	float gg;      /* (m/s) conductance to CO2 */
	float Rd;		/* (umol/m2/s) dark respiration rate  */
//	float lnc;		/* (kg Nleaf/m2) leaf N concentration, area units */
//	float flnr;	/* (kg NRub/kg Nleaf) fraction of leaf N in Rubisco */
	
	float tk;     /* (K) absolute temperature */
	float O2;     /* (Pa) atmospheric partial pressure O2 */ 
	float Ca;     /* (Pa) atmospheric partial pressure CO2 */
	float gamma;  /* (Pa) co2 compensation point, no dark respiration */
	float Kc;     /* (Pa) MM constant for carboxylase reaction */
	float Ko;     /* (Pa) MM constant for oxygenase reaction */
	float act;    /* (umol/kgRubisco/s) Rubisco activity */
	float Vmax;   /* (umol/m2/s) maximum carboxylation velocity */
	float Jmax;   /* (umol/m2/s) maximum rate of electron transport */
	float J;      /* (umol/m2/s) maximum rate of Rubisco regeneration */
	float Av;     /* (umol/m2/s) Rubisco limited assimilation rate */
	float Aj;     /* (umol/m2/s) RuBP regeneration limited assim rate */
	float A;      /* (umol/m2/s) net assimilation rate */
//    float Vmax25;  /* block it, just because the code is change. X.F. July */
    float Nratio;  /* block it, just because the code is change. X.F. July */
     int  short lc_p; /* block it, just because the code is change. X.F. July */
	float kk,aa,bb,cc,dd,ee, term1,term2,term3;  
//lixuansong:ȥ����static
	 float fnr = 7.16;   	/* kg Rub/kg NRub */
	 float Kc25 = 30.0; 		/* (Pa) MM const carboxylase, 25 deg C */ 
	 float q10Kc = 2.1;    	/* (DIM) Q_10 for kc */
	 float Ko25 = 30000.0;   /* (Pa) MM const oxygenase, 25 deg C */
	 float q10Ko = 1.2;   	/* (DIM) Q_10 for ko */
	 float act25 = 3.6;    	/* (umol/mgRubisco/min) Rubisco activity */
	 float q10act = 2.4;  	/* (DIM) Q_10 for Rubisco activity */			
      float ft;
	

	int i;	                    	/* index for sunlit/shaded leaves, 0: sun, 1 shade */			
  	for (i=0;i<=1;i++)
        {

	/* convert w/m2 to umol/m2/s */
	
/*	if (i==0) irad=2.04*z[34]*;
	else 
          irad=2.04*z[35]; 
*/

	if (i==0) irad=4.55*z[34]*0.5;
	else 
          irad=4.55*z[35]*0.5; 


	t = z[14];                                 //Average temperature during day time
	tk = t + 273.15;
	
	if (i==0) gg=g[20];
  	else
	  gg=g[21];

	/* convert conductance from m/s --> umol/m2/s/Pa ����Ҫ��*/ 
	
	gg =gg/1.6 * 1000000 / (8.3143 * tk); 
	
	/* calculate the atomsheric pressure */
	
	pa=100000;

	/* convert atmospheric CO2 from ppm --> Pa */
	
	Ca =co * pa / 1e6;  // zfm4.22  ԭco=360
  	
	/* calculate atmospheric O2 in Pa, assumes 21% O2 by volume */
	
	O2 = 0.21 * pa;

	/* correct kinetic constants for temperature, and do unit conversions */

	Ko = Ko25 * pow(q10Ko, float((t-25.0)/10.0));
	Kc = Kc25 * pow(q10Kc, (float)((t-25.0)/10.0));
	act = pow(q10act, (float)((t-25.0)/10.0));


ft=1.0/(1+exp((-220000+712.0*(t+273.15))/(8.32*(t+273.15)))); 

//2018-06-11: 710 Change to 712   Decrease earlier at high temperatures

//ft=1.0/(1+exp((-215600+700*(t+273.15))/(8.32*(t+273.15)))); 

	/* calculate gamma (Pa), assumes Vomax/Vcmax = 0.21 */

	gamma = 4.02*pow((float)1.75, (float)(t-25)/10);    //compensation point
	 

/**************** Modification according to Bonan ************* */
/********************Modified by X.F. Aug************************/
	lc_p=( int short)z[23];
   
	//Vmax25=b[49];
	//Nratio=b[50];
    
	 Nratio= b[50]/CNl;

     if(Nratio>1.0) Nratio=1.0;
     if(Nratio<0.45) Nratio=0.45;
  


	 Vmax=b[49]*Nratio*act*ft;                                                           //juw

	/**************** End of the modification ********************/

	/* calculate Jmax = f(Vmax) */

	Jmax = 29.1 + 1.64*Vmax;
	
	/* calculate J = f(Jmax, I) */

 //  J = Jmax * irad / (irad + 2.1*Jmax);
	J=0.24*irad/sqrt(1+0.24*0.24*irad*irad/Jmax/Jmax);   //F.Ian Woodward   Global Biogeochemical Cycles, 1995,9(4):471-490,����11
	/* Rd */

  	Rd=0.015*Vmax;
	g[23]=Rd*z[18]*12.01;

/***************** With daily integration *****************************/ 
	kk=Kc*(1+O2/Ko);
  
/* Av */   
	Av=0;

    	aa = (kk+Ca)*(kk+Ca);
    	bb = 2*(2*gamma+kk-Ca)*Vmax+2*(Ca+kk)*Rd;
    	cc = (Vmax-Rd)*(Vmax-Rd);

     	if (aa>0 && cc>0)
        {
	dd = sqrt(aa*gg*gg+bb*gg+cc);
	ee = (bb*bb-4*aa*cc)/(8*aa*sqrt(aa));

	term1=sqrt(aa)*gg*gg/2+sqrt(cc)*gg;
	term2=((2*aa*gg+bb)*dd-bb*sqrt(cc))/(4*aa);
	term3=ee*(log(2*aa*gg+bb+2*sqrt(aa)*dd)-log(bb+2*sqrt(aa)*sqrt(cc)));

	if (gg!=0) Av=0.5*1.27*(term1-term2+term3)/gg;

	}

/* Aj */
 	Aj=0;
    
    	aa = (2.33*gamma+Ca)*(2.33*gamma+Ca);
    	bb = 0.4*(4.3*gamma-Ca)*J+2*(Ca+2.33*gamma)*Rd;
    	cc = (0.22*J-Rd)*(0.22*J-Rd);

      	if (aa>0 && cc>0)
        {
	dd = sqrt(aa*gg*gg+bb*gg+cc);
	ee = (bb*bb-4*aa*cc)/(8*aa*sqrt(aa));

	term1=sqrt(aa)*gg*gg/2+sqrt(cc)*gg;
	term2=((2*aa*gg+bb)*dd-bb*sqrt(cc))/(4*aa);
	term3=ee*(log(2*aa*gg+bb+2*sqrt(aa)*dd)-log(bb+2*sqrt(aa)*sqrt(cc)));

	if (gg!=0) Aj=0.5*1.27*(term1-term2+term3)/gg;

	}

	/* calculate A as the minimum of (Av,Aj) */
	if (Av < Aj) A = Av; 
	else         A = Aj;

	/* primary output */
	 
if(A<0) A=0;                            //JUW-2013-10-31 Very important

	if (i==0) g[28]=A;
	else g[29]=A;
   
		
   }
   return 1;
 }	

