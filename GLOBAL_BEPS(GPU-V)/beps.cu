#include "hip/hip_runtime.h"

//===========================================2018-04-26-22:17


/*************************************************************************
Prograh: beps.c (Boreal Ecosystem Productivity Simulator)

===============================================================================================

BEPS model was originally developed by Chen and Liu in Canada. It is designed to calculate key
components in carbon and water cycles at regional and global scales using remotely sensed leaf
area index (LAI).This is the updated daily version of BEPS coded using GPU technique. 

The improvement of this daily version of BEPS over the earlier ones include:
(1) The carbon and nitrogen cycles are coupled. 
(2) Soil moisture dynamics is simulated using an implicit algorithm (Ju et al., 2010);

b[]  �������;  g[]  ����������ɢ;  x[]  ����ɢ;  z[]  ����Ҫ��
=================================================================================================

***********************************************************************************************/

#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 

#include "gpucoef.cu"
#include "gpudoxx.cu"
#include "gpumodel.cu"
#include "gpureadb.cu"
#include "gpupreInitial.cu"
#include "gpucarbon.cu"
#include "gpudoflux.cu"
#include "gpulw.cu"


#include "gpuinter.cu"
#include "gpumelt.cu"
#include "gpurs.cu"
#include "gpusolution.cu"
#include "gpuzcomp.cu"
#include "gpuv_moisture.cu"
#include "gpupenmon.cu"
#include "gpusoilresp.cu"
#include "gpureadb_init.cu"
#include "ncar.h"


float **dmatrix();		              // Function which allocates mem for matrix 

#define THREAD_NUM  512 //512 //512    // 512     // 512    //2017-10-02ǰΪ 512
#define BLOCK_NUM   512   // 256 //16//4         //2017-10-02ǰΪ   16        //2018-02-27: 256

#define cutilSafeCall(err) __cudaSafeCall (err, __FILE__, __LINE__)

bool InitCUDA()
{
	int count;

	cutilSafeCall(hipGetDeviceCount(&count));
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		cutilSafeCall(hipGetDeviceProperties(&prop, i));
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}

	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	cutilSafeCall(hipSetDevice(i));
	printf("%d %d\n",count,i);
	
	return true;
}

//===========================================   LAI����ת���ӳ���:����ЧLAIת��Ϊ��ʵLAI ==========================

__global__ static void gpuLAI(int short*lc,float*laie,float*lai)  
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
 	float omega;
	int long pix;
	
	for(pix=bid * THREAD_NUM + tid;pix<npixels;pix+=BLOCK_NUM * THREAD_NUM)
	{ 
		/*
		switch(lc[pix])
		{
		case 6 : case 9://mixed =0.6, for MODIS 
		omega=0.65;           //0.6   //2013-0828��0.7��Ϊ0.65 
		break;

	    case 1: case 2: case 3:case 7: case 8:    //broadleaf forest =0.7, for MODIS 
		omega=0.75;      //0.7       //2013-0828��0.8��Ϊ0.75    
		break;

	    case 4: case 5: case 10: //coni=0.5, for MODIS 
		omega=0.55;   //0.55  7-21         //2013-0828��0.6��Ϊ0.55  
		break;

		case 11:  case 12: case 14:     //shrub=0.5, for MODIS, shrub same as conif, closed shrub and woody savanna
		omega=0.5;
		break;

		case 13: case 16: case 17: case 18: 	// grass=0.9, for MODIS, opened shrub & savanna & grassland
		omega=0.9;
		break;

		default:
		omega=0.8;
		}  
*/
		//==========��LAIe ת��ΪLAI==================================
lai[pix]=laie[pix];/// omega; //ȫ��LAI������������ʵLAI,����Ҫ��ת��
	
if(lai[pix]>10.0)  lai[pix]=10.0;
if(lai[pix]<0.01)  lai[pix]=0.01;


	}          //Pixelѭ������

}

//==================����̼���ʼ����Ҫ��NPP================================================
	__global__ static void gpuModel1(long* jday, long* pix_offset,  int short*lc, float*CI,float *TI,short *Climatedata,float*lai,float*npp, xvalue*xx,						
float* lat,float*soilw1_old,float*soilw2_old,float*soilw3_old,hy_c1*HY1,hy_c2*HY2,hy_c3*HY3,float*co,float*lambd, float *lambd2)
	
	{
	float b[SIZEB],tmean,lambdt,lambdw1,lambdw2,lambdw3,x[SIZEX];
	float t_d,t_n,d_l, lambdt_d,lambdt_n;

	float soilw1_new, soilw2_new, soilw3_new;
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int long pix,index;
 	float CNl,coef[60];
    float a1,a2,a3;
	float w1,w2,w3;
	float lambd1_min;

   float  aa1,aa2,aa3,FC1,FC2,FC3;

	gpureadb_init(b);
		for(pix=pixoffset+bid * THREAD_NUM + tid;pix<npixels;pix+=BLOCK_NUM * THREAD_NUM)
	{ //================��ʼ��Ԫѭ��===========================

			if(lc[pix]<=0|| lc[pix]==21  || lc[pix]==19 || lai[pix]<0 ) { //���ڷ�½����Ԫ��
			
			npp[pix]=0.00;  xx[pix].x11=0;  xx[pix].x21=0; xx[pix].x4=0; xx[pix].x5=0;
		}		
		else {//����½����Ԫ����ʼ����
			
		
			gpureadb(b,lc[pix]);	    	//Read parameters according to cover type          // read ��Ӧ���������	   	
							
			index=0;
		
            gpucoef_c(pix,lc[pix],coef,HY2);
	       
		  CNl=coef[46];


			gpumodel(CNl,*jday,pix,index,lat[pix],lai[pix],lc[pix],CI[pix],TI[pix],soilw1_old[pix],soilw2_old[pix],soilw3_old[pix],
				HY1,HY2,HY3,x,b,Climatedata,xx,*co,&soilw1_new, &soilw2_new, &soilw3_new,&tmean, &t_d,&t_n,&d_l);

			npp[pix]=xx[pix].x6*1000.0;  //ÿ���NPP  //juw
		
			//����3������ʪ��=================================
			soilw1_old[pix]=soilw1_new; soilw2_old[pix]=soilw2_new; soilw3_old[pix]=soilw3_new;


			//=======�����¶ȶԷֽ�ĵ���ϵ���� ���������¶���������ȣ����Ƿǳ����Ƶļ���
			
	

			if(t_d>35.0)           lambdt_d=1.0;
			else if( t_d<-4.0  )   lambdt_d=0.03;
			else  {
				a1=1.0/(35.0+46.32)-1/(t_d+46.32);
				a2=a1*308.56;
				lambdt_d=exp(a2);
			}

			if(t_n>35.0)        lambdt_n=1.0;
			else if( t_n<-4.0  )   lambdt_n=0.03;
			else  {
				a1=1.0/(35.0+46.32)-1/(t_n+46.32);
				a2=a1*308.56;
				lambdt_n=exp(a2);
			}


aa1=soilw1_old[pix]/HY1[pix].PR*100;   //======================================================��һ������ʵ�ʺ�ˮ��ռ���ͺ�ˮ���ı���
FC1=HY1[pix].FC/HY1[pix].PR*100; 


aa2=soilw2_old[pix]/HY2[pix].PR*100;   //======================================================��2������ʵ�ʺ�ˮ��ռ���ͺ�ˮ���ı���
FC2=HY2[pix].FC/HY2[pix].PR*100; 


aa3=soilw3_old[pix]/HY3[pix].PR*100;   //======================================================��3������ʵ�ʺ�ˮ��ռ���ͺ�ˮ���ı���
FC3=HY3[pix].FC/HY3[pix].PR*100; 


if(FC1<10) FC1=10;    if(FC1>90) FC1=90;
if(FC2<10) FC2=10;    if(FC2>90) FC2=90;
if(FC3<10) FC3=10;    if(FC3>90) FC3=90;

gpulw1(FC1,FC2,FC3, aa1,aa2,aa3,&lambdw1,&lambdw2,&lambdw3);


/*
if(lc[pix]==16 ||lc[pix]==17 || lc[pix]==18){
w1=60;w2=40;w3=0;
}
else if(lc[pix]==1 ||lc[pix]==7 || lc[pix]==8){
w1=70;w2=30;w3=0;	
}

else if(lc[pix]==5 ||lc[pix]==4){
w1=50;w2=50;w3=0;	
}

else {
w1=60;w2=40;w3=0;	
}
*/

/*
if(lc[pix]==16 ||lc[pix]==17 || lc[pix]==18){
	w1=70;w2=30;w3=0;
}
else if(lc[pix]==1 ||lc[pix]==7 || lc[pix]==8){
	w1=70;w2=30;w3=0;	
}

else if(lc[pix]==5 ||lc[pix]==4){
	w1=50;w2=50;w3=0;	
}

else {
	w1=60;w2=40;w3=0;	
}
*/

if(lc[pix]==16 ||lc[pix]==17 || lc[pix]==18){
	w1=50;w2=40;w3=10;
}

else if(lc[pix]==1 ||lc[pix]==7 || lc[pix]==8){
	w1=45;w2=35;w3=20;	
}

else if(lc[pix]==5 ||lc[pix]==4){
	w1=30;w2=40;w3=30;	
}

else {
	w1=35;w2=40;w3=25;	
}

a2=lambdt_d*(w1*lambdw1+w2*lambdw2+w3*lambdw3)/(w1+w2+w3);
if(a2<0.0075) a2=0.0075;

a3=lambdt_n*(w1*lambdw1+w2*lambdw2+w3*lambdw3)/(w1+w2+w3);
if(a3<0.0075) a3=0.0075;

lambd[pix]=lambd[pix]+a2*d_l/86400+a3*(1.0-d_l/86400.0);

//=========================================================================================================================================================

/*
if(lc[pix]==16 ||lc[pix]==17 || lc[pix]==18){
	w1=30;w2=50;w3=20;
}

else if(lc[pix]==1 ||lc[pix]==7 || lc[pix]==8){
	w1=30;w2=50;w3=20;	


///w1=40;w2=40;w3=20;	


}

else if(lc[pix]==5 ||lc[pix]==4){
	w1=20;w2=50;w3=30;	
}

else {
	w1=40;w2=40;w3=20;	

//	w1=30;w2=50;w3=20;	

}


if(lc[pix]==16 ||lc[pix]==17 || lc[pix]==18){
	w1=50;w2=40;w3=10;
}

else if(lc[pix]==1 ||lc[pix]==7 || lc[pix]==8){
	w1=55;w2=30;w3=15;	
}

else if(lc[pix]==5 ||lc[pix]==4){
	w1=30;w2=40;w3=30;	
}

else {
	w1=40;w2=40;w3=20;	
}
*/



a2=lambdt_d*(w1*lambdw1+w2*lambdw2+w3*lambdw3)/(w1+w2+w3);
if(a2<0.0075) a2=0.0075;

a3=lambdt_n*(w1*lambdw1+w2*lambdw2+w3*lambdw3)/(w1+w2+w3);
if(a3<0.0075) a3=0.0075;

lambd2[pix]=lambd2[pix]+a2*d_l/86400+a3*(1.0-d_l/86400.0);


}			   			   
	} //=========================End of PIXEL cycle PPPPPPPPPP     ��Ԫѭ������
}

//=============================================����ÿ���̼ͨ����ˮͨ��==================================================
					
__global__ static void gpuModel2(long* jday,long* pix_offset, int short*lc,float*CI,float *TI,short * climatedata ,float *Ndep,float *Ndep0,float*lai,float*npp, xvalue*xx,float* lat,
								 float*soilw1_old,float*soilw2_old,float*soilw3_old,hy_c1*HY1,hy_c2*HY2,hy_c3*HY3,float*co,float*lambd, float *lambd2,
								 float*soilw1_n,float*soilw2_n,float*soilw3_n, carbonpool *Cpoolold,CNratio *CNRold,carbonpool *Cpoolnew,CNratio *CNRnew,
 							     float*hr,float*nep, int short *outputnep)
{
	float b[SIZEB],tmean,lambdt,lambdw1,lambdw2,lambdw3,x[SIZEX];
     float t_d,t_n,d_l,lambdt_d,lambdt_n;


	float soilw1_new, soilw2_new, soilw3_new;
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int long pix,index;
 	float coef[40],initialValues[40];
    int count;
    int  long  pix1; 
   float a1,a2,a3;
   float w1,w2,w3;
   float lam1,lam2, Net_dep;;

float  aa1,aa2,aa3,FC1,FC2,FC3;

	gpureadb_init(b);


	for(pix=pixoffset+bid * THREAD_NUM + tid;pix<npixels;pix+=BLOCK_NUM * THREAD_NUM)
	{ //================��ʼ��Ԫѭ��===========================

     count=pix/4950;      //4950��ʵ�����ݵ�����
     pix1=   4950*count*2;

   if(lc[pix]<=0|| lc[pix]==21 || lc[pix]==19  || lai[pix]<0 ) { //���ڷ�½����Ԫ��
			 
			npp[pix]=0.00; 	  xx[pix].x11=0; 	 xx[pix].x21=0; 	 xx[pix].x4=0;       xx[pix].x5=0;
		
			Cpoolnew[pix].Cstem =0.1;     Cpoolnew[pix].Cleaf =0.1;        Cpoolnew[pix].Ccroot=0.1;       Cpoolnew[pix].Cfroot=0.1; 

			Cpoolnew[pix].Ccd   = 0.1;    Cpoolnew[pix].Csmd= 0.1;         Cpoolnew[pix].Cssd= 0.1;        Cpoolnew[pix].Cfmd = 0.1;  
			
			Cpoolnew[pix].Cfsd= 0.1;      Cpoolnew[pix].Csm  = 0.1;
			Cpoolnew[pix].Cm  = 0.1;      Cpoolnew[pix].Cs   = 0.1;  Cpoolnew[pix].Cp=0.1;   
	
			
			hr[pix]    =0.0001;	nep[pix]   =0.0001;
		
            soilw1_n[pix]=0;  soilw2_n[pix]=0;soilw3_n[pix]=0;

			CNRnew[pix].CNcd   =0.1;      	CNRnew[pix].CNssd=0.1;      	CNRnew[pix].CNsmd  =0.1;

			CNRnew[pix].CNfsd  =0.1;     	CNRnew[pix].CNfmd  =0.1;     	CNRnew[pix].CNsm   =0.1;

			CNRnew[pix].CNm    =0.1;       CNRnew[pix]. CNs   =0.1;          CNRnew[pix]. CNp    =0.1;

			CNRnew[pix].CNstem =0.1;     	CNRnew[pix].CNcroot=0.1;     	CNRnew[pix].CNleaf=0.1 ;

			CNRnew[pix].CNfroot=0.1;     	CNRnew[pix].Nav     =0.1;      CNRnew[pix].  Nt     =0.1 ;


            outputnep[pix+pix1]        =-30000;      //signed short ǿ�ư�Npp��Ϊ����������					
			outputnep[4950+pix+pix1]   =-30000;      //signed short ǿ�ư�Npp��Ϊ����������					
			outputnep[4950*2+pix+pix1] =-30000; 
			//outputnep[4950*3+pix+pix1] =0; 
			//outputnep[4950*4+pix+pix1] =0; 
 

		}		
		else {                                            //����½����Ԫ����ʼ����
        
			index=1;
       	
			gpureadb(b,lc[pix]);	    	//Read parameters according to cover type          // read ��Ӧ���������	   	
			
			xx[pix].x41 =Cpoolold[pix].Cleaf;   	xx[pix].x42 =Cpoolold[pix].Cstem; 	xx[pix].x43=Cpoolold[pix].Cfroot;   	xx[pix].x44=Cpoolold[pix].Ccroot;	
			

			gpumodel(CNRold[pix].CNleaf,*jday,pix,index,lat[pix],lai[pix],lc[pix],CI[pix],TI[pix],soilw1_old[pix],soilw2_old[pix],soilw3_old[pix],
			HY1,HY2,HY3,x,b, climatedata,xx,*co,&soilw1_new, &soilw2_new, &soilw3_new,&tmean,&t_d,&t_n,&d_l);

			soilw1_n[pix]=soilw1_new; soilw2_n[pix]=soilw2_new; soilw3_n[pix]=soilw3_new;

			if(lc[pix]==16 || lc[pix]==17 || lc[pix]==18){

if (soilw1_n[pix]<(HY1[pix].WP+(HY1[pix].FC-HY1[pix].WP)*0.1) )  soilw1_n[pix]=(HY1[pix].WP+(HY1[pix].FC-HY1[pix].WP)*0.1);
if (soilw2_n[pix]<(HY2[pix].WP+(HY2[pix].FC-HY2[pix].WP)*0.1) )  soilw2_n[pix]=(HY2[pix].WP+(HY2[pix].FC-HY2[pix].WP)*0.1);
if (soilw3_n[pix]<(HY3[pix].WP+(HY3[pix].FC-HY3[pix].WP)*0.1) )  soilw3_n[pix]=(HY3[pix].WP+(HY3[pix].FC-HY3[pix].WP)*0.1);

			}

			npp[pix]=xx[pix].x6*1000.0;  //ÿ���NPP 
			if(t_d>35.0)        lambdt_d=1.0;
			else if( t_d<-4.0  )   lambdt_d=0.03;
			else  {
				a1=1.0/(35.0+46.32)-1/(t_d+46.32);
				a2=a1*308.56;
				lambdt_d=exp(a2);
			}

			if(t_n>35.0)        lambdt_n=1.0;
			else if( t_n<-4.0  )   lambdt_n=0.03;
			else  {
				a1=1.0/(35.0+46.32)-1/(t_n+46.32);
				a2=a1*308.56;
				lambdt_n=exp(a2);
			}


			aa1=soilw1_n[pix]/HY1[pix].PR*100;   //======================================================��һ������ʵ�ʺ�ˮ��ռ���ͺ�ˮ���ı���
			FC1=HY1[pix].FC/HY1[pix].PR*100; 


			aa2=soilw2_n[pix]/HY2[pix].PR*100;   //======================================================��2������ʵ�ʺ�ˮ��ռ���ͺ�ˮ���ı���
			FC2=HY2[pix].FC/HY2[pix].PR*100; 


			aa3=soilw3_n[pix]/HY3[pix].PR*100;   //======================================================��3������ʵ�ʺ�ˮ��ռ���ͺ�ˮ���ı���
			FC3=HY3[pix].FC/HY3[pix].PR*100; 


			if(FC1<10) FC1=10;    if(FC1>90) FC1=90;
			if(FC2<10) FC2=10;    if(FC2>90) FC2=90;
			if(FC3<10) FC3=10;    if(FC3>90) FC3=90;

			
			gpulw1(FC1,FC2,FC3, aa1,aa2,aa3,&lambdw1,&lambdw2,&lambdw3);
/*
			if(lc[pix]==16 ||lc[pix]==17 || lc[pix]==18){
				w1=70;w2=30;w3=0;
			}
			else if(lc[pix]==1 ||lc[pix]==7 || lc[pix]==8){
				w1=70;w2=30;w3=0;	
			}

			else if(lc[pix]==5 ||lc[pix]==4){
				w1=50;w2=50;w3=0;	
			}

			else {
				w1=60;w2=40;w3=0;	
			}
*/


			if(lc[pix]==16 ||lc[pix]==17 || lc[pix]==18){
				w1=50;w2=40;w3=10;
			}

			else if(lc[pix]==1 ||lc[pix]==7 || lc[pix]==8){
				w1=45;w2=35;w3=20;	
			}

			else if(lc[pix]==5 ||lc[pix]==4){
				w1=30;w2=40;w3=30;	
			}

			else {
				w1=35;w2=40;w3=25;	
			}

			a2=lambdt_d*(w1*lambdw1+w2*lambdw2+w3*lambdw3)/(w1+w2+w3);
			if(a2<0.0075) a2=0.0075;

			a3=lambdt_n*(w1*lambdw1+w2*lambdw2+w3*lambdw3)/(w1+w2+w3);
			if(a3<0.0075) a3=0.0075;

			lambd[pix]=a2*d_l/86400+a3*(1.0-d_l/86400.0);

			//=========================================================================================================================================================
/*
			if(lc[pix]==16 ||lc[pix]==17 || lc[pix]==18){
				w1=30;w2=50;w3=20;
			}

			else if(lc[pix]==1 ||lc[pix]==7 || lc[pix]==8){
				w1=30;w2=50;w3=20;	

//w1=40;w2=40;w3=20;	


			}

			else if(lc[pix]==5 ||lc[pix]==4){
				w1=20;w2=50;w3=30;	
			}

			else {


			w1=40;w2=40;w3=20;	
			
			//w1=30;w2=50;w3=20;	
			
			}



			if(lc[pix]==16 ||lc[pix]==17 || lc[pix]==18){
				w1=50;w2=40;w3=10;
			}

			else if(lc[pix]==1 ||lc[pix]==7 || lc[pix]==8){
				w1=55;w2=30;w3=15;	
			}

			else if(lc[pix]==5 ||lc[pix]==4){
				w1=30;w2=40;w3=30;	
			}

			else {
				w1=40;w2=40;w3=20;	
			}
*/

a2=lambdt_d*(w1*lambdw1+w2*lambdw2+w3*lambdw3)/(w1+w2+w3);
if(a2<0.0075) a2=0.0075;
a3=lambdt_n*(w1*lambdw1+w2*lambdw2+w3*lambdw3)/(w1+w2+w3);
if(a3<0.0075) a3=0.0075;

lambd2[pix]=a2*d_l/86400+a3*(1.0-d_l/86400.0);


//=========================================================================================================================================================
/*
if(lc[pix]==16 ||lc[pix]==17 || lc[pix]==18){
	w1=50;w2=40;w3=10;
}

else if(lc[pix]==1 ||lc[pix]==7 || lc[pix]==8){
	w1=55;w2=30;w3=15;	
}

else if(lc[pix]==5 ||lc[pix]==4){
	w1=30;w2=40;w3=30;	
}

else {
	w1=40;w2=40;w3=20;	
}
*/


			
	//==========���´�����model1��ͬ=================================			
	
	  gpucoef_c(pix,lc[pix],coef,HY2);



/*
     initialValues[16]=CNcd[pix] ;      	 initialValues[17]=CNssd[pix];     	 initialValues[18]=CNsmd[pix];
	
	 initialValues[19]=CNfsd[pix];      	 initialValues[20]=CNfmd[pix];   	 initialValues[21]=CNsm[pix];

	 initialValues[22]=CNm[pix];             initialValues[23]=CNs[pix];         initialValues[24]=CNp[pix];

	 initialValues[25]=CNstem[pix] ;    	 initialValues[26]=CNleaf[pix];   	 initialValues[27]=Nav[pix]  ;

	 initialValues[29]=Nt[pix]  ;
 */
	 //=================================����̼��Ͳ�̼���̼����==========================================

	 gpusoilresp(pix,lc[pix],coef,*jday,lat[pix],tmean,xx[pix].x24,Ndep[pix],Ndep0[pix],Cpoolold,CNRold,Cpoolnew,CNRnew,lambd[pix],lambd2[pix]);

/*
			Cstem_n[pix] =initialValues[1]; 			Cleaf_n[pix] =initialValues[3];          		Ccroot_n[pix]=initialValues[2];
			
			Cfroot_n[pix]=initialValues[4]; 			Ccd_n[pix]   = initialValues[5];  			Csmd_n[pix]  = initialValues[7]; 
			
			Cssd_n[pix]  = initialValues[6];			Cfmd_n[pix]  = initialValues[9];  			Cfsd_n[pix]  = initialValues[8];
			
			Csm_n[pix]   = initialValues[10];			Cm_n[pix]    = initialValues[11]; 			Cs_n[pix]    = initialValues[12];
			
			Cp_n[pix]    =initialValues[13];  

			
	CNcd_n[pix]   =initialValues[16];      	CNssd_n[pix]  =initialValues[17];      	CNsmd_n[pix]  =initialValues[18];

	CNfsd_n[pix]  =initialValues[19];     	CNfmd_n[pix]  =initialValues[20];     	CNsm_n[pix]   =initialValues[21];

    CNm_n[pix]    =initialValues[22];         CNs_n[pix]    =initialValues[23];         CNp_n[pix]    =initialValues[24];

    CNstem_n[pix] =initialValues[25];     	CNcroot_n[pix]=initialValues[25];     	CNleaf_n[pix]=initialValues[26] ;

	CNfroot_n[pix]=initialValues[26];     	Nav_n[pix]     =initialValues[27];        Nt_n[pix]     =initialValues[29] ;

*/



			hr[pix]    = hr[pix]+Cpoolnew[pix].Hr;  //��������	
			nep[pix]   = npp[pix]-hr[pix];           //NEP
			
             outputnep[pix+pix1]        =(nep[pix]*10.0); //signed short ǿ�ư�Npp��Ϊ����������					
			 outputnep[4950+pix+pix1]   =(npp[pix]*10.0); //signed short ǿ�ư�Npp��Ϊ����������					
			 outputnep[4950*2+pix+pix1] =(xx[pix].x11*5000.0); 
			 //outputnep[4950*3+pix+pix1] =((xx[pix].x4+xx[pix].x5)*10.0); 
			// outputnep[4950*4+pix+pix1] =((xx[pix].x31+xx[pix].x5*0)*10.0); 
}			   			   
} //=========================End of PIXEL cycle PPPPPPPPPP     ��Ԫѭ������
}

//==========================================����gpumodel1 �����NPP�ͷֽ��ٶ���ʪ�ȵ���������ƽ��ֵ��ʼ̼================

__global__ static void gpuInit(long*pix_offset,hy_c2*HY2,int short*lc,float *CI,float *TI,float*lambd,float *lambd2,float*tnpp,carbonpool *Cpoolold, CNratio *CNRold)

{
	  const int tid = threadIdx.x;
	  const int bid = blockIdx.x;
 	  float coef[60];
	  int long pix;
      int long j;

      float Fm,Nav;
   
	
	 float fw, fcr, fl, ffr; 
	 float kw_cd, kcr_cd;
	 float kl_sl, kfr_fl,kl_sl1;/* sl: surface litter;fl:fine root litter*/ 
	 float kssd_a, kssd_sm, kssd_s, ksmd_a, ksmd_sm;
   	 float kfsd_a, kfsd_m, kfsd_s, kfmd_a, kfmd_m;
     float kcd_s, kcd_a, kcd_m;
	 float km_p, km_a,km_s;
	 float ksm_a, ksm_s, ks_p,ks_a,ks_m, kp_a, kp_m; 	

	float a1,a2,a3,a4,a5,a6,a7,a8;
    float lam1,lam;
    
    long year,day;

    float Cw0, Ccr0, Cl0, Cfr0, Ccd0,Cssd0,Csmd0,Cfsd0,Cfmd0, Cm0, Csm0, Cs0, Cp0;
    float Cw1, Ccr1, Cl1, Cfr1, Ccd1,Cssd1,Csmd1,Cfsd1,Cfmd1, Cm1, Csm1, Cs1, Cp1;
	
	float lam_u,lam_d,part1,part2;
     
	float dCcd,dCssd,dCsmd,dCfsd,dCfmd,dCsm,dCm,dCs, dCp;
	
    float Sc, Sn, NP, Nt;///,u1,u2,u3;
  
    float  b2;
	
    float CNl_av;

 	for(pix=pixoffset+bid * THREAD_NUM + tid;pix<npixels;pix+=BLOCK_NUM * THREAD_NUM)

	{
        if(lc[pix]>0 && lc[pix]!=21 && lc[pix]!=19) { //����½����Ԫ
	
        lambd[pix]=lambd[pix]/((endyear1-startyear+1)*365.0);
          
 lambd2[pix]=lambd2[pix]/((endyear1-startyear+1)*365.0);

        gpucoef_c(pix,lc[pix],coef,HY2);
     	  
		//gpupreInitials(tnpp[pix],lambd[pix],lamb,Tmean_m,NPPd_m,coef, initValues);

  
	Nav=0; 
     
    fw     =coef[0];     //The ratio of NPP allocated to stem
	fcr    = coef[1];     //The ratio of NPP allocated to coarse roots
	fl     =coef[2];     //The ratio of NPP allocated to leaves
	ffr    = coef[3];     //The ratio of NPP allocated to fine roots
	kw_cd  = coef[4];     //The turn over rate of stem pool
	kcr_cd =coef[5];     //The turn over rate of coarse root pool
	kl_sl1  =coef[6];     //The turn over rate of leaf pool
	
	 if(lc[pix]==16 || lc[pix]==17 || lc[pix]==18) kl_sl=kl_sl1;    //����ũ���������������0.6���ջ�
	 else                                          kl_sl=kl_sl1;
	
	kfr_fl = coef[7];     //The turn over rate of fine root pool

    kssd_a = coef[8]*365;     //surface structural litter pool
    kssd_sm =coef[9]*365;      
    kssd_s = coef[10]*365;
	
	ksmd_a = coef[11]*365;   //surface metabolic litter  pool
	ksmd_sm =coef[12]*365;
    
    kfsd_a = coef[13]*365;   //soil structural litter pool
    kfsd_m = coef[14]*365;      
    kfsd_s = coef[15]*365;
	
	kfmd_a = coef[16]*365;  //soil metabolic litter  pool
	kfmd_m = coef[17]*365;	

	kcd_a =  coef[18]*365;   //coarse detritus litter  pool
    kcd_m =  coef[19]*365;
    kcd_s =  coef[20]*365;

	km_a =   coef[21]*365;   //soil microbial C pool
	km_p =   coef[22]*365;
    km_s =   coef[23]*365 ;

    ksm_a=   coef[24]*365;   //surface microbial C pool
	ksm_s=   coef[25]*365;

	ks_a =   coef[26]*365;  //slow C  pool
	ks_p =   coef[27]*365;
    ks_m =   coef[28] *365;

	kp_a =   coef[29]*365;   //passive C pool
    kp_m =   coef[30]*365;


   	lam1  =lambd[pix];  //for surface pools
    lam   =lambd2[pix];  // for soil pools      
  
 	
	//totalNup=0;


    if(lc[pix]==13 || lc[pix]==16 || lc[pix]==17 || lc[pix]==18) Fm=0.6;
	else 	Fm=0.3;//0.85-0.018*0.6*CNl;   //2014��8��20����0.2 ��Ϊ0.3 
    	 
	Cw1 =   (fw/kw_cd)* tnpp[pix] ;
	Ccr1=  (fcr/kcr_cd)* tnpp[pix];
	Cl1 =   (fl/kl_sl1) * tnpp[pix];
	Cfr1=  (ffr/kfr_fl)*tnpp[pix];
 
    Cssd1=(1-Fm)*kl_sl*Cl1/(kssd_a+kssd_sm+kssd_s);
    Csmd1=Fm*kl_sl*Cl1/(ksmd_a+ksmd_sm);
    Cfsd1=(1-Fm)*kfr_fl*Cfr1/(kfsd_a+kfsd_m+kfsd_s);
    Cfmd1=Fm*kfr_fl*Cfr1/(kfmd_a+kfmd_m);
    Ccd1 =((kw_cd*Cw1+kcr_cd*Ccr1)/(kcd_a+kcd_m+kcd_s));
    Cssd1= Cssd1/lam1; 
    Csmd1= Csmd1/lam1;
    Cfsd1= Cfsd1/lam; 
    Cfmd1= Cfmd1/lam;
    Ccd1 = Ccd1/lam; 

   Csm1 = (Cssd1*kssd_sm+Csmd1*ksmd_sm)/(ksm_a+ksm_s);

   a1=Cfsd1*(kfsd_m*lam*(ks_a*lam+ks_p*lam+ks_m*lam)*(kp_a*lam+kp_m*lam)+(kp_a*lam+kp_m*lam)*ks_m*lam*kfsd_s*lam+ks_p*lam*kfsd_s*lam*kp_m*lam);
   
   a2=Cfmd1*kfmd_m*lam*(kp_a*lam+kp_m*lam)*(ks_a*lam+ks_p*lam+ks_m*lam);
   
   a3=Ccd1*(kcd_m*lam*(ks_a*lam+ks_p*lam+ks_m*lam)*(kp_a*lam+kp_m*lam)+(kp_a*lam+kp_m*lam)*ks_m*lam*kcd_s*lam +kp_m*lam*ks_p*lam*kcd_s*lam);
   
   a4=Csm1*(ksm_s*lam1*ks_m*lam*(kp_a*lam+kp_m*lam)+ks_p*lam*kp_m*lam*ksm_s*lam1);
   
   a5=Cssd1*(ks_m*lam*kssd_s*lam1*(kp_a*lam+kp_m*lam)+ks_p*lam*kp_m*lam*kssd_s*lam1);
   
   a6=(km_a*lam+km_p*lam+km_s*lam)*(ks_a*lam+ks_p*lam+ks_m*lam)*(kp_a*lam+kp_m*lam);
   
   a7=km_s*lam*ks_m*lam*(kp_a*lam+kp_m*lam);
      
   a8=kp_m*lam*(km_s*lam*ks_p*lam+km_p*lam*(ks_a*lam+ks_p*lam+ks_m*lam)); 
  
   Cm1=(a1+a2+a3+a4+a5)/(a6-a7-a8);
   Cs1=(Csm1*ksm_s*lam1+Cssd1*kssd_s*lam1+Cfsd1*kfsd_s*lam+Cm1*km_s*lam+Ccd1*kcd_s*lam)/
	   (ks_a*lam+ks_p*lam+ks_m*lam);    
   Cp1=(ks_p*Cs1+km_p*Cm1)/(kp_a+kp_m);

	 
	Cpoolold[pix].Cstem =Cw1; 
	Cpoolold[pix].Cleaf =Cl1;
	Cpoolold[pix].Ccroot=Ccr1;
		
	Cpoolold[pix].Cfroot=Cfr1;
	Cpoolold[pix].Ccd   =Ccd1; 
	Cpoolold[pix].Csmd  =Csmd1 ;

	Cpoolold[pix].Cssd  =Cssd1; 
	Cpoolold[pix].Cfmd  =Cfmd1; 	
	Cpoolold[pix].Cfsd  =Cfsd1;

	Cpoolold[pix].Csm   =Csm1;
	Cpoolold[pix].Cm    =Cm1; 
	Cpoolold[pix].Cs    =Cs1;

	
	Cpoolold[pix].Cp    =Cp1;
			
    CNRold[pix].CNcd  =coef[35];     //Cd  ��̼����
	 CNRold[pix].CNssd =coef[36];     //ssd ��̼����
	 CNRold[pix].CNsmd =coef[37];     //smd ��̼���� 
	 CNRold[pix].CNfsd =coef[38];     //fsd ��̼����
	 CNRold[pix].CNfmd =coef[39] ;    //fmd ��̼����
	 CNRold[pix].CNsm  =coef[41];     //sm  ��̼����
     CNRold[pix].CNm   =coef[42];     //sm  ��̼����
    CNRold[pix]. CNs   =coef[40];     //slow��̼����
     CNRold[pix].CNp   =coef[43];     
     CNRold[pix].CNstem=coef[47];    //Setm C:NinitialValues[25] ;
	 CNRold[pix].CNcroot=coef[47];   //Setm C:NinitialValues[25] ;
	 CNRold[pix].CNleaf=coef[46] ;   //ҶC:N
	 CNRold[pix].CNfroot=coef[46];   //ҶC:N 
	
   Nav=(coef[0]*tnpp[pix]/CNRold[pix].CNstem+coef[1]*tnpp[pix]/CNRold[pix].CNstem+coef[2]*tnpp[pix]/CNRold[pix].CNleaf+coef[3]*tnpp[pix]/CNRold[pix].CNleaf)*20; 
	
   
   CNRold[pix].Nav     =Nav;
	
	 
	 CNRold[pix].Nt=0;	

	}

		else {

         
			Cpoolold[pix].Cstem =0.1;
			Cpoolold[pix].Cleaf =0.1;
			Cpoolold[pix].Ccroot=0.1;

			Cpoolold[pix].Cfroot=0.1;
			Cpoolold[pix].Ccd   =0.1;
			Cpoolold[pix].Csmd  =0.1;

			Cpoolold[pix].Cssd  =0.1;
			Cpoolold[pix].Cfmd  =0.1;
			Cpoolold[pix].Cfsd  =0.1;

			Cpoolold[pix].Csm   =0.1;
			Cpoolold[pix].Cm    =0.1;
			Cpoolold[pix].Cs    =0.1;


			Cpoolold[pix].Cp    =0.1;

			CNRold[pix].CNcd  =0.1;
			CNRold[pix].CNssd =0.1;
			CNRold[pix].CNsmd =0.1;
			CNRold[pix].CNfsd =0.1;
			CNRold[pix].CNfmd =0.1;
			CNRold[pix].CNsm  =0.1;
			CNRold[pix].CNm   =0.1;
			CNRold[pix]. CNs   =0.1;
			CNRold[pix].CNp   =0.1;
			CNRold[pix].CNstem=0.1;
			CNRold[pix].CNcroot=0.1;
			CNRold[pix].CNleaf=0.1;
			CNRold[pix].CNfroot=0.1;
			CNRold[pix].Nav =0.1;
			CNRold[pix].Nt=0;	

	}
	
              }

return;
}

 
//============================================================================================================================================================//
//                                                                                                                                                            //                                                                                                                                          
//                                                                                                                                                            //  
//                                                                                                                                                            //     
//                                                                                                                                                            //     
//============================================================================================================================================================//

int main()
{
    
	FILE  *ff,*ff2,*f_lat,*f_lon,*outfilenpp,*f8,*f9;


    clock_t ttt=clock();
   
    char  outfnpp_name[255],filename[200];
	unsigned char *output,*LC;
		
	int short startyearII;   
	long ptr;
    int control,control1;
	
 int short *outputnep;  
	long yearindex[200];
	int long i,j,ii;		          // zfmtry09-2-2 
	long tmpyear; 
	int short *lc;	
	float  *CI,*TI; 
	// Land cover for a line in a given study area  
	int long pix,lin;		           
	long line,pixel;                  // Position of line,pixel in China, referenced to 0,0 NOT 1,1 		  
	long jday,year,year1;             //zfm3.25
	int long lin_offset,pix_offset,factor;
    short  rcode;	    	          // Error code 
	
	short *ClimateD; 
	 
	float coef[60];                   //NPP���������̼����ת���ʲ�������   
	float b[SIZEB];		              // Biological parameters
	float *lai,*laie,*Ndep0,*Ndep ;	
	float *x;				          //X ��������  
	float *npp,*nppold, *nep,*hr;	  //NPP:net primary productivity; nep:net ecosystem productivity; hr:heterotrophic respiration
	float c[200];
	float co;
	float *tnpp,*thr;    
	
	float *lambd, *lambd2;
	     float *value;
	
  	float *soilw1_old,*soilw2_old,*soilw3_old;
    float *soilw1_n,*soilw2_n,*soilw3_n;


	float *awc;   			          // Av soil water hold cap for a line in a given study area  
	float *lon,*lat;		          // Long, lat values for a line in study area  

  
	struct xvalue *xx;		          // X variables for a line in a given study area 
	struct hy_c1 *HY1;
	struct hy_c2 *HY2;
	struct hy_c3 *HY3;
   
	//===========================================================gpu ��ʼ��==============================================//
	//int *gpunum;
	long* gpupix_offset;
	int short *gpulc;
	float *gpuCI,*gpuTI;
	 int short *gpuoutputnep;
	short *gpusdat;
    long *gpujday;


 //   float *gpulambdd,*gpuTmean,*gpuNPPd,
	float *gpulambd,*gpulambd2,*gputnpp,*gpulai,*gpulaie,*gpunpp,*gpulat,*gpuco;
    float *gpusoilw1_old,*gpusoilw2_old,*gpusoilw3_old, *gpusoilw1_n,*gpusoilw2_n,*gpusoilw3_n;

	float *gpuNdep0,*gpuNdep;


     struct carbonpool *Cpoolold;
     struct carbonpool *Cpoolnew;

     struct CNratio  *CNRold;
     struct CNratio  *CNRnew;

float *gpuhr,*gpunep;
	
	struct xvalue *gpuxx;
	struct hy_c1 *gpuHY1;
	struct hy_c2 *gpuHY2;
	struct hy_c3 *gpuHY3;


	struct carbonpool *gpuCpoolold;
	struct carbonpool *gpuCpoolnew;

	struct CNratio  *gpuCNRold;
	struct CNratio  *gpuCNRnew;





     float ratio;
	
lin_offset=linoffset;
pix_offset=pixoffset;


	//***********============================================gpu ��ʼ������======================================**********//
	 printf("CUDA initialized?\n");
	if(!InitCUDA()) {
		return 0;
	}

	printf("CUDA initialized.\n");
 	
	rcode = NOERROR;   // 0
	
	cycle_display=1;
	output_NPP='Y';
	output_GPP='Y';
	output_res='y';
	output_eva='y';
	output_tra='y';   
	output_site='n'; 
 

for(control=6;control>=1;control=control-1){   // 1: all no change; 2: Climate change only; 3 :CO2 change only; 4 : LAI change only; 5:Ndepostion 6: all change  

 //===============================================================================================================

    //-------	Read default biological paramenters  --------------------------
	readb_init(b);     //��ʼ���� ��45��
//========================================================================

    //����CO2Ũ������
    ff2=fopen("\\Global_Input\\1951_2008co2.txt","rt");	
	for(year=startyear0;year<=endyear2;year++){
	fscanf(ff2,"%d %f",&ii,&c[year-startyear0]);
	printf("%d,%6.2f\n",ii, c[year-startyear0]);
    	}
    fclose(ff2);

	if((f_lat=fopen("\\Global_Input\\Global_latlon\\globallat0727.dat","rb"))==NULL) printf("Can not open latitude file\n");    
	if((f_lon=fopen("\\Global_Input\\Global_latlon\\globallon0727.dat","rb"))==NULL) printf("Can not open longitude file"); // read lat file, X.F April

	hipMalloc((void**) &gpupix_offset, sizeof(long));     //right                                  //1
	hipMemcpy(gpupix_offset, &pix_offset, sizeof(long), hipMemcpyHostToDevice);

	control1=control;
	
	//==== ================================================================================================================= ��ʼ��ѭ��===============================  
			
	for (lin =0 ; lin <(nlines-lin_offset);lin=lin+1 ) { //===========================================================================================================
	    	
	line = lin_offset + lin;     //line Ϊʵ���к�
	//printf("line=%d  %d  %d\n",line, lin_offset,lin);

    LC=(unsigned char*)malloc(npixels*sizeof(unsigned char));   //1       //γ��, ����
    
	lat=(float*)malloc(npixels*sizeof(float));   //2       //γ��, ����
	lon=(float*)malloc(npixels*sizeof(float));      //3    //����, ����
	 
	lc=(int short*)malloc(npixels*sizeof(int short));//4
	awc=(float*)malloc(npixels*sizeof(float));	  //5   
	lai=(float*)malloc(npixels*sizeof(float));  //6
	laie=(float*)malloc(npixels*sizeof(float));  //7
    
	Ndep0=(float*)malloc(npixels*sizeof(float));  //8
	Ndep=(float*)malloc(npixels*sizeof(float));  //9
	
	nppold=(float*)malloc(npixels*sizeof(float)); //10
	npp=(float*)malloc(npixels*sizeof(float));//11
	nep=(float*)malloc(npixels*sizeof(float));//12
	hr=(float*)malloc(npixels*sizeof(float));//13
	
value=(float*)malloc(npixels*sizeof(float));//13;




	output=(unsigned char*)malloc(npixels*sizeof(unsigned char));//14
	outputnep=( int short*)malloc(npixels3*sizeof( int short));//15  //X.F June 20, 2003

	tnpp = (float *)malloc(npixels*sizeof(float));//16  //X.F June 20, 2003   
   
	thr = (float *)malloc(npixels*sizeof(float));//16  //X.F June 20, 2003   


	lambd=(float *)malloc(npixels*sizeof(float));//17   //X.F June 20, 2003;  
	lambd2=(float *)malloc(npixels*sizeof(float));//17   //X.F June 20, 2003;  	
		
	Cpoolold=(struct carbonpool*)malloc(npixels*sizeof(struct carbonpool));  //49
	Cpoolnew=(struct carbonpool*)malloc(npixels*sizeof(struct carbonpool));  //49


	CNRold=(struct CNratio*)malloc(npixels*sizeof(struct CNratio));  //49
	CNRnew=(struct CNratio*)malloc(npixels*sizeof(struct CNratio));  //49


	soilw1_old= (float *)malloc(npixels*sizeof(float));//46  //��һ������ˮ�ֺ��� 
	soilw2_old= (float *)malloc(npixels*sizeof(float));//47  //�ڶ�������ˮ�ֺ��� 
	soilw3_old= (float *)malloc(npixels*sizeof(float));//48  //����������ˮ�ֺ��� 


	soilw1_n= (float *)malloc(npixels*sizeof(float));//46  //��һ������ˮ�ֺ��� 
	soilw2_n= (float *)malloc(npixels*sizeof(float));//47  //�ڶ�������ˮ�ֺ��� 
	soilw3_n= (float *)malloc(npixels*sizeof(float));//48  //����������ˮ�ֺ��� 



CI= (float *)malloc(npixels*sizeof(float));//49  //�ۼ���ָ��  
TI= (float *)malloc(npixels*sizeof(float));//50  //��ƽ���¶�



	for(i=0;i<npixels;i++) {

	lat[i]=0;    	lon[i]=0;      //��γ���м����, ��������
	
	lc[i]=0;     	awc[i]=0; 	lai[i]=0;   	laie[i]=0;
    
	Ndep0[i]=0; 	Ndep[i]=0;
	
	nppold[i]=0; 	npp[i]=0;    	nep[i]=0; 	hr[i]=0;
	//******lixuansong***** 
	output[i]=0; 	outputnep[i]=0; tnpp[i]=0; 	lambd[i]=0;thr[i]=0;lambd2[i]=0;

Cpoolold[i].Cstem=0;    Cpoolold[i].Cleaf=0; Cpoolold[i].Ccroot=0; Cpoolold[i].Cfroot=0;

Cpoolold[i].Ccd=0;      Cpoolold[i].Csmd=0;  Cpoolold[i].Cssd=0;   Cpoolold[i].Cfmd=0; 

 Cpoolold[i].Cfsd=0;    Cpoolold[i].Csm=0;   Cpoolold[i].Cm=0;    Cpoolold[i].Cs=0;     Cpoolold[i].Cp=0; 
	
CNRold[i].CNleaf=20;    CNRold[i].CNstem=20; CNRold[i].CNcroot=20;  CNRold[i].CNfroot=20;  CNRold[i].CNcd=20; CNRold[i].CNsmd=20; 

CNRold[i].CNssd=20;    CNRold[i].CNfmd=20;   CNRold[i].CNfsd=20;    CNRold[i].CNsm=20;     CNRold[i].CNm=20;  CNRold[i].CNs=20;	CNRold[i].CNp=20;  CNRold[i].Nav=20; 

CNRold[i].Nt=20; 



	soilw1_old[i]=0.1; 	soilw2_old[i]=0.1;   soilw3_old[i]=0.1; 
	}

	HY1=(struct hy_c1*)malloc(npixels*sizeof(struct hy_c1));  //49
	HY2=(struct hy_c2*)malloc(npixels*sizeof(struct hy_c2));  //50
	HY3=(struct hy_c3*)malloc(npixels*sizeof(struct hy_c3));   //51

	//lambdd=(float *)malloc(npixels*366*sizeof(float)); //��������Ӱ�������ֽ��ϵ��
	//Tmean=(float *)malloc(npixels*366*sizeof(float));  //Ԥ���ڼ����ƽ���¶�
	//NPPd=(float *)malloc(npixels*366*sizeof(float));  //Ԥ���ڼ��NPP
	
	//for(i=0;i<npixels;i++)lambdd[i]=(float *)malloc(366*sizeof(float)); 

	//*******	Daily Model Parameters  (Currently there are 24) ****
	x=(float*)malloc(SIZEX*sizeof(float));                          //52

	//*****	Memory for model parameters which change daily ****

	xx=(struct xvalue*)malloc(npixels*sizeof(struct xvalue));       //53

//sdat=(struct climatedata*)malloc(npixels*sizeof(struct climatedata));
    ClimateD=(short *) malloc(npixels5*sizeof(short));                 //54

	    
	    fseek(f_lat,1L*(long) line* (long) npixels*sizeof(float),SEEK_SET);  
	    fread(&lat[0],sizeof(float),npixels,f_lat);     
		
        fseek(f_lon,1L*(long) line* (long) npixels*sizeof(float),SEEK_SET); 
		fread(&lon[0],sizeof(float),npixels,f_lon);      
		

		for(pix=0; pix<npixels; pix++) {
		pixel = pix_offset + pix; 
		zeroxx1(pix,x,xx);              //�����м���� 
		}	     


		printf("\n Firest SIMULATION IN LINE control=%d line=%d\n", control,line+1);	  

        printf("Land cover 1\n");

        f9=fopen("\\Global_Input\\Global_cover\\Global_0727_landcover.raw", "rb");
	    ptr=(line*npixels )*sizeof(char);   
        fseek(f9,ptr,SEEK_SET);
     	fread(&LC[0],1,npixels,f9);  
    	printf("Land cover 2\n");	
		for(pix=0; pix<npixels; pix++) lc[pix]=LC[pix];
    	fclose(f9);
        printf("Land cover3 \n");  
 
   
 
		f9=fopen("\\Global_Input\\Global_cover\\Global_CI_072727.img", "rb");
		ptr=(line*npixels )*sizeof(float);   
		fseek(f9,ptr,SEEK_SET);
		fread(&CI[0],4,npixels,f9);  
	    fclose(f9);
	

		f9=fopen("\\Global_Input\\Global_cover\\Global_Tmean_072727.dat", "rb");
		ptr=(line*npixels )*sizeof(float);   
		fseek(f9,ptr,SEEK_SET);
		fread(&TI[0],4,npixels,f9);  
		fclose(f9);



 //����������������  
		readsoildata(line,HY1,HY2,HY3,&rcode);              //
		if(rcode == ERROR) {
			printf("���������������ݳ���");
			exit(0);
		}

  printf("Land cover4 \n");  

		for(pix=0; pix<npixels; pix++) { 
			// ��ʼ��������ˮ��������������ʼ��ˮ����������ˮ�����ü���Ըɺ������ԣ�����ƫ��

			soilw1_old[pix]= HY1[pix].FC;                    soilw2_old[pix]= HY2[pix].FC;        soilw3_old[pix]= HY3[pix].FC; 
			
			if(soilw1_old[pix]<0.025) soilw1_old[pix]=0.025; 	 if(soilw1_old[pix]>0.6) soilw1_old[pix]=0.6;
            if(soilw2_old[pix]<0.025) soilw2_old[pix]=0.025; 	 if(soilw2_old[pix]>0.6) soilw2_old[pix]=0.6;
			if(soilw3_old[pix]<0.025) soilw3_old[pix]=0.025; 	 if(soilw3_old[pix]>0.6) soilw3_old[pix]=0.6;
			
			//soilw2_old[pix]=__max(0.05,soilw2_old[pix]);     soilw2_old[pix]=__min(0.6,soilw2_old[pix]);
			//soilw3_old[pix]=__max(0.05,soilw3_old[pix]);     soilw3_old[pix]=__min(0.6,soilw3_old[pix]);

		}
		/////////////gpu==================================================================22222222222222222222222
		
		  printf("Land cover5 \n");  
		
		hipMalloc((void**) &gpuHY1, npixels*sizeof(struct hy_c1));   //2
		hipMalloc((void**) &gpuHY2, npixels*sizeof(struct hy_c2));   //3
		hipMalloc((void**) &gpuHY3, npixels*sizeof(struct hy_c3));    //4 
		  printf("Land cover6 \n");  
		
		hipMalloc((void**) &gpulat, npixels*sizeof(float));   //5
		hipMalloc((void**) &gpulc, npixels*sizeof(int short)); //6
		
		hipMalloc((void**) &gpusoilw1_old, npixels*sizeof(float)); //7
		hipMalloc((void**) &gpusoilw2_old, npixels*sizeof(float)); //8
		hipMalloc((void**) &gpusoilw3_old, npixels*sizeof(float)); //9


		hipMalloc((void**) &gpusoilw1_n, npixels*sizeof(float)); //7
		hipMalloc((void**) &gpusoilw2_n, npixels*sizeof(float)); //8
		hipMalloc((void**) &gpusoilw3_n, npixels*sizeof(float)); //9


		hipMalloc((void**) &gpuCI, npixels*sizeof(float)); //10
        hipMalloc((void**) &gpuTI, npixels*sizeof(float)); //10

		hipMemcpy(gpuHY1,HY1,npixels*sizeof(struct hy_c1),hipMemcpyHostToDevice);//c2
		hipMemcpy(gpuHY2,HY2,npixels*sizeof(struct hy_c2),hipMemcpyHostToDevice);//c3
		hipMemcpy(gpuHY3,HY3,npixels*sizeof(struct hy_c3),hipMemcpyHostToDevice);//c4
		
		hipMemcpy(gpulat,lat,npixels*sizeof(float),hipMemcpyHostToDevice); //c5
		hipMemcpy(gpulc,lc,npixels*sizeof(int short),hipMemcpyHostToDevice);//c6
       	
		hipMemcpy(gpusoilw1_old,soilw1_old,npixels*sizeof(float),hipMemcpyHostToDevice);//c7
		hipMemcpy(gpusoilw2_old,soilw2_old,npixels*sizeof(float),hipMemcpyHostToDevice);//c8
		hipMemcpy(gpusoilw3_old,soilw3_old,npixels*sizeof(float),hipMemcpyHostToDevice);  ////c9

        hipMemcpy(gpuCI,CI,npixels*sizeof(float),hipMemcpyHostToDevice);//c10
        hipMemcpy(gpuTI,TI,npixels*sizeof(float),hipMemcpyHostToDevice);//c10
		
		
		hipMalloc((void**) &gpuCpoolold,  npixels*sizeof(struct carbonpool));  //28

        hipMalloc((void**) &gpuCpoolnew,  npixels*sizeof(struct carbonpool));  //28

		hipMalloc((void**) &gpuCNRold,    npixels*sizeof(struct CNratio));  //28

		hipMalloc((void**) &gpuCNRnew,    npixels*sizeof(struct CNratio));  //28

		
		
		
		
		
		
		
		
		for(pix=0; pix<npixels; pix++){
		xx[pix].x1=0;   	tnpp[pix]=0; 		lambd[pix]=0; thr[pix]=0; lambd2[pix]=0;
		   
	   }

		
		if(control==6){//***************************************************************************************************************************************
	
		hipMalloc((void**) &gpulambd, npixels*sizeof(float));                                   //10
		hipMemcpy(gpulambd,lambd,npixels*sizeof(float),hipMemcpyHostToDevice);

		hipMalloc((void**) &gpulambd2, npixels*sizeof(float));                                   //10
		hipMemcpy(gpulambd2,lambd2,npixels*sizeof(float),hipMemcpyHostToDevice);

       	//=======================================================================================��һ����ѭ��================================================================/
		for(year=startyear;year<=endyear1;year++){//��ѭ����ʼ
		//===================================================================================================================================================================
			printf("\n year= %d...\n", year);	
			
				for(pix=0; pix<npixels; pix++) {
				pixel = pix_offset + pix; 
				npp[pix]=0.0; 	zeroxx(pix,x,xx);   output[pix]= 0;
				         
			} // end zfm����ʼֵ 

			//����LAI�����ֵ���ڹ��������� 
           tmpyear=startyear2;
        	readlaimax(line,xx,lc,&rcode,tmpyear);   

			if (rcode == ERROR) {
				printf ("�������LAI���� "); exit(0);
			}
		
           		    
			co=c[0];   
			printf("%6.2f\n", co);

			//2013-10-25
			    hipMalloc((void**) &gpusdat, npixels5*sizeof(short));    //11
				hipMalloc((void**) &gpujday, sizeof(long));              //12 
				hipMalloc((void**) &gpunpp, npixels*sizeof(float));      //14
			    hipMalloc((void**) &gpuco, sizeof(float));                //16
			    hipMalloc((void**) &gpulai, npixels*sizeof(float));       //18
     		    hipMalloc((void**) &gpuxx, npixels*sizeof(struct xvalue));   //20
			    hipMemcpy(gpuco, &co, sizeof(float), hipMemcpyHostToDevice);
						
			for (jday=jday_start; jday<=jday_end; jday=jday++) {   //===================================================��ʱ��ѭ����ʼ=========================
	           if(jday%90==1) printf("jday=%d",jday);
				clock_t ttt1=clock();
                clock_t ttt2;//,ttt3,ttt4;

				//========================================������������============================================================================================		
			    //	getchar();

             readclim(year,jday,line,ClimateD,&rcode); //����ÿ�����������
	
	          if (rcode == ERROR) {
		      printf("�����������ݷ�������");	 exit(0);
              }


             ttt2=clock();
             if(jday==jday_start)
             printf("timh:%d\n",ttt2-ttt1);

			 
 //JUW2013  
     if(control1== 4 || control1==6){    // 1: all no change; 2: Climate change only; 3 :CO2 change only; 4 : LAI change only; 5:N_deposition,6: all change
	 
		 if(year<=2000){		 //2000��ǰ����AVHRR��LAI����

	if( jday==1||jday==16||jday==32||jday==47||jday==60||jday==75||jday==91||jday==106||jday==121
		||jday==136||jday==152||jday==167||jday==182||jday==197||jday==213||jday==228||jday==244
		||jday==259||jday==274||jday==289||jday==305||jday==320||jday==335||jday==350)
	{	               
		tmpyear=year;              
		               
		//printf("tmpyear0=%d    \n",tmpyear);
		
		readlai(line,laie,jday,&rcode,tmpyear);     //��ȡ��ЧҶ���ָ��
						if (rcode == ERROR){
							printf("����LAI���ݴ���"); 	  
							exit(0);
 	
						}
						
						} 
   			}  //the end of year<2000


					else {  //2000�꿪ʼ����MODIS ��LAI����
 
						if((jday-1)%8==0){ //�ж��Ƿ��LAI���ݣ�MODIS��LAI��8��һ�� 
							readlai(line,laie,jday,&rcode,tmpyear);     //��ȡ��ЧҶ���ָ��
							if (rcode == ERROR){
								printf("����LAI���ݴ���"); 	  
								exit(0);
							}

						}  // the end of if((jday-1)%8==0)
					
                      } //control1== 4    ����LAI���ݽ���
					
}  
    else{  //����control==1,2,3���龰��
// 1: all no change; 2: Climate change only; 3 :CO2 change only; 4 : LAI change only; 5: all change
//========================================================================================================================================================================================//			
	
		tmpyear=year;   
		
		if( jday==1||jday==16||jday==32||jday==47||jday==60||jday==75||jday==91||jday==106||jday==121
			||jday==136||jday==152||jday==167||jday==182||jday==197||jday==213||jday==228||jday==244
			||jday==259||jday==274||jday==289||jday==305||jday==320||jday==335||jday==350)
	                 
	                    {
                       	readlai(line,laie,jday,&rcode,tmpyear);     //��ȡ��ЧҶ���ָ��
						if (rcode == ERROR){
						printf("����LAI���ݴ���"); 	  
						exit(0);
						} 
                    	}
//========================================================================================================================================================================================//
 	
}	 	
					////////gpu//lixuansong
					hipMalloc((void**) &gpulaie, npixels*sizeof(float));      //21
					hipMemcpy(gpulaie,laie,npixels*sizeof(float),hipMemcpyHostToDevice);

					gpuLAI<<<BLOCK_NUM, THREAD_NUM,0>>>(gpulc,gpulaie,gpulai);
					hipMemcpy(lai,gpulai,npixels*sizeof(float),hipMemcpyDeviceToHost);
					hipFree(gpulaie);  //right                     //21
   
                  //2013-10-25
			     
		
				hipMemcpy(gpunpp,npp,npixels*sizeof(float),hipMemcpyHostToDevice);
				hipMemcpy(gpusdat,ClimateD,npixels*5*sizeof(short),hipMemcpyHostToDevice);
				hipMemcpy(gpujday,&jday,sizeof(long),hipMemcpyHostToDevice);

				hipMemcpy(gpuxx,xx,npixels*sizeof(struct xvalue),hipMemcpyHostToDevice);
		      
               hipMemcpy(gpusoilw1_old,soilw1_old,npixels*sizeof(float),hipMemcpyHostToDevice);
               hipMemcpy(gpusoilw2_old,soilw2_old,npixels*sizeof(float),hipMemcpyHostToDevice);
               hipMemcpy(gpusoilw3_old,soilw3_old,npixels*sizeof(float),hipMemcpyHostToDevice);
			
gpuModel1<<<BLOCK_NUM, THREAD_NUM,0>>>(gpujday,gpupix_offset,gpulc,gpuCI,gpuTI,gpusdat,gpulai,gpunpp,gpuxx,gpulat,gpusoilw1_old,gpusoilw2_old,gpusoilw3_old,
									   gpuHY1,gpuHY2,gpuHY3,gpuco,gpulambd,gpulambd2);
		
				hipMemcpy(npp,gpunpp,npixels*sizeof(float),hipMemcpyDeviceToHost);
				hipMemcpy(xx,gpuxx,npixels*sizeof(struct xvalue),hipMemcpyDeviceToHost);
			           				 
				hipMemcpy(soilw1_old,gpusoilw1_old,npixels*sizeof(float),hipMemcpyDeviceToHost);
				hipMemcpy(soilw2_old,gpusoilw2_old,npixels*sizeof(float),hipMemcpyDeviceToHost);
				hipMemcpy(soilw3_old,gpusoilw3_old,npixels*sizeof(float),hipMemcpyDeviceToHost);
					 				
				j=j++;

                if(jday==jday_start)
                printf("time:%d\n",clock()-ttt2);
			
                 }//===========================//dayѭ������===================================================================================

           //2013-10-25
			hipFree(gpujday);  //13
			hipFree(gpusdat);  //12
			hipFree(gpunpp);   //14
						
			hipFree(gpuxx);     //20
			hipFree(gpulai);    //18
			hipFree(gpuco);     //16
	
			for(pix=pix_offset;pix<npixels;pix++){
	    	tnpp[pix]=(float) tnpp[pix]+npp[pix] /(endyear1-startyear+1);
         	}
           



   }//=================================��ѭ������===============================================================================
	 

				
		clock_t t0=clock();

	     
		hipMalloc((void**) &gputnpp, npixels*sizeof(float));     //26
		hipMemcpy(gputnpp,tnpp,npixels*sizeof(float),hipMemcpyHostToDevice);
		
		
		//clock_t* time;
		
		//��ʼ������̼��
		 
		 gpuInit<<<BLOCK_NUM, THREAD_NUM,0>>>(gpupix_offset,gpuHY2,gpulc,gpuCI,gpuTI,gpulambd,gpulambd2,gputnpp,gpuCpoolold,gpuCNRold) ;
 
	 
	 	hipMemcpy(Cpoolold,gpuCpoolold,npixels*sizeof(struct carbonpool),hipMemcpyDeviceToHost);
		hipMemcpy(CNRold,  gpuCNRold,  npixels*sizeof(struct CNratio),   hipMemcpyDeviceToHost);
		 


        hipFree(gpulambd);  //10
        hipFree(gpulambd2);  //10

        hipFree(gputnpp);    //26

	pix=1533;
		
		printf("Nt=%f,Nav=%f\n",CNRold[pix].Nt,CNRold[pix].Nav); 

		printf("\n");printf("\n");

	
		}  //the end of control==6====================================================================================================================================
  
else {  //���������龰��ֱ�Ӷ�������ʼ������
	  
	
	ptr=(lin*npixels )*sizeof(float); 
      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_stem%d_%d.img",startyear2-1,365); //1
	  f9=fopen(outfnpp_name,"rb");

      fseek(f9,ptr,SEEK_SET);
     
	  fread(&value[0],sizeof(float),npixels,f9);
      for(i=0;i<npixels;i++) Cpoolold[i].Cstem=value[i];
	  fclose(f9);  

      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_leaf%d_%d.img",startyear2-1,365); //2
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
     
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++) Cpoolold[i].Cleaf=value[i];
	  fclose(f9);  

	

      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_Ccroot%d_%d.img",startyear2-1,365); //3
	  f9=fopen(outfnpp_name,"rb");
     fseek(f9,ptr,SEEK_SET);

	 fread(&value[0],sizeof(float),npixels,f9);
	 for(i=0;i<npixels;i++) Cpoolold[i].Ccroot=value[i];
	 fclose(f9);  
  

      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_froot%d_%d.img",startyear2-1,365); //4
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++) Cpoolold[i].Cfroot=value[i];
	  fclose(f9);  


      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_Cd%d_%d.img",  startyear2-1,365);  //5
	  f9=fopen(outfnpp_name,"rb");
     fseek(f9,ptr,SEEK_SET);
	 fread(&value[0],sizeof(float),npixels,f9);
	 for(i=0;i<npixels;i++) Cpoolold[i].Ccd=value[i];
	 fclose(f9);  
	 
	
      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_smd%d_%d.img", startyear2-1,365);  //6
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);

	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++) Cpoolold[i].Csmd=value[i];
	  fclose(f9);  

      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_ssd%d_%d.img", startyear2-1,365);  //7
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++) Cpoolold[i].Cssd=value[i];
	  fclose(f9);  

      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_fmd%d_%d.img",startyear2-1,365);  //8
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++) Cpoolold[i].Cfmd=value[i];
	  fclose(f9);  

      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_fsd%d_%d.img",startyear2-1,365);  //9
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++) Cpoolold[i].Cfsd=value[i];
	  fclose(f9);  

      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_sm%d_%d.img",startyear2-1,365);  //10
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++) Cpoolold[i].Csm=value[i];
	  fclose(f9);  

      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_m%d_%d.img",startyear2-1,365);  //11
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++) Cpoolold[i].Cm=value[i];
	  fclose(f9);  

      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_s%d_%d.img",startyear2-1,365);  //12
	  f9=fopen(outfnpp_name,"rb");
     fseek(f9,ptr,SEEK_SET);
	 fread(&value[0],sizeof(float),npixels,f9);
	 for(i=0;i<npixels;i++) Cpoolold[i].Cs=value[i];
	 fclose(f9);    

      sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_p%d_%d.img",startyear2-1,365);  //13
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++) Cpoolold[i].Cp=value[i];
	  fclose(f9);  

//==========================================================for C:N files========================================

      ptr=(lin*npixels )*sizeof(float); 
      
	  sprintf(outfnpp_name,"h:\\P7_output_all\\CN_stem%d_%d.img",startyear2-1,365); //1
	 
	
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
     

	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNstem=value[i];
	  fclose(f9);  

     sprintf(outfnpp_name,"h:\\P7_output_all\\CN_leaf%d_%d.img",startyear2-1,365); //2
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
     
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNleaf=value[i];
	  fclose(f9);  




      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_Ccroot%d_%d.img",startyear2-1,365); //3
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNcroot=value[i];
	  fclose(f9);  


      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_froot%d_%d.img",startyear2-1,365); //4
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNfroot=value[i];
	  fclose(f9);  


      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_Cd%d_%d.img",startyear2-1,365);  //5
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNcd=value[i];
	  fclose(f9);  


      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_smd%d_%d.img",startyear2-1,365);  //6
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNsmd=value[i];
	  fclose(f9);  


      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_ssd%d_%d.img",startyear2-1,365);  //7
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNssd=value[i];
	  fclose(f9);  


      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_fmd%d_%d.img",startyear2-1,365);  //8
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNfmd=value[i];
	  fclose(f9);  

      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_fsd%d_%d.img",startyear2-1,365);  //9
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNfsd=value[i];
	  fclose(f9);  


      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_sm%d_%d.img",startyear2-1,365);  //10
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNsm=value[i];
	  fclose(f9);  



      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_m%d_%d.img",startyear2-1,365);  //11
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNm=value[i];
	  fclose(f9);  



      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_s%d_%d.img",startyear2-1,365);  //12
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNs=value[i];
	  fclose(f9);  



      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_p%d_%d.img",startyear2-1,365);  //13
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].CNp=value[i];
	  fclose(f9);  



  
      sprintf(outfnpp_name,"h:\\P7_output_all\\Nav_%d_%d.img",startyear2-1,365);  //14
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
	  fread(&value[0],sizeof(float),npixels,f9);
	  for(i=0;i<npixels;i++)CNRold[i].Nav=value[i];
	  fclose(f9);  


 
      sprintf(outfnpp_name,"h:\\P7_output_all\\SW1_%d_%d.img",startyear2-1,365);  //15
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
      fread(&soilw1_old[0],sizeof(float),npixels,f9);
      fclose(f9);  

      
      sprintf(outfnpp_name,"h:\\P7_output_all\\SW2_%d_%d.img",startyear2-1,365);  //15
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
      fread(&soilw2_old[0],sizeof(float),npixels,f9);
      fclose(f9);  
  
      sprintf(outfnpp_name,"h:\\P7_output_all\\SW3_%d_%d.img",startyear2-1,365);  //15
	  f9=fopen(outfnpp_name,"rb");
      fseek(f9,ptr,SEEK_SET);
      fread(&soilw3_old[0],sizeof(float),npixels,f9);
      fclose(f9);  
     

}   //the end of control!=6 


    	//==================================================================================2018-03-07�����漸�йر� 
	//	for(pix=0; pix<npixels; pix++) {
		//	pixel = pix_offset + pix; 
			//zeroxx1(pix,x,xx);              //�ӳ���
		//}	     






//===============================================================================================================================================
		printf("\n Second SIMULATION IN control=%d   LINE=%d...\n", control,line+1);	  

		// 	  Read landcover values (for each pixel in the line)  
		//readlc(line,lc,&rcode);                               
		//if (rcode == ERROR) {
		//	exit(0);
		//}

        f9=fopen("\\Global_Input\\Global_cover\\Global_0727_landcover.raw", "rb");
	    ptr=(line*npixels )*sizeof(char);   
        fseek(f9,ptr,SEEK_SET);
     	fread(&LC[0],1,npixels,f9);  for(pix=0; pix<npixels; pix++) lc[pix]=LC[pix];
        fclose(f9);
 

		f9=fopen("\\Global_Input\\Global_cover\\Global_CI_072727.img", "rb");
		ptr=(line*npixels )*sizeof(float);   
		fseek(f9,ptr,SEEK_SET);
		fread(&CI[0],4,npixels,f9);  
		fclose(f9);

		f9=fopen("\\Global_Input\\Global_cover\\Global_Tmean_072727.dat", "rb");
		ptr=(line*npixels )*sizeof(float);   
		fseek(f9,ptr,SEEK_SET);
		fread(&TI[0],4,npixels,f9);  
		fclose(f9);




       hipMemcpy(gpulc,lc,npixels*sizeof(int short),hipMemcpyHostToDevice);//c6

	   hipMemcpy(gpuCI,CI,npixels*sizeof(float),hipMemcpyHostToDevice);//c6
       hipMemcpy(gpuTI,TI,npixels*sizeof(float),hipMemcpyHostToDevice);//c6


    	// Read available water holding capacity for pixels in a line 
		readsoildata(line,HY1,HY2,HY3,&rcode); //            
		if(rcode == ERROR) {
		exit(0);
		}
		
		
		for(pix=0; pix<npixels; pix++){
			xx[pix].x1=0;   
				}

 /*	
f8=fopen("test.txt","wt");
fprintf(f8,"%s,%s,%s,%s,%s,","jday","Cstem","Ccroot","Cfroot", "Cleaf"); 
fprintf(f8,"%s,%s,%s,%s,%s,%s,%s,%s,%s,", "Ccd","Csmd","Cssd","Cfmd","Cfsd","Csm","Cm","Cs","Cp"); 
fprintf(f8,"%s,%s,%s,%s,","CNstem","CNcroot","CNfroot", "CNleaf"); 
fprintf(f8,"%s,%s,%s,%s,%s,%s,%s,%s,%s,%s,%s,%s,%s\n", "CNcd","CNsmd","CNssd","CNfmd","CNfsd","CNsm","CNm","CNs","CNp","Nav","NPP","Hr","NEP"); 
*/
 	

//==================================================��ʼ��ʽģ�����===============================================================================/

		if(control==6){//�����龰��ֱ�Ӷ�������
	
			for(pix=0; pix<npixels; pix++) { 
			// ��ʼ�������� ˮ��������������ʼ��ˮ����������ˮ�����ü���Ըɺ������ԣ�����ƫ��
			soilw1_old[pix]= HY1[pix].FC;                   soilw2_old[pix]= HY2[pix].FC;        soilw3_old[pix]= HY3[pix].FC; 
			//soilw1_old[pix]=__max(0.05,soilw1_old[pix]); 	soilw1_old[pix]=__min(0.6,soilw1_old[pix]);
			//soilw2_old[pix]=__max(0.05,soilw2_old[pix]);     soilw2_old[pix]=__min(0.6,soilw2_old[pix]);
			//soilw3_old[pix]=__max(0.05,soilw3_old[pix]);     soilw3_old[pix]=__min(0.6,soilw3_old[pix]);


			if(soilw1_old[pix]<0.025) soilw1_old[pix]=0.025; 	 if(soilw1_old[pix]>0.6) soilw1_old[pix]=0.6;
			if(soilw2_old[pix]<0.025) soilw2_old[pix]=0.025; 	 if(soilw2_old[pix]>0.6) soilw2_old[pix]=0.6;
			if(soilw3_old[pix]<0.025) soilw3_old[pix]=0.025; 	 if(soilw3_old[pix]>0.6) soilw3_old[pix]=0.6;

		}
	 }  //�龰6����

		

		hipMemcpy(gpusoilw1_old,soilw1_old,npixels*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(gpusoilw2_old,soilw2_old,npixels*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(gpusoilw3_old,soilw3_old,npixels*sizeof(float),hipMemcpyHostToDevice);
		//above================================== juw===========================================================================================================
			
	
		
		hipMalloc((void**) &gpulambd, npixels*sizeof(float));    //55       
		hipMemcpy(gpulambd,lambd,npixels*sizeof(float),hipMemcpyHostToDevice);
	

    
		hipMalloc((void**) &gpulambd2, npixels*sizeof(float));    //55       
		hipMemcpy(gpulambd2,lambd2,npixels*sizeof(float),hipMemcpyHostToDevice);


	
		srand(32730);
		for(year=1970;year<=endyear2;year++){//===================================================2013-10-06
        yearindex[year-startyear]=1970+rand()%10; 
       	}
	
       for(year=startyear;year<1980;year++){
        yearindex[year-startyear]=year;
        }


              hipMalloc((void**) &gpunpp, npixels*sizeof(float));	  //15	
			  hipMalloc((void**) &gpuhr, npixels*sizeof(float));       //23
			  hipMalloc((void**) &gpunep, npixels*sizeof(float));       //24   
 		      hipMalloc((void**) &gpuxx, npixels*sizeof(struct xvalue));    //21
		
		
		        hipMalloc((void**) &gpuoutputnep, npixels3*sizeof( int short));  //25
                hipMalloc((void**) &gpusdat, npixels5*sizeof(short));  //12
				hipMalloc((void**) &gpujday, sizeof(long));            //13         
			


			//============================================================================================================================================
            ff2=fopen( "\\Global_Input\\Global_Ndeposition\\1891_I_Annual_dep_00727.dat","rb");
			ptr=sizeof( float)*(line*npixels + pix_offset);
			fseek(ff2,ptr,0);     
            fread(&Ndep0[0],4,npixels,ff2);
            fclose(ff2);
            if (rcode == ERROR) {
			exit(0);
			}


			for(pix=0; pix<npixels; pix++) {
				tnpp[pix]=0.0;  thr[pix]=0.0;
			}	
		//===================================================================================================================================================/

		//=========================================================================================�ڶ�����ѭ����ʼ===========================================

		//===================================================================================================================================================/

		if(control==6) 	startyearII=startyear1;
		else 	        startyearII=startyear2;
			
		for(year=startyearII;year<=endyear2;year++){       //���� startyear1=1871 
		printf("year=%d\n",year);
	
		for(pix=0; pix<npixels; pix++) {
		npp[pix]=0.0;  nppold[pix]=0.0;		nep[pix]=0.0;			hr[pix]=0.0; 		output[pix]= 0;
		zeroxx(pix,x,xx); 	
					}	                            // end zfm����ʼֵ 
        
		hipMalloc((void**) &gpuNdep0, npixels*sizeof(float));    //56
		hipMemcpy(gpuNdep0,Ndep0,npixels*sizeof(float),hipMemcpyHostToDevice);


			//=======����LAI maximum ���ݣ����ڹ���������=================================================================================================== 
			
             if(year<startyear2)  tmpyear=startyear2; 
			 else {
			if(control==4 || control==6 ) tmpyear=year;           //1: ALL no; 2: Climate change only; 3��CO2 change only;4: LAI 5 Ndeposion 6: all change; 
			else tmpyear=startyear2;   //JUW2013
			 }
			readlaimax(line,xx,lc,&rcode,tmpyear);    //��LAI�����ֵ
			if (rcode == ERROR) {
			exit(0);
			}
			
            //============================================================���뵪��������======================================================================
			if(year<startyear2)  tmpyear=year; 
			 else {
			if(control==5 || control==6 ) tmpyear=year;           //1: ALL no; 2: Climate change only; 3��CO2 change only;4: LAI 5 Ndeposion 6: all change; 
			else tmpyear=startyear2;   //JUW2013
			 }

		    sprintf(filename, "\\Global_Input\\Global_Ndeposition\\%d%s",tmpyear,"_I_Annual_dep_00727.dat");
			ff2=fopen( filename,"rb");
			ptr=sizeof( float)*(line*npixels + pix_offset);
			fseek(ff2,ptr,0);     
            fread(&Ndep[0],4,npixels,ff2);
            fclose(ff2);
           //========================================================================================================================================================= 
		
			                  
            hipMalloc((void**) &gpuNdep, npixels*sizeof(float));  //57
			hipMemcpy(gpuNdep,Ndep,npixels*sizeof(float),hipMemcpyHostToDevice);

		  if(year>=startyear2){                                        //startyear=1951 ;startyear0=1901
           if(control==3 || control==6) co=c[year-startyear0];           //1: ALL no; 2: Climate change only; 3��CO2 change only;4: LAI 5:N deposition 6: all change; 
		   else            	            co=c[startyear2-startyear0];      //co=c[startyear-startyear];   //2013-11-06
			}	
	    	else                        co=c[year-startyear0];  
	
//printf("year=%d co=%f %f %f\n",year,co,c[0],c[1]);


		    hipMalloc((void**) &gpuco, sizeof(float));      //17
			hipMemcpy(gpuco, &co, sizeof(float), hipMemcpyHostToDevice);

                

//==================================================================================================================================================
 
			
for (jday=jday_start; jday<=jday_end; jday=jday++) {   //===============================ʱ��ѭ����ʼ=============================
                 if(jday%90==1) printf("jday=%d",jday);		

				 //start to read climate
	    		if (year>=startyear2){      //startyear2=1981
                if(control==2 || control==6  ) readclim(year,jday,line,ClimateD,&rcode);         //1: ALL no; 2: Climate change only; 3��CO2 change only;4: LAI 5 Ndepostion 6: all change; 
	                      else                 readclim(yearindex[year-startyear],jday,line,ClimateD,&rcode);			
		 
				}    //startyear2���Ժ�
				else   {  //1980����ǰ
                
				  if(year<startyear)  year1=startyear+(year-startyear1)%10;	 //startyear:�������ݿ�ʼ��			
				   else   year1=year;	

				   readclim(year1,jday,line,ClimateD,&rcode);                            
				}

						 
					
					if (rcode == ERROR) {
						printf("�����������ݷ�������") ;	    
						exit(0);
					}
						
	        //the end of reading climate data			
 
                    
					if(control==4  || control==6	){//1: ALL no; 2: Climate change only; 3��CO2 change only;4: LAI 5 Ndepostion 6: all change; 
				if(year<=2000){
                  if(year<startyear2)      tmpyear=1980;                           //
				  else                     tmpyear=year;
			
				  if( jday==1||jday==16||jday==32||jday==47||jday==60||jday==75||jday==91||jday==106||jday==121
					  ||jday==136||jday==152||jday==167||jday==182||jday==197||jday==213||jday==228||jday==244
					  ||jday==259||jday==274||jday==289||jday==305||jday==320||jday==335||jday==350)

					readlai(line,laie,jday,&rcode,tmpyear);     //��ȡ��ЧҶ���ָ��
					
									
					if (rcode == ERROR){
						printf("����LAI���ݴ���"); 	  
						exit(0);
					} 
					}  //the end of year<2000

			else {
			if((jday-1)%8==0){ //�ж��Ƿ��LAI���ݣ�MODIS��LAI��8��һ��
					
				tmpyear=year;
				readlai(line,laie,jday,&rcode,tmpyear);     //��ȡ��ЧҶ���ָ��
						if (rcode == ERROR){
							printf("����LAI���ݴ���"); 	  
							exit(0);

						}
					}  //the end of (jday-1)%8==0
					
			         }  // the end of elso : year>=2000
				 
					}   //the end of control==4 || control==5

 
					else{  //if(control==1 || control==2 || control==3 || control==5)

						tmpyear=1980;    
					
						if( jday==1||jday==16||jday==32||jday==47||jday==60||jday==75||jday==91||jday==106||jday==121
							||jday==136||jday==152||jday==167||jday==182||jday==197||jday==213||jday==228||jday==244
							||jday==259||jday==274||jday==289||jday==305||jday==320||jday==335||jday==350)
		  
		  
		  {
                     								
				  readlai(line,laie,jday,&rcode,tmpyear);     //��ȡ��ЧҶ���ָ��
						if (rcode == ERROR){
							printf("����LAI���ݴ���"); 	  
							exit(0);
						} 
                    	}  // the end of  if(control==1 || control==2 || control==3|| control==5 )
//========================================================================================================================================================================================//
            
					}
					
			hipMalloc((void**) &gpulaie, npixels*sizeof(float));                              //22
			hipMemcpy(gpulaie,laie,npixels*sizeof(float),hipMemcpyHostToDevice);

			hipMalloc((void**) &gpulai, npixels*sizeof(float));                                 //19
			gpuLAI<<<BLOCK_NUM, THREAD_NUM,0>>>(gpulc,gpulaie,gpulai);
			hipMemcpy(lai,gpulai,npixels*sizeof(float),hipMemcpyDeviceToHost);
			hipFree(gpulaie);                                                                   //22
           
				
				hipMemcpy(gpusdat,ClimateD,npixels5*sizeof(short),hipMemcpyHostToDevice);
				hipMemcpy(gpujday,&jday,sizeof(long),hipMemcpyHostToDevice);
				hipMemcpy(gpuxx,xx,npixels*sizeof(struct xvalue),hipMemcpyHostToDevice);
 			
				
                hipMemcpy(gpunpp,npp,npixels*sizeof(float),hipMemcpyHostToDevice);
				hipMemcpy(gpuhr,hr,npixels*sizeof(float),hipMemcpyHostToDevice);
				hipMemcpy(gpunep,nep,npixels*sizeof(float),hipMemcpyHostToDevice);



				 if(jday==1){
	 
                hipMemcpy(gpusoilw1_old,soilw1_old,npixels*sizeof(float),hipMemcpyHostToDevice);
				hipMemcpy(gpusoilw2_old,soilw2_old,npixels*sizeof(float),hipMemcpyHostToDevice);
				hipMemcpy(gpusoilw3_old,soilw3_old,npixels*sizeof(float),hipMemcpyHostToDevice);		

				hipMemcpy(gpuCpoolold,Cpoolold,npixels*sizeof(struct carbonpool),hipMemcpyHostToDevice);
				
	            hipMemcpy(gpuCNRold,CNRold,npixels*sizeof(struct CNratio),hipMemcpyHostToDevice);

				  }

				 
				 
				 
				 
				 /*
	pix=1533;					
	printf("line=%d 1=%d 2=%d 3=%d 4=%d 5=%d \n",line,ClimateD[pix],ClimateD[pix+4950],ClimateD[pix+4950*2],ClimateD[pix+4950*3],ClimateD[pix+4950*4]);
    printf("line=%d 1=%f 2=%f 3=%f 4=%f 5=%f 6=%f\n",line,soilw1_old[pix],soilw2_old[pix],soilw1_old[pix],lai[pix],Ndep[pix],Ndep[pix]);   

 
	printf("year=%d,stem=%f,croot=%f,frrot=%f,leaf=%f\n",year,Cpoolold[pix].Cstem,Cpoolold[pix].Ccroot,Cpoolold[pix].Cfroot,Cpoolold[pix].Cleaf); 

	printf("cd=%f,smd=%f,ssd=%f,fmd=%f,fsd=%f,sm=%f,m=%f,s=%f,p=%f\n",Cpoolold[pix].Ccd,Cpoolold[pix].Csmd,Cpoolold[pix].Cssd,Cpoolold[pix].Cfmd,
		                                                              Cpoolold[pix].Cfsd,Cpoolold[pix].Csm,Cpoolold[pix].Cm,Cpoolold[pix].Cs,Cpoolold[pix].Cp); 
	
	
	printf("Nstem=%f,Ncroot=%f,Nfroot=%f,Nleaf=%f\n",CNRold[pix].CNstem,CNRold[pix].CNcroot,CNRold[pix].CNfroot, CNRold[pix].CNleaf); 
	
	
	
	printf("Ncd=%f,Nsmd=%f,Nssd=%f,Nfmd=%f,Nfsd=%f,Nsm=%f,Nm=%f,Ns=%f,Nt=%f,Nav=%f,npp=%f,hr=%f,nep=%f\n",CNRold[pix].CNcd,CNRold[pix].CNsmd,CNRold[pix].CNssd,CNRold[pix].CNfmd,
		                                           CNRold[pix].CNfsd,CNRold[pix].CNsm,CNRold[pix].CNm,CNRold[pix].CNs,CNRold[pix].Nt,CNRold[pix].Nav,npp[pix],hr[pix],nep[pix]); 

printf("\n");printf("\n");
*/

				gpuModel2<<<BLOCK_NUM, THREAD_NUM,0>>>(gpujday,gpupix_offset,gpulc,gpuCI,gpuTI,gpusdat,gpuNdep,gpuNdep0,gpulai,gpunpp,gpuxx,gpulat,
				gpusoilw1_old,gpusoilw2_old,gpusoilw3_old,gpuHY1,gpuHY2,gpuHY3,gpuco,gpulambd,gpulambd2,gpusoilw1_n,gpusoilw2_n,gpusoilw3_n,
				gpuCpoolold,gpuCNRold,gpuCpoolnew,gpuCNRnew,gpuhr,gpunep,gpuoutputnep);

//===================================================================================================================================================/
               // hipMemcpy(lambd,gpulambd,npixels*sizeof(float),hipMemcpyDeviceToHost);
			  	hipMemcpy(xx,gpuxx,npixels*sizeof(struct xvalue),hipMemcpyDeviceToHost);
     
				hipMemcpy(hr,gpuhr,npixels*sizeof(float),hipMemcpyDeviceToHost);
				hipMemcpy(nep,gpunep,npixels*sizeof(float),hipMemcpyDeviceToHost);
				hipMemcpy(npp,gpunpp,npixels*sizeof(float),hipMemcpyDeviceToHost);
				hipMemcpy(outputnep,gpuoutputnep,npixels3*sizeof( int short),hipMemcpyDeviceToHost);	


				hipMemcpy(gpusoilw1_old,gpusoilw1_n,npixels*sizeof(float),hipMemcpyDeviceToDevice);
				hipMemcpy(gpusoilw2_old,gpusoilw2_n,npixels*sizeof(float),hipMemcpyDeviceToDevice);
				hipMemcpy(gpusoilw3_old,gpusoilw3_n,npixels*sizeof(float),hipMemcpyDeviceToDevice);

				hipMemcpy(gpuCpoolold,gpuCpoolnew,npixels*sizeof(struct carbonpool),hipMemcpyDeviceToDevice);
				hipMemcpy(gpuCNRold,  gpuCNRnew,npixels*sizeof(struct CNratio),hipMemcpyDeviceToDevice);
			


//===============================================================================================================================================/
				  if(jday==365){

if(year<(startyear1+20)){
					  for(pix=0; pix<npixels; pix++) {

						  thr[pix]=hr[pix];
						  tnpp[pix]=npp[pix];

					  }	                            // end zfm����ʼֵ 
}
			    hipMemcpy(soilw1_old,gpusoilw1_old,npixels*sizeof(float),hipMemcpyDeviceToHost);
				hipMemcpy(soilw2_old,gpusoilw2_old,npixels*sizeof(float),hipMemcpyDeviceToHost);
		    	hipMemcpy(soilw3_old,gpusoilw3_old,npixels*sizeof(float),hipMemcpyDeviceToHost);	
					

				hipMemcpy(Cpoolold,gpuCpoolold,npixels*sizeof(struct carbonpool),hipMemcpyDeviceToHost);
				hipMemcpy(CNRold,  gpuCNRold,npixels*sizeof(struct CNratio),hipMemcpyDeviceToHost);

  }
			/*
    pix=903;
	fprintf(f8,"%d,%f,%f,%f,%f,",year,Cstem[pix],Ccroot[pix],Cfroot[pix], Cleaf[pix]); 
	fprintf(f8,"%f,%f,%f,%f,%f,%f,%f,%f,%f,", Ccd[pix],Csmd[pix],Cssd[pix],Cfmd[pix],Cfsd[pix],	Csm[pix],Cm[pix],Cs[pix],Cp[pix]); 
    fprintf(f8,"%f,%f,%f,%f,",CNstem[pix],CNcroot[pix],CNfroot[pix], CNleaf[pix]); 
	fprintf(f8,"%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f\n", CNcd[pix],CNsmd[pix],CNssd[pix],CNfmd[pix],CNfsd[pix],CNsm[pix],CNm[pix],CNs[pix],Nt[pix],Nav[pix],npp[pix],hr[pix],nep[pix]); 
 	 */
  
/*
    pix=903;
	printf("year=%d,stem=%f,croot=%f,frrot=%f,leaf=%f\n",year,Cstem[pix],Ccroot[pix],Cfroot[pix], Cleaf[pix]); 
	printf("cd=%f,smd=%f,ssd=%f,fmd=%f,fsd=%f,sm=%f,m=%f,s=%f,p=%f\n", Ccd[pix],Csmd[pix],Cssd[pix],Cfmd[pix],Cfsd[pix],	Csm[pix],Cm[pix],Cs[pix],Cp[pix]); 
    printf("Nstem=%f,Ncroot=%f,Nfroot=%f,Nleaf=%f\n",CNstem[pix],CNcroot[pix],CNfroot[pix], CNleaf[pix]); 
	printf("Ncd=%f,Nsmd=%f,Nssd=%f,Nfmd=%f,Nfsd=%f,Nsm=%f,Nm=%f,Ns=%f,Nt=%f,Nav=%f,npp=%f,hr=%f,nep=%f\n", CNcd[pix],CNsmd[pix],CNssd[pix],CNfmd[pix],CNfsd[pix],CNsm[pix],CNm[pix],CNs[pix],Nt[pix],Nav[pix],npp[pix],hr[pix],nep[pix]); 
 
  		




				pix=1533;					
				printf("line=%d 1=%d 2=%d 3=%d 4=%d 5=%d \n",line,ClimateD[pix],ClimateD[pix+4950],ClimateD[pix+4950*2],ClimateD[pix+4950*3],ClimateD[pix+4950*4]);
				printf("line=%d 1=%f 2=%f 3=%f 4=%f 5=%f 6=%f\n",line,soilw1_old[pix],soilw2_old[pix],soilw1_old[pix],lai[pix],Ndep[pix],Ndep[pix]);   


				printf("year=%d,stem=%f,croot=%f,frrot=%f,leaf=%f\n",year,Cpoolold[pix].Cstem,Cpoolold[pix].Ccroot,Cpoolold[pix].Cfroot,Cpoolold[pix].Cleaf); 

				printf("cd=%f,smd=%f,ssd=%f,fmd=%f,fsd=%f,sm=%f,m=%f,s=%f,p=%f\n",Cpoolold[pix].Ccd,Cpoolold[pix].Csmd,Cpoolold[pix].Cssd,Cpoolold[pix].Cfmd,
					Cpoolold[pix].Cfsd,Cpoolold[pix].Csm,Cpoolold[pix].Cm,Cpoolold[pix].Cs,Cpoolold[pix].Cp); 


				printf("Nstem=%f,Ncroot=%f,Nfroot=%f,Nleaf=%f\n",CNRold[pix].CNstem,CNRold[pix].CNcroot,CNRold[pix].CNfroot, CNRold[pix].CNleaf); 



				printf("Ncd=%f,Nsmd=%f,Nssd=%f,Nfmd=%f,Nfsd=%f,Nsm=%f,Nm=%f,Ns=%f,Nt=%f,Nav=%f,npp=%f,hr=%f,nep=%f\n",CNRold[pix].CNcd,CNRold[pix].CNsmd,CNRold[pix].CNssd,CNRold[pix].CNfmd,
					CNRold[pix].CNfsd,CNRold[pix].CNsm,CNRold[pix].CNm,CNRold[pix].CNs,CNRold[pix].Nt,CNRold[pix].Nav,npp[pix],hr[pix],nep[pix]); 

				printf("\n");printf("\n");


*/




//===============================================================================================================================================/
	
				
 			if(((jday%30)==1 && year>=startyear) || (jday==365 && year>=startyear1) ) {  
	
//===============================================================================================================================================/
				 
			//============================���NEP========================================       
					if(output_NPP=='y' || output_NPP=='Y') {       //����Ԫ���NeP         
		
					
					if(control==1)	sprintf(outfnpp_name,"h:\\P7_output_all_no\\gnep%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					if(control==2)	sprintf(outfnpp_name,"h:\\P7_output_climate\\gnep%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
                    if(control==3)	sprintf(outfnpp_name,"h:\\P7_output_CO2\\gnep%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j 
                    if(control==4)	sprintf(outfnpp_name,"h:\\P7_output_LAI\\gnep%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					if(control==5)  sprintf(outfnpp_name,"h:\\P7_output_Ndep\\gnep%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
                    if(control==6)  sprintf(outfnpp_name,"h:\\P7_output_all\\gnep%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
                    

					//	sprintf(outfnpp_name,".\\output\\gnep%d_%d.img",year,jday);    
						//sprintf(outfnpp_name,"cnep%d_%d.img",year,jday); 

					if(lin==0) {
						if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL)
						{
							printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							getchar();
							exit(0);
						}
						fwrite(&outputnep[0],sizeof( int short),npixels3,outfilenpp);

						//printf("1=%f   ", outputnep[0]);
						fclose(outfilenpp);
					
					}  //the end of lin==0
					
					else  {
						if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)
						{
							printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							getchar();
							exit(0);
						}
						fwrite(&outputnep[0],sizeof( int short),npixels3,outfilenpp);

						//printf("1=%f   ", outputnep[0]);
						fclose(outfilenpp);
					
					}
					
					}  //  //���NEP����  
//output Carbon pools
			}

 //=================================================��ʼ����C��N�ļ�============================================


					//if(year>=startyear && control==6&&(jday==365)){
                  
							if( control==6&&(jday==365)){	
						
						sprintf(outfnpp_name,"h:\\P7_output_all\\CN_leaf%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					
						if(lin==0) {
							if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
								printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							getchar();
							exit(0);
							}
						}		   
						else  			
						{
							if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
							printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							getchar();
							exit(0);
							}
							
							}
for(i=0;i<npixels;i++) value[i]=CNRold[i].CNleaf;
						fwrite(&value[0],sizeof(float),npixels,outfilenpp);
                       	fclose(outfilenpp);





						}


	   //      if(year>=startyear&& control==6 &&jday==365){    //2018-03-02


                     if(control==6 &&jday==365){    //2018-03-02
//============================================2==============================================================================
                      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_stem%d_%d.img",year,jday);     						
					  
					  
					  
					  if(lin==0) {
						  if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }
					  }		   
					  else  			
					  {
						  if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }

							}

					  for(i=0;i<npixels;i++) value[i]=CNRold[i].CNstem;
					  fwrite(&value[0],sizeof(float),npixels,outfilenpp);
					  fclose(outfilenpp);


//============================================3==============================================================================
                      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_Ccroot%d_%d.img",year,jday);    
					 
					  if(lin==0) {
						  if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }
					  }		   
					  else  			
					  {
						  if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }

							}


					  for(i=0;i<npixels;i++) value[i]=CNRold[i].CNcroot;
					  fwrite(&value[0],sizeof(float),npixels,outfilenpp);
					  fclose(outfilenpp);

				   
	
//============================================4==============================================================================
                      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_froot%d_%d.img",year,jday);    
					
					  if(lin==0) {
						  if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }
					  }		   
					  else  			
					  {
						  if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }

							}

					  for(i=0;i<npixels;i++) value[i]=CNRold[i].CNfroot;
					  fwrite(&value[0],sizeof(float),npixels,outfilenpp);
					  fclose(outfilenpp);

			
	
//============================================5==============================================================================
                     sprintf(outfnpp_name,"h:\\P7_output_all\\CN_Cd%d_%d.img",year,jday);       
					 
					 
					 if(lin==0) {
						 if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
							 printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							 getchar();
							 exit(0);
							}
					 }		   
					 else  			
						{
							if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
								printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
								getchar();
								exit(0);
							}

					 }


					 for(i=0;i<npixels;i++) value[i]=CNRold[i].CNcd;
					 fwrite(&value[0],sizeof(float),npixels,outfilenpp);
					 fclose(outfilenpp);
						
//============================================6=============================================================================
                  sprintf(outfnpp_name,"h:\\P7_output_all\\CN_smd%d_%d.img",year,jday);     
				 
				  if(lin==0) {
					  if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
						  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
						  getchar();
						  exit(0);
							}
				  }		   
				  else  			
						{
							if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
								printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
								getchar();
								exit(0);
							}

				  }

				  for(i=0;i<npixels;i++) value[i]=CNRold[i].CNsmd;
				  fwrite(&value[0],sizeof(float),npixels,outfilenpp);
				  fclose(outfilenpp);


//============================================7==============================================================================
                     sprintf(outfnpp_name,"h:\\P7_output_all\\CN_ssd%d_%d.img",year,jday);      
					 if(lin==0) {
						 if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
							 printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							 getchar();
							 exit(0);
							}
					 }		   
					 else  			
						{
							if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
								printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
								getchar();
								exit(0);
							}

					 }
					 for(i=0;i<npixels;i++) value[i]=CNRold[i].CNssd;
					 fwrite(&value[0],sizeof(float),npixels,outfilenpp);
					 fclose(outfilenpp);


						
//============================================8==============================================================================
                   sprintf(outfnpp_name,"h:\\P7_output_all\\CN_fmd%d_%d.img",year,jday);      
				   if(lin==0) {
					   if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
						   printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
						   getchar();
						   exit(0);
					   }
				   }		   
				   else  			
				   {
					   if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
						   printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
						   getchar();
						   exit(0);
					   }

							}

				   for(i=0;i<npixels;i++) value[i]=CNRold[i].CNfmd;
				   fwrite(&value[0],sizeof(float),npixels,outfilenpp);
				   fclose(outfilenpp);


					
//============================================9==============================================================================
                sprintf(outfnpp_name,"h:\\P7_output_all\\CN_fsd%d_%d.img",year,jday);      
				if(lin==0) {
					if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
						printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
						getchar();
						exit(0);
					}
				}		   
				else  			
				{
					if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
						printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
						getchar();
						exit(0);
					}

				}
				for(i=0;i<npixels;i++) value[i]=CNRold[i].CNfsd;
				fwrite(&value[0],sizeof(float),npixels,outfilenpp);
				fclose(outfilenpp);


				//============================================10=============================================================================
                sprintf(outfnpp_name,"h:\\P7_output_all\\CN_sm%d_%d.img",year,jday);      
				if(lin==0) {
					if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
						printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
						getchar();
						exit(0);
					}
				}		   
				else  			
				{
					if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
						printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
						getchar();
						exit(0);
					}

				}

				for(i=0;i<npixels;i++) value[i]=CNRold[i].CNsm;
				fwrite(&value[0],sizeof(float),npixels,outfilenpp);
				fclose(outfilenpp);

//============================================11============================================================================
            sprintf(outfnpp_name,"h:\\P7_output_all\\CN_m%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
			if(lin==0) {
				if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
					printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
					getchar();
					exit(0);
				}
			}		   
			else  			
			{
				if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
					printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
					getchar();
					exit(0);
				}

			}
			for(i=0;i<npixels;i++) value[i]=CNRold[i].CNm;
			fwrite(&value[0],sizeof(float),npixels,outfilenpp);
			fclose(outfilenpp);
			
		
//============================================12==============================================================================
                      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_s%d_%d.img",year,jday);  					
					  if(lin==0) {
						  if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }
					  }		   
					  else  			
					  {
						  if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }

							}
					  for(i=0;i<npixels;i++) value[i]=CNRold[i].CNs;
					  fwrite(&value[0],sizeof(float),npixels,outfilenpp);
					  fclose(outfilenpp);

//============================================13==============================================================================
                      sprintf(outfnpp_name,"h:\\P7_output_all\\CN_p%d_%d.img",year,jday);     
					  if(lin==0) {
						  if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }
					  }		   
					  else  			
					  {
						  if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }

							}
					  for(i=0;i<npixels;i++) value[i]=CNRold[i].CNp;
					  fwrite(&value[0],sizeof(float),npixels,outfilenpp);
					  fclose(outfilenpp);
					
//============================================14==============================================================================
                      sprintf(outfnpp_name,"h:\\P7_output_all\\Nav_%d_%d.img",year,jday);     
					  if(lin==0) {
						  if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }
					  }		   
					  else  			
					  {
						  if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
							  printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							  getchar();
							  exit(0);
						  }

							}

					  for(i=0;i<npixels;i++) value[i]=CNRold[i].Nav;
					  fwrite(&value[0],sizeof(float),npixels,outfilenpp);
					  fclose(outfilenpp);

					
	//============================================15==============================================================================JUW
                      sprintf(outfnpp_name,"h:\\P7_output_all\\SW1_%d_%d.img",year,jday);     
                       if(lin==0) {
						if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL)
						{
							printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							getchar();
							exit(0);
						}
                    fwrite(&soilw1_old[0],sizeof(float),npixels,outfilenpp);		
					
						
					  //  fwrite(&Nt[0],sizeof(float),npixels,outfilenpp);		
						
						fclose(outfilenpp);
					
					}  //the end of lin==0
					
					else  {
						if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)
						{
							printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							getchar();
							exit(0);
						}
                      fwrite(&soilw1_old[0],sizeof(float),npixels,outfilenpp);		
                      fclose(outfilenpp);
					}	

//============================================16==============================================================================
                      sprintf(outfnpp_name,"h:\\P7_output_all\\SW2_%d_%d.img",year,jday);     
                       if(lin==0) {
						if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL)
						{
							printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							getchar();
							exit(0);
						}
                    fwrite(&soilw2_old[0],sizeof(float),npixels,outfilenpp);		
					fclose(outfilenpp);
					
					}  //the end of lin==0
					
					else  {
						if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)
						{
							printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							getchar();
							exit(0);
						}
                      fwrite(&soilw2_old[0],sizeof(float),npixels,outfilenpp);		
                      fclose(outfilenpp);
					}	


//============================================17==============================================================================
                      sprintf(outfnpp_name,"h:\\P7_output_all\\SW3_%d_%d.img",year,jday);     
                       if(lin==0) {
						if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL)
						{
							printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							getchar();
							exit(0);
						}
                    fwrite(&soilw3_old[0],sizeof(float),npixels,outfilenpp);		
					fclose(outfilenpp);
					
					}  //the end of lin==0
					
					else  {
						if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)
						{
							printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
							getchar();
							exit(0);
						}
                      fwrite(&soilw3_old[0],sizeof(float),npixels,outfilenpp);		
                      fclose(outfilenpp);
					}	
                     }  //the end  if(control==6 && year==(startyear2-1))

//=======================================����C��N�ļ�����=============================================

					 
//===========================================��ʼ����C���ļ�======================================================					 
					 if(control==6 &&jday==365){
//================================================================1================================================
                   sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_leaf%d_%d.img",year,jday);      
					
				   if(lin==0) {
					   if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
						   printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
						   getchar();
						   exit(0);
					   }
				   }		   
				   else  			
				   {
					   if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
						   printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
						   getchar();
						   exit(0);
					   }

							}
				   for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Cleaf;
				   fwrite(&value[0],sizeof(float),npixels,outfilenpp);
				   fclose(outfilenpp);

					
//============================================2==============================================================================

sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_stem%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					
if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}	 
                 
for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Cstem;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);

//============================================3==============================================================================

sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_Ccroot%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					
if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}

for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Ccroot;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);

//============================================4==============================================================================

sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_froot%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					
if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}
		
for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Cfroot;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);


//============================================5==============================================================================

sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_Cd%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					
if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}

for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Ccd;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);


//============================================6=============================================================================

sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_smd%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					
if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}
for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Csmd;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);					


//============================================7==============================================================================

sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_ssd%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					
if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}
	

for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Cssd;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);

//============================================8==============================================================================
sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_fmd%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j


if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}						
     
for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Cfmd;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);


//============================================9==============================================================================

sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_fsd%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					
if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}

for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Cfsd;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);


//============================================10=============================================================================

sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_sm%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					
if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}

for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Csm;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);

//============================================11============================================================================

sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_m%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}

for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Cm;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);

//============================================12==============================================================================

sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_s%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}

for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Cs;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);


//============================================13==============================================================================

sprintf(outfnpp_name,"h:\\P7_output_all\\Carbon_p%d_%d.img",year,jday);      ///////sprintf ����"cnpp90_%d.img"+j
					
if(lin==0) {
	if ((outfilenpp=fopen(outfnpp_name, "wb"))== NULL){	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}
}		   
else  			
{
	if ((outfilenpp=fopen(outfnpp_name, "ab"))== NULL)	{	
		printf("\n Unable to open file <%s>,  exitting program ...\n\n", outf_name);
		getchar();
		exit(0);
	}

}

for(i=0;i<npixels;i++) value[i]=Cpoolold[i].Cp;
fwrite(&value[0],sizeof(float),npixels,outfilenpp);
fclose(outfilenpp);


//=====================================================================================================================/


}  //  //���carbon pools����  


					j=j++;
 


				
				
				hipFree(gpulai);                              //19



	}    //=====================================================dayѭ������=======================
			
     
			hipFree(gpuco);  //17
            hipFree(gpuNdep);  //57
            hipFree(gpuNdep0);  //56


if(year<(startyear1+20)){

for(pix=0;pix<npixels;pix++){
if(lc[pix]!=0&& lc[pix]!=21&& lc[pix]!=19 ) {

	
	coef_c(pix,lc[pix],coef,HY2);
ratio=tnpp[pix]/thr[pix];
/*
if(pix==3666){
printf("tnpp=%f  hr=%f %f\n",tnpp[pix],thr[pix],ratio);

printf("%f  %f   %f   %f   %f   %f   %f   %f   %f   %f   %f   %f %f\n",

Cpoolold[pix].Cstem,Cpoolold[pix].Ccroot,Cpoolold[pix].Cfroot,Cpoolold[pix].Cleaf,Cpoolold[pix].Ccd,Cpoolold[pix].Csmd,Cpoolold[pix].Cssd,

Cpoolold[pix].Cfmd,Cpoolold[pix].Cfsd,Cpoolold[pix].Csm,Cpoolold[pix].Cm,Cpoolold[pix].Cs,Cpoolold[pix].Cp);

}
*/

/*
Cpoolold[pix].Cstem=Cpoolold[pix].Cstem*ratio;
Cpoolold[pix].Ccroot=Cpoolold[pix].Ccroot*ratio;

Cpoolold[pix].Cfroot=Cpoolold[pix].Cfroot*ratio;
Cpoolold[pix].Cleaf=Cpoolold[pix].Cleaf*ratio;
*/

Cpoolold[pix].Ccd=Cpoolold[pix].Ccd*ratio;

Cpoolold[pix].Csmd=Cpoolold[pix].Csmd*ratio;

Cpoolold[pix].Cssd=Cpoolold[pix].Cssd*ratio;

Cpoolold[pix].Cfmd=Cpoolold[pix].Cfmd*ratio;

Cpoolold[pix].Cfsd=Cpoolold[pix].Cfsd*ratio;

Cpoolold[pix].Csm=Cpoolold[pix].Csm*ratio;

Cpoolold[pix].Cm=Cpoolold[pix].Cm*ratio;

Cpoolold[pix].Cs=Cpoolold[pix].Cs*ratio;

Cpoolold[pix].Cp=Cpoolold[pix].Cp*ratio;

/*
if(pix==3666){
	printf("tnpp=%f  hr=%f %f\n",tnpp[pix],thr[pix],ratio);

	printf("%f  %f   %f   %f   %f   %f   %f   %f   %f   %f   %f   %f %f\n",

		Cpoolold[pix].Cstem,Cpoolold[pix].Ccroot,Cpoolold[pix].Cfroot,Cpoolold[pix].Cleaf,Cpoolold[pix].Ccd,Cpoolold[pix].Csmd,Cpoolold[pix].Cssd,

		Cpoolold[pix].Cfmd,Cpoolold[pix].Cfsd,Cpoolold[pix].Csm,Cpoolold[pix].Cm,Cpoolold[pix].Cs,Cpoolold[pix].Cp);


pix=pix;


}

*/


}
}// pix loop
}// year<(startyear1+20


if(year<(startyear1+10)){

	for(pix=0;pix<npixels;pix++){
	if(lc[pix]!=0&& lc[pix]!=21&& lc[pix]!=19 ) {
	coef_c(pix,lc[pix],coef,HY2);

   CNRold[pix].CNcd  =coef[35];     //Cd  ��̼����
	CNRold[pix].CNssd =coef[36];     //ssd ��̼����
	CNRold[pix].CNsmd =coef[37];     //smd ��̼���� 
	CNRold[pix].CNfsd =coef[38];     //fsd ��̼����
	CNRold[pix].CNfmd  =coef[39] ;    //fmd ��̼����
	CNRold[pix].CNsm  =coef[41];     //sm  ��̼����
    CNRold[pix].CNm    =coef[42];     //sm  ��̼����
   CNRold[pix]. CNs    =coef[40];     //slow��̼����
    CNRold[pix].CNp    =coef[43];     
    CNRold[pix].CNstem =coef[47];    //Setm C:NinitialValues[25] ;
	CNRold[pix].CNcroot=coef[47];   //Setm C:NinitialValues[25] ;
	CNRold[pix].CNleaf  =coef[46] ;   //ҶC:N
	CNRold[pix].CNfroot =coef[46];   //ҶC:N 
}
}
} // year<(startyear1+10

//JUW1


}//====================================��ѭ������======================================================

          
           

            hipFree(gpunpp);   //15
			hipFree(gpuhr);    //23
			hipFree(gpunep);   //24
			hipFree(gpuxx);         //21




            hipFree(gpuoutputnep);  //25
            hipFree(gpujday);  //13
			hipFree(gpusdat);  //12
			


			hipFree(gpuCpoolold);
hipFree(gpuCpoolnew);
				

hipFree(gpuCNRold);
hipFree(gpuCNRnew);
				hipFree(gpusoilw1_n);//7
				hipFree(gpusoilw2_n); //8
				hipFree(gpusoilw3_n);  //9

  	           hipFree(gpuHY1);   //2
		       hipFree(gpuHY2);   //3
		       hipFree(gpuHY3);   //4
		       hipFree(gpulat);  //5
		       hipFree(gpulc);   //6
		       hipFree(gpusoilw1_old);//7
		       hipFree(gpusoilw2_old); //8
    	       hipFree(gpusoilw3_old);  //9
	   
 hipFree(gpuCI);   //10
hipFree(gpuTI);
		        hipFree(gpulambd);   //55
  hipFree(gpulambd2);   //55



	printf("time used: %d\n", clock()-ttt);
	free(LC);       printf ("1");  free(CI);   free(TI);  
	free(lc);       printf ("2");  
	free(awc);		printf ("3");
	free(lai);	    printf ("4");
	free(lon);	    printf ("5");
	free(lat);	    printf ("6");
	free(npp);	    printf ("7");
	
	free(output);	printf ("8");
	free(outputnep); printf ("9"); 
	free(ClimateD);  //10
 
     
     free(laie);     //12
	 free(nppold);   //13
	 free(nep);      //14
	 free(hr);       //15
 	 free(tnpp);     //16
    free(thr);     //16

	 free(lambd);    //17
 free(lambd2);    //17
	
	free(soilw1_old);//46  
	free(soilw2_old);   //47
	free(soilw3_old);   //48


	free(soilw1_n);//46  
	free(soilw2_n);   //47
	free(soilw3_n);   //48


free(Cpoolold);
free(Cpoolnew);

free(CNRold);
free(CNRnew);
free(value);


	free(HY1);  //49
	free(HY2); //50
	free(HY3); //51


 	free(x) ; //52
 	free(xx);  //53

    free(Ndep0) ; //54
 	free(Ndep);  //55

    //fclose(f8);
//exit(0);

 }  // ==================================================================��ѭ������ zfm4.22==============================================================
hipFree(gpupix_offset);  //1

} // the end of control loop

return 1;

}	/* end of main */





