#include "hip/hip_runtime.h"
/*************************************************************************
  Program:     zcomp.c
  --------
 	 Description:
	 ------------ 
	 Calculate z variables.
	 
	 Details:
	 -------

	Z 	Description
	-------------------------------------------------------------

    2	day of year  
   	3	precipitation 				(m)  
	4 	max. Air temperature 		(deg c) 	
 	5  	min. Air temperature 		(deg c)    
   	6  	relative humidity			(%)  
   	7  	soil temperature, ave 24 hr temp   	(deg c)  
   	8  	daily total incoming solar radiation  (kj/m2/d)
   	9  	average incoming solar radiation	(W/m2)		
	10	LAI

    14 	daylight average air temperature 	(deg c)  
    15 	average night min. temperature  	(deg c)  
    16 	vapor pressure deficit        		(mb)  
    17 	absolute humidity deficit   		(mic. gm/m3)  
   	18 	running periods (daylength for daily model)  (sec)  
   	19 	daily total absorbed radiation    	(kj/m2/d)
	20	daily averaged absorbed radiation  	(W/m2)
       		
	22	available water capacity 	 	(m)
	23	land cover type

	26	leaf nitrogen concentration  fraction	

	30	Cos(Theta_noon)
	31	Cos(Theta_mean)
	32	sunlit LAI 
	33	shaded LAI
	34	daily mean radiation over sunlit leaves (W/m2)
	35	daily mean radiation over shaded leaves (W/m2)
	36	radiation over sunlit leaves at noon (W/m2)
	37	radiation over shaded leaves at noon (W/m2)
	38	net radiation for sunlit leaves (W/m2)
	39	net radiation for shaded leaves (W/m2)
	40	net radiation for understory (W/m2)
	41	net radiation for ground (W/m2)
	42  understory LAI

***************************************************************************
  CCRS (EMS/Applications Division)
  Written by:   J. Liu      
  Modified by: X.F. Feng
  Last update:  July 2003
*****************************************************************************/


//#include"beps.h"

#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 


   __device__ void gpuzcomp(
    long jday,
    int long pix,
    float lat_p,
    short int  lc_p,float CI,float TI,
    float awc_p,
    float lai_p,
    float b[],float   x[],float z[],
    short climatedata[],float *tmean)
    { 
    float   xtmax, xtmin, xrad, xppt, tave,  h;
	float   etemp1, etemp2, esd, es, ampl, day, xd; 

    float gfn;			       /* gap function at noon */
    float omega;		       /* clumping index */
    float theta_m;		       /* solar zenith angle at noon */
    float alfa;
    float ssun;		       /* radiation for sunlit leaves */
    float sshade;		       /* radiation for shaded leaves */  
    
	// for shortwave radiation
	float lai_u;			   // understory lai
	float n_ssun;			   // net shortwave radiation for sun lit leaves
	float n_sshade;		   // net shortwave radiation for sun lit leaves
	float n_sunder;	       // net shortwave radiation for understory
	float n_sground;		   // net shortwave radiation for gound

	// for longwave radiation
	float m_lnet_o;		   // mean longwave radiation for overstory
	float m_lnet_u;		   // mean longwave radiation for understory
	float lnet_g;			   // net longwave radiation for ground
    int long pixx;
    
    int count;



    count=pix/4950;
    pixx=pix+4950*count*4;

    xtmax =climatedata[3*4950+pixx]*0.1-273.15;   ///CRU ���ϵ��¶ȵ�λ��K, �������   
	xtmin =climatedata[4*4950+pixx]*0.1-273.15;   ///CRU ���ϵ��¶ȵ�λ��K���������   
	h	  =climatedata[2*4950+pixx]*0.1;            //  relative humidity(%)
	xrad  =climatedata[pixx];//*0.1// MJ/d
	xppt  =climatedata[4950+pixx]*0.1;           // the unit of the original input data is 0.1 mm.

     z[45]=TI;      //ÿ��pixel �Ķ���ƽ���¶ȣ�



   if(h<15)  h=15;
   if(h>100) h=100;

   if(xtmax<-50)  xtmax=-50;
   if(xtmax>45)   xtmax=45;

   if(xtmin<-50)  xtmin=-50;
   if(xtmin>45)   xtmin=45;

   if(xppt<0) xppt=0;
   if(xppt>1500) xppt=1500;

   if(xrad<0.5) xrad=0.5;
   if(xrad>600) xrad=600;



	z[2] = jday;
     
/* convert ppt. 0.1mm to meters */       
    if ( xppt < 0 )       z[3] = 0;
	else                  z[3] = xppt / 1000.0 ;   ///zfm preci unit is mm , convert to m
   
	z[4] = xtmax;     
    z[5] = xtmin;    


   *tmean=(xtmax+xtmin)/2.0;

    tave = (z[4]+z[5])/2.0; 
 	
    if (jday <= 1) 
	z[1] = b[22];

    //z[1] = __max((z[1]+tave), b[22]);
   if((z[1]+tave)>b[22]) z[1]=z[1]+tave;
   else                  z[1]=b[22];    

/* average temperatures, soil, air */ 
    
    z[7]  = tave;

/* if snowpack exists, soil temp defined as 0 deg */ 

    if (x[1] > 0.0) 	
	z[7] = 0.0;

//================================================================================LAI 
  
	omega=CI/100.0; 
	if(omega<0.4) omega=0.4;

	z[10]  =lai_p;

/* daylight mean temp. z[14], nighttime mean temp. z[15] */
    z[14]  =  0.212 * (z[4] - tave) + tave;   //�����¶�
   //  z[15]  = (z[14]+2*z[5])/3.0;   //ҹ��ƽ������     2017-11-29ǰ�� ҹ������ƫ��
 
z[15]  = (z[14]+z[5])/2.0;
	
	//etemp1 = 16.78*tave-116.9;
	//etemp2 = tave + 237.3; 

	//2017-11-23���������ں͹����Ҫ�ڰ�����У������ð����¶ȼ���VPD
	etemp1 = 16.78*z[14]-116.9;
	//etemp2 = tave + 237.3; 

etemp2 = z[14]+ 237.3; 


	esd = exp (etemp1/etemp2);
    es = esd * h *0.01 ;


//	z[16]= (100-h)*0.01*esd;                 // in kpa
//   z[16]= (100-h)*0.1*esd;                  // in mbar


z[16] =(esd - es)*10.0;// __max(((esd - es) * 10),0.0) ;                 // in mbar
 if(z[16]<0) z[16]=0;
      
z[46]=h*0.01;



/* compute daylength in seconds */ 
      if(lat_p<0)  xd  =  (float) jday +105;
else               xd  =  (float) jday -79.0;
   //2017-11-23��ǰ��ȫ��  xd  =  (float) jday -79.0;   �÷��̽������ñ�����

if (xd < 0.0)  	xd = 286.0 + (float) jday;

    ampl = exp(7.42+0.045* abs(lat_p))/3600.0; 
  
	day  = ampl * (sin(xd*0.01721)) + 12.0; 
    z[18]  =  day * 3600.0;// * 0.95;     //running period   ??????????????????????????????????//20171128:0.95

/* total incoming solar radiation in kj/m2/s */ 
  // z[8]  = xrad *24.0*3600*0.85/1000;        ///zfm ע�ⵥλ�� ����������̫������    NCAR ������һ��24Сʱ��ֵ��0.85��ϵ�������ݷ�ȷ���������վ�����ϱȣ� NCAR ���������һ��ϵ�� 
      
	
	z[8]  = xrad*100000;
/* average incoming solar radiation in watt/m2 */ 

    z[9] = z[8]/z[18];  ///zfm ע�ⵥλ   // ??????????????????????????????????
  // z[9] = z[8];    ///zfm rad unit is w/m2


/* FPAR */ 
   theta_m=fabs((lat_p-23.5*sin((jday-81)*2*PI/365.0))*PI/180.0);    //����̫���춥��   //2017-10-26��ȡ����ֵ����
   
   if (fabs(PI/2-theta_m)<0.01)	alfa=0.05;
 
	else alfa=(cos(theta_m)-(PI/2-theta_m)*sin(theta_m))/((PI/2-theta_m)*(1-sin(theta_m)));
 

	/*
	switch(lc_p)
    {

	case 6 : case 9://mixe =0.6, for MODIS 
		omega=0.65;           //0.6   //2013-0828��0.7��Ϊ0.65 
		break;

	case 1: case 2: case 3:case 7: case 8:    //broadleaf forest =0.7, for MODIS 
		omega=0.75;      //0.7       //2013-0828��0.8��Ϊ0.75    
		break;

	case 4: case 5: case 10: //coni=0.5, for MODIS 
		omega=0.55;   //0.55  7-21         //2013-0828��0.6��Ϊ0.55  
		break;

    case 11:  case 12: case 14:     //shrub=0.5, for MODIS, shrub same as conif, closed shrub and woody savanna
		omega=0.5;
		break;

	case 13: case 16: case 17: case 18: 	// grass=0.9, for MODIS, opened shrub & savanna & grassland
		omega=0.9;
		break;
	
	default:
		omega=0.8;


    }

*/


	if (fabs(PI/2-theta_m)<0.01)	gfn=0.1;
	else
	   gfn = exp(-0.4*omega*z[10]/cos(theta_m));    //gap function at noon

	z[19]=(0.95-0.94*alfa*gfn);         //__max(0.0,(0.95-0.94*alfa*gfn));   
	if(z[19]<0) z[19]=0;
	
	
	z[19]=z[8]*z[19];           //ÿ��overstorey������̫������
    

/*  daily averaged absorbed radiation in W/m2 */
    z[20]=z[19]/z[18];      //��λs����̫�����伴w/m2/s
	//z[20]=z[19];//6.10

/* leaf nitrogen */ 
    z[21] = b[26];        //Ҷ��

/* awc */
    z[22]=awc_p;              //awc
	
	//z[22]=__max(0.1,z[22]);     z[22]=__min(0.6,z[22]);
if(z[22]<0.1) z[22]=0.1;
if(z[22]>0.6) z[22]=0.6;


/* land cover */
    z[23]=(float)lc_p;            // land cover

/* cos(Thita_m)  */

  /* cos(Thita_m)  */
    z[30]=cos(theta_m);
    if(z[30]<0.01) z[30]=0.01;
/* cos(Thita_avg) */
    z[31]=cos(PI/8+3*theta_m/4);    ////??????????
   if(z[30]<0.01) z[30]=0.01;

/* average sunlit leaves */
	if (z[31]<0.01)
		z[32]=0;
	else
		z[32]=2*z[31]*(1-exp(-0.5*omega*lai_p/z[31]));   //��Ҷlai  z[31]: ƽ��̫���춨������

/* average shaded leaves */
    z[33]=lai_p-z[32];                    //��Ҷlai

/* daily averaged radiation for sunlit and shaded leaves */
    gpurad_ssl(z[9],z[31],z[10],lc_p,omega,&ssun,&sshade);     //z[10]:��LAI ��Z[31]��ƽ��̫���춨������
	z[34]=ssun;      //�վ���Ҷ̫������
    z[35]=sshade;    //�վ���Ҷ̫������

// printf ("z[9]=%f z[31]=%f z[10]=%f omega=%f &ssun=%f &sshade=%f\n", z[9], z[31], z[10], omega, &ssun, &sshade);

/*  radiation for sunlit and shaded leaves at noon*/
    gpurad_ssl(z[9]*PI/2,z[30],z[10],lc_p,omega,&ssun,&sshade); 
	z[36]=ssun;            //������Ҷ��̫������
    z[37]=sshade;          //������Ҷ��̫������  ??????????????����Ҷ�����䣨���棩�к�����
	

// understory lai
	switch(lc_p)
	{

/************** for LCTM, July*   //���²�lai**********/
	case 1:  case 3:            // conifer  //for umd ,zhoulei 4.1
		lai_u=1.175*exp(-0.991*z[10]);            //���²�lai
       	break;	

	case 2:  case 4:         //broad leaf forest  //EBF & DBF ;zhoulei 4.1
		lai_u=1.5;
       	break;

	case 5:                             // mixed between conifer and broadleaf //mixed forest;zhoulei 4.1
		lai_u=0.5*(1.5+1.175*exp(-0.991*z[10]));
		break;	

	default:
		lai_u=0.0;

  	}
lai_u=0.05;

	// net shortwave radiation calculation
	gpunet_shortwave(z[9],z[31],z[10],lc_p,omega,&ssun,&sshade,lai_u,&n_ssun,&n_sshade,&n_sunder,&n_sground);

	// net longwave radiation calculation

	// z[14]: daytime average temperature
	gpunet_longwave(lai_p,lai_u,omega,es,(z[14]+275.0),&m_lnet_o,&m_lnet_u,&lnet_g);
	z[38]=n_ssun+m_lnet_o;           //��Ҷ̫��������
	//net shortwave radiation for sun lit leaves + mean longwave radiation for overstory 
	z[39]=n_sshade+m_lnet_o;         //��Ҷ������
	//net shortwave radiation for shaded lit leaves + mean longwave radiation for overstory 
	z[40]=n_sunder + m_lnet_u;        //���²�ľ�����
	//net shortwave radiation for understory +  mean longwave radiation for understory
	z[41]=n_sground + lnet_g;           //����ľ�����
	//net shortwave radiation for gound + net longwave radiation for ground
	z[42]=lai_u;                     //���²�Ҷ���ָ��
	

    return;
}

__device__ int gpurad_ssl(float sg, float cos_theta,  float lai_p, int short lc_p,float omega,float *ssun,float *sshade)
{		   
    float theta_avg; 	 	 // Mean cos(thita) 
    float s0;			     // solar constant (=1367 W m-2)
    float rr;			     // ratio of sdif_over to sg
    float sdir;		         // direct radiation W m-2 */
    float sdif_over;		 // diffusive radiation over plant canopy 
    float sdif_under;		 // diffusive radiation under plant canopy
    float c;			     // radiation from multiple scattering

/************* calcuate sdir and sdif_over ********************/

    s0=1367;

    if (cos_theta<0.01)
	{
		sdif_over=0;
		sdir=0;
	}
	else
	{
		rr=sg/(s0*cos_theta);                 // �ж�ָ��
		
                                                   //ֱ��
/*
		if(lc_p==1 || lc_p==7 || lc_p==8){
     	sdif_over=sg*(0.7327+3.8453*rr-16.31*pow(rr,2)+18.96*pow(rr,3)-7.0802*pow(rr,4)); 	//ɢ��	
        if(sdif_over>sg) sdif_over=sg;	
		sdir=sg-sdif_over;                                                     //ֱ��
		}

		else{
		if (rr>0.8) sdif_over=0.13*sg;
		else
			sdif_over=sg*(0.943+0.732*rr-4.9*pow(rr,2)+1.796*pow(rr,3)+2.058*pow(rr,4)); 	//ɢ��	
		sdir=sg-sdif_over;  

		}
		*/
		if (rr>0.75) sdif_over=0.15*sg;      //2018-01-26�ģ�if (rr>0.8) sdif_over=0.13*sg;  
		else
			sdif_over=sg*(0.943+0.732*rr-4.9*pow(rr,2)+1.796*pow(rr,3)+2.058*pow(rr,4)); 	//ɢ��	
		sdir=sg-sdif_over;  


		//sdif_over=sg*1.0/(1+exp(-3.98+8.05*rr)); 	//ɢ��	
		//if(sdif_over>sg) sdif_over=sg;	
		//sdir=sg-sdif_over;                                                     //ֱ��

	}
		
/************* calculate ssun and sshade ***********************/

/* radiation from multiple scattering */
	//c=0.07*omega*sdir*(1.1-0.1*lai_p)*exp(-cos_theta);     //���ɢ��    //2013-08-27��  
	c=0.07*omega*sdir*(1.1-0.1*lai_p)*exp(-cos_theta);     //���ɢ��    //2013-08-27��  

	
	
	
	/* sdif_under */
    theta_avg=0.537+0.025*lai_p;
    sdif_under=sdif_over*exp(-0.5*omega*lai_p/theta_avg);    //���²�ɢ��
	
/* radiation for shaded leaves, X.F., Sep 2003 */
   // if ((lai_p<0.01) && (sdif_over-sdif_under)<0)
    if ((lai_p<0.01) && (sdif_over-sdif_under)<=0)      //lai_p��Ҷ���
		*sshade=0;
	else	
		*sshade=(sdif_over-sdif_under)/lai_p+c;

/* radiation for sunlit leaves */
    if (cos_theta <0.01)
		*ssun=0;
	else
		*ssun=0.5*sdir/cos_theta +*sshade;         //��1999����



    return 1;             //???????????????????????????????????
}

__device__ int gpunet_shortwave(float sg,float cos_theta,float lai_p, int short lc_p,
								float omega,float *ssun,float *sshade,float lai_u, float *n_ssun,float *n_sshade,float *n_sunder,float *n_sground)		 

{		   
    float theta_avg; 		 // Mean cos(theta) 
    float s0;			     // solar constant (=1367 W m-2)
    float rr;			     //ratio of sdif_over to sg 
    float sdir;		         // direct radiation W m-2
    float sdif_over;		 // diffusive radiation over plant canopy
    float sdif_under;		 // diffusive radiation under plant canopy
    float c;			     // radiation from multiple scattering
	float c_for_net;
	float alpha_l=0.25;
	float alpha_g=0.2;
	float theta_avg_under;  // Mean cos(theta) for understory  

//************* calcuate sdir and sdif_over ********************

    s0=1367;

    if (cos_theta<0.01)
	{
		sdif_over=0;
		sdir=0;
	}
	else
	{
		rr=sg/(s0*cos_theta);
/*
		if(lc_p==1 || lc_p==7 || lc_p==8){
			sdif_over=sg*(0.7327+3.8453*rr-16.31*pow(rr,2)+18.96*pow(rr,3)-7.0802*pow(rr,4)); 	//ɢ��	
			if(sdif_over>sg) sdif_over=sg;	
			sdir=sg-sdif_over;                                                     //ֱ��
		}

		else{
			if (rr>0.8) sdif_over=0.13*sg;
			else
				sdif_over=sg*(0.943+0.732*rr-4.9*pow(rr,2)+1.796*pow(rr,3)+2.058*pow(rr,4)); 	//ɢ��	
			sdir=sg-sdif_over;  
		}
		*/

		if (rr>0.75) sdif_over=0.15*sg;     //2018-01-26�ģ�if (rr>0.8) sdif_over=0.13*sg; 
		else
			sdif_over=sg*(0.943+0.732*rr-4.9*pow(rr,2)+1.796*pow(rr,3)+2.058*pow(rr,4)); 	//ɢ��	
		sdir=sg-sdif_over;  

		
		
		
		
		/*
		sdif_over=sg*1.0/(1+exp(-3.98+8.05*rr)); 	//ɢ��	
		if(sdif_over>sg) sdif_over=sg;	
		sdir=sg-sdif_over;                                                     //ֱ��
  */ 
	}
		
/************* calculate ssun and sshade ***********************/

/* radiation from multiple scattering */
	c=0.07*omega*sdir*(1.1-0.1*lai_p)*exp(-cos_theta);
	
/* sdif_under */
    theta_avg=0.537+0.025*lai_p;
    sdif_under=sdif_over*exp(-0.5*omega*lai_p/theta_avg);
	
 
/* radiation for shaded leaves  X.F. Sep 2003*/
    // if ((lai_p<0.01 ) && (sdif_over-sdif_under)<0)
	if ((lai_p<0.01 ) && (sdif_over-sdif_under)<=0)
        
		*sshade=0.0;
	else	
		*sshade=(sdif_over-sdif_under)/lai_p+c;

 

/* radiation for sunlit leaves */
    if (cos_theta <0.01)
		*ssun=0.0;
	else
		*ssun=0.5*sdir/cos_theta +*sshade;
 
	
/* net short radiation for shaded leaves, X.F., Sep 2003 */
	c_for_net=alpha_l*omega*sdir*(1.1-0.1*lai_p)*exp(-cos_theta);;      //??????????????????????????????
     //if ((lai_p<0.01) && (sdif_over-sdif_under)<0.0)
      if ((lai_p<0.01) && (sdif_over-sdif_under)<=0.0)   //X.F. Sep 2003
		*n_sshade=0;
	else	
		*n_sshade=(sdif_over-sdif_under)/lai_p+c_for_net;       //????????????????????????��sshade/?????????????

/* net short radiation for sun lit leaves */
	*n_ssun=(1-alpha_l)**ssun + *n_sshade;
  
/* net short radiation for understory */

    if(cos_theta<0.05) cos_theta=0.05;   //==========================================================JUW 2013_06_16===================================================

	*n_sunder=(1-alpha_l)*(sdir*exp(-0.5*omega*lai_p/cos_theta)+sdif_under);
   
	
	if(*n_sunder<0)*n_sunder=0;



/* net short radiation for ground */
	theta_avg_under=theta_avg=0.537+0.025*lai_u;
	*n_sground=(1-alpha_g)*(sdir*exp(-0.5*omega*(lai_p+lai_u)/cos_theta)+sdif_under*exp(-0.5*omega*lai_u/theta_avg_under));
    if(*n_sground<0) *n_sground=0;
	
	// printf("*n_sground=%f \n", *n_sground);
    return 1;
}

__device__ int gpunet_longwave(
	float lai_o,
	float lai_u,
	float omega,
	float es,
	float ta,
	float *m_lnet_o,
	float *m_lnet_u,		             
	float *lnet_g)			             
{
	float sigma;			             //Stefan-Boltzmann constant =5.67*10^(-8) W m^-2 K^-4
	float epsilon_a, epsilon_o,epsilon_u,epsilon_g;
	float to,tu,tg;		             //temperature, in K
	float l_a,l_o,l_u,l_g;				 // longwave radiation
	float r_ctheta_o, r_ctheta_u;       //representive zenith angle
	float exponent_o, exponent_u;
	float lnet_o, lnet_u;

	sigma=5.67/100000000;

	epsilon_o=0.98;
	epsilon_u=0.98;
	epsilon_g=0.95;
//	epsilon_a=1.24*pow(((es)/ta), (1.0/7.0));  // es in mbar, ta in K
epsilon_a=1.72*pow(float((es)/ta), float(1.0/7.0));  // es in kpa, ta in K

	to=ta;	                              // in K
	tu=ta;
	tg=ta;

	// longwave radiation
	l_a=epsilon_a*sigma*pow(ta,(float)4.0);   //air
	l_o=epsilon_o*sigma*pow(to,(float)4.0);    //overstorey
	l_u=epsilon_u*sigma*pow(tu,(float)4.0);    //understorey
	l_g=epsilon_o*sigma*pow(tg,(float)4.0);     //ground
	
	// represntive angle
	r_ctheta_o=0.573+0.025*lai_o;
	r_ctheta_u=0.573+0.025*lai_u;

	exponent_o=exp(-0.5*lai_o*omega/r_ctheta_o);
	exponent_u=exp(-0.5*lai_u*omega/r_ctheta_u);

	//net longwave radiation
	lnet_o=(l_a + l_u* (1-exponent_u) + l_g* exponent_u - 2*l_o)*(1-exponent_o);
	lnet_u=(l_a*exponent_o + l_o* (1-exponent_o) + l_g - 2*l_u)*(1-exponent_u);
	*lnet_g=(l_a*exponent_o + l_o* (1-exponent_o))*exponent_u + l_u*(1-exponent_u) - l_g ;

	if (lai_o>0.01) *m_lnet_o =lnet_o/lai_o;
		else *m_lnet_o=0.0;
	if (lai_u !=0) *m_lnet_u = lnet_u/lai_u;
		else *m_lnet_u=0.0;


	return 1;
}