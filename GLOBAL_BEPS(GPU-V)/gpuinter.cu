#include "hip/hip_runtime.h"
/*************************************************************************
  Program:     inter.c
  --------
  Description:
  ----------- 
	Compute rain, snow to ground and to air.
  Details:
  -------
	input:
	b[4]: snow & rain interception coefficient (=0.00025 m/lai/day)
	z[3]: precipitation
	z[8]: daily total short wave radiation (kj/m2/d)��=J/(3.6*m2*s)=(w/m2)/3.6??????????
	z[10]: lai
	z[14]: daytime average temperature (C deg)		
	output:
	g[1]:ground rain 
	g[2]:ground snow
	g[4]:evaporation of precipitation from canopy (m of h2o/ha/day)	
	g[5]:sublimation of precipitation from canopy (m of h2o/ha/day)		
	g[6]:ground rain from canopy
	g[7]:ground snow from canopy
	g[11]:intercepted precip. 
***************************************************************************
  CCRS (EMS/Applications Division)
  Written by:   J. Liu      
  Modified by: X.F. Feng
  Last update:  July 2003
*****************************************************************************/
	
#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 

/*float min(float a,float b)
{
	if (a<b)
		return a;
	return b;
}
float max(float a,float b)
{
	if (a>b)
		return a;
	return b;
}
  */
__device__ void gpuinter(
    float  b[],float g[],float x[],float z[])
{
    // float lh_evp=2.5E+6; 
    // float lh_sub=2.8E+6;
    float  Eva_inter_sunlit,Eva_inter_shaded, Eva_inter,Rn;
float  a1;

	/* fresh snow */
	
	b[8]=0.05; 
 
/* intercepted precip */
	a1=b[4]*(1-exp(-0.5*z[10]))*5;     //*0.5;    // 2012-10-15�ջָ�, ����ϵ��0.5   z[10]ΪҶ���ָ�� 
    g[11]= min(z[3], a1);              // 2012-10-15�ջָ�  z[3]Ϊ��ˮ��    g[11] �ڲ������ˮ�� 

//g[11]= __min(z[3],z[10]* b[4]*0.25);      // 2012-10-15��ǰ�õķ���



   
/* rain vs snow */     //g[1]Ϊ 
 
    if (z[14]  > 0.0 ) {    
        g[1] = z[3] - g[11];
		g[2] = 0.0; 
    } 
    else {
	    g[1]=0.0; 
        g[2] = z[3] - g[11];		
     }

/* water loss from canopy */
/*	
if (z[14]  > 0.0 ) {
        g[4] = min(g[11], z[8]*b[7]/lh_evp);
		g[5] =0.0;
 
    } 
    else {
		g[4] =0.0;
        g[5] = min(g[11], z[8]*b[8]/lh_sub);	
     }
*/

   


 
if (z[14]  > 0.0 ) {
    Rn=z[38]*z[32]+z[39]*z[33];

	Eva_inter_sunlit= z[18]*gpupenmon(z[14],z[16],Rn,g[40],0.05)*z[32];

    //Eva_inter_sunlit= z[18]*penmon(z[14],z[16],z[38],g[40],0.005)*z[32];

	Eva_inter_shaded= 0;//z[18]*penmon(z[14],z[16],z[39],g[40],0.005)*z[33];  

    Eva_inter=Eva_inter_sunlit+Eva_inter_shaded;
   
	if(g[11]>0) {
		g[90]=__min(1.0,g[11]/Eva_inter);     
	}
		else g[90]=0.0;
	
	   g[4] = __min(g[11], Eva_inter);   //2009_8_03 Ju �Ķ�
	// g[4] = min(g[11], z[8]*b[7]/lh_evp); //ԭ�����㷨
	
	   
	   g[5] =0.0;
 
    } 
    else {
		g[4] =0.0;
    Rn=z[38]*z[32]+z[39]*z[33];

	Eva_inter_sunlit= z[18]*gpupenmon(z[14],z[16],Rn,g[40],0.05)*z[32];

    //Eva_inter_sunlit= z[18]*penmon(z[14],z[16],z[38],g[40],0.005)*z[32];

	Eva_inter_shaded= 0;//z[18]*penmon(z[14],z[16],z[39],g[40],0.005)*z[33];   

    Eva_inter=Eva_inter_sunlit+Eva_inter_shaded;

    if(g[11]>0) g[90]=__min(1.0,g[11]/Eva_inter);

	else g[90]=0.0;

	g[5] = __min(g[11], Eva_inter);   //2009_8_03 Ju �Ķ� 

	//g[5] = __max(g[11], 0)	;


  if(g[5]<0.0)  g[5]=0;



		
	//	g[5] = min(g[11], z[8]*b[8]/lh_sub);//ԭ�����㷨	
     }



/* canopy water to ground */
	if (z[14]  > 0.0 ) {      
		g[6] = __max(0.0, g[11]-g[4]);
		g[7] =0.0;
    } 
    else { 
		g[6]=0.0;
		g[7] = __max(0.0, g[11]-g[5]);
     }

    return;
}
 
