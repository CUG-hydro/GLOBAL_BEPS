#include "hip/hip_runtime.h"
/*************************************************************************
  Program:     doflux.c
  --------
  Description:
  ----------- 
	Calculate and update the fluxes in carbon and water cycles.
***************************************************************************
  CCRS (EMS/Applications Division)
  Written by:   J. Liu      
  Last update:	December 2000
*****************************************************************************/
	 
#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 


   __device__ void gpudoflux(float CNl,int long pix,long index,float b[],float g[],float x[],float z[],
                              float co, float TI,float soilw1,float soilw2,float soilw3, struct hy_c1 HY1[],struct hy_c2 HY2[], struct hy_c3 HY3[],
                               float *soilwn1,float*soilwn2,float*soilwn3)
    {          
   float aa,bb,B[5],K[5],Psi[5],depth[5],kk[5];
   float soilw_in=0, theta_o[5],theta_n1,theta_n2,theta_n3,ET=0;
   float rr[4],rrr[4],x22_old,x21_old;
    float rrr1,rrr2,rrr3;
    float baita; //��ϵ�ڲ�ͬ������ı仯�ٶ�   // 2013-7-20��Ϊ0.96  // 2013-9-17��Ϊ0.95
    float depth3;
    float AT,ratio; 

	if (z[23]==13 || z[23]==16 || z[23]==17 || z[23]==18 ) {  //13 �ݵأ� 16��17��18Ϊũ��
     baita=0.93; depth3=0.6;
	 } 
	else if (z[23]==1 || z[23]==7 || z[23]==8  ) {  //������Ҷ��
    baita=0.965;
	depth3=4.0;                                                            //2018-3-1: 5.0 ��Ϊ3.0
	 

	
	} 
    else if (z[23]==2 || z[23]==3   ) {  //��Ҷ��Ҷ��
     baita=0.96; depth3=1.5;   //2014-08-19 ��2.0 ��Ϊ1��0 
	 } 

    else if (z[23]==5   ) {  //��Ҷ��Ҷ��
     baita=0.96; depth3=1.0;
	 } 
     else if (z[23]==6 || z[23]==9 ) {  //�콻��
     baita=0.96; depth3=1.0;
	 } 

     else if (z[23]==10 ) {  //������Ҷ��
     baita=0.965; depth3=1.5;   //2014-08-19 ��2.0 ��Ϊ1��0 
	 } 
 
	 else if (z[23]==4 ) {  //������Ҷ��
		 baita=0.96; depth3=1.0;   //2014-08-19 ��2.0 ��Ϊ1��0 
	 } 



     else if (z[23]==11 || z[23]==12 || z[23]==14) {  //��ľ
     baita=0.955; depth3=1.5;  //2014-08-19 ��2.0 ��Ϊ1��0 
	 } 
    else {
    baita=0.955; depth3=1.5;  //2014-08-19 ��2.0 ��Ϊ1��0 
    }


depth[0]=depth1+depth2+depth3;       
depth[1]=depth1;  depth[2]=depth2;   depth[3]=depth3;  //�����������

//��ϵ�ķֲ�����, baita ȡСֵ����ϵ�ֲ�����ȼ�С��

rr[1]=1-pow(baita, depth[1]*100);
rr[2]=pow(baita, depth[1]*100)-pow(baita, (depth[1]+depth[2])*100);
rr[3]=1.0-rr[1]-rr[2];

//����B 
Psi[1]=HY1[pix].SP;  Psi[2]=HY2[pix].SP; Psi[3]=HY3[pix].SP;

B[1]  =HY1[pix].B;     B[2]=HY2[pix].B;    B[3]=HY3[pix].B;

K[1]  =HY1[pix].K;     K[2]=HY2[pix].K;    K[3]=HY3[pix].K;

if(z[4]<-2.0) {   //JUW20180125
kk[1]=K[1] *0.5; 
kk[2]=K[2] *0.5;
kk[3]=K[3] *0.5;
}
else{
	kk[1]=K[1]; 
	kk[2]=K[2];
	kk[3]=K[3];
}



   x[21]=soilw1;
   x[22]=soilw2;
   x[23]=soilw3;
   b[34]=HY1[pix].WP;//0.12; ��ήϵ��	        
   b[35]=HY1[pix].FC;//0.35; ����ˮ��	 
   b[36]=HY1[pix].PR;//0.52;    


//  resistance 
//	ra(b, g, x, z); 
	if( z[23]==16 && z[23]==17  && z[23]==18 || z[23]==13) g[40]=1.0/30.0;   //�����߽���迹ȡ����
	else	g[40]=1.0/5.0; 





//  canopy interception  
    gpuinter(b, g, x, z);    

//  snowmelt routine  
     gpumelt (b, g, x, z);

	 gpurs(pix,b, g, x, z,rr,&rrr1,&rrr2,&rrr3,TI,HY1,HY2,HY3);  //���������迹

	 //rsoil(b, g, x, z); 
                                                
	 gpucarbon(CNl,index,b, g, x, z,co,TI); //����GPP��NPP
   
// Penman - Monteith estimate of transpiration, overstory  
	g[16]=		gpupenmon(z[14],z[16],z[38],g[40],g[20])*z[32]; //��Ҷ��������
	g[16]=g[16]+gpupenmon(z[14],z[16],z[39],g[40],g[21])*z[33]; //��Ҷ��������
  
// �ڲ���������canopy transpiration  
	 g[17] = g[16]* z[18];

     //g[17]=__max(0,g[17]);  g[17]=__min(100,g[17]);
 if(g[17]<0.0 )   g[17]=0.0;
  if(g[17]>100.0)   g[17]=100.0;



// Penman - Monteith estimate of evaporation from soil  
	// if snowpack>0 or snow/ice area  evp_soil=0  
	if((z[14]<0) || (z[23]==31)) g[12]=0.0;
	else g[12]= z[18]*gpupenmon(z[14],z[16],z[41],0.5*g[40],g[41]);        //2012-10-15��Ϊ0.5*g[40];  
  
// Urban area 
	if (z[23]==13) g[12]=0.8*g[12];
	 // g[12]=__max(0,g[12]);
    if(g[12]<0.0)  g[12]=0.0;

	
// Penman - Monteith estimate of evaporation from understory  
// if LAI_u z[42]=0, temperature z[14]<0 or  ice z[23]=31, trans_under=0  
	if( ((z[14]<0) || (z[23]==31)) || z[42]==0) g[19]=0.0;
	else g[19]=z[18]*gpupenmon(z[14],z[16],z[40],g[40],g[19]);
	 
	
	//g[19]=__max(0,g[19]);   g[19]= __min(10,g[19]);
	
if(g[19]<0.0001) g[19]=0.0001;  
if(g[19]>100.0)  g[19]=100.0;

	//g[19]=0;// z[18]*penmon(z[14],z[16],z[40],g[40],g[19]); 
  
// ********** Water update ***************** 

//  snow = snow + ground snow + snow from canopy - melt - sublimation  
    x[1] = x[1] + g[2]+ g[7] - g[3]- g[13]; 
 
// swc[i]=swc[i-1] + ground rain + snowmelt + rain from tree - evap from soil-trans 

//==       ----------------------------------------------------------------------------------------------------------------------------��������������������²�������ˮ��
/*
if(TI>0)ratio=0.5;
else {
//ratio=0.5+0.0*TI;
//if(ratio<0.25) ratio=0.25;
ratio=0.5;
}
*/

ratio=0.5+0.2*z[45];

if(ratio<0.01) ratio=0.01;
if(ratio>=0.5)  ratio=0.5;


	if(soilw3>HY3[pix].FC ) {
       aa=ratio*kk[3]*pow((float)(soilw3/HY3[pix].PR),float(2.0*B[3]*1.2+3));
     	if(aa>(soilw3-HY3[pix].FC)*depth[3])  aa=(soilw3-HY3[pix].FC)*depth[3];
         }
	else aa=0;

//aa=0;

        //������澶������   
        /*
		if(soilw1<b[35] ) bb=0;  
		else{
		bb=pow(soilw1/b[36],3.0);
	    x[1]=x[1];
		}*/
        //������澶������   

       if((g[1]+ g[3] + g[6])<0.005) bb=0;
	   else{
        if(soilw1>(b[35]*0.75)) bb=pow(soilw1/b[36],(float)3.0);
		else  bb=0;
	   }
	
		
		//���� ����ս�ˮ��С��5mm, û�е��澶��

//=====================================================================================================================================================================
	
	soilw_in = (g[1]+ g[3] + g[6])*(1.0-bb);         //g[1]Ϊ��Ч��ˮ����g[3]Ϊ��ѩ���룬g[6]Ϊ�ڲ�����������Ľ�ˮ  
	
	ET = g[12]+g[17]+g[19];
	

	//���ں���������

	x[21] = soilw1+(soilw_in - (g[17]*rrr1+g[12]))/depth[1];
    
	x[22] = soilw2 - (g[17]*rrr2)/depth[2];
    
	x[23] = soilw3 - (g[17]*rrr3+aa)/depth[3];
	
	if(x[21]>HY1[pix].PR){ //���������,�����ˮ�ֵ��ڶ���
	x21_old= x[21];
	x[22]  = x[22]+ (x[21]-HY1[pix].PR)*depth[1]/depth[2];
    x[21]=HY1[pix].PR;
      
	
	if(x[22]>HY2[pix].PR){//���һ�͵ڶ��������,�����ˮ���γɵر���
     x22_old= x[22];
	// x[22]=__min(x[22],b[36]);

 x[22]=HY2[pix].PR;

	 g[33]=(x22_old-HY2[pix].PR)*depth[2];
	  }
	}
   /* if soil water goes negative set, b[34]: wilting point   */ 
   
   /*
   b[34]=HY1[pix].WP;//0.12; ��ήϵ��	        
   b[35]=HY1[pix].FC;//0.35; ����ˮ��	 
   b[36]=HY1[pix].PR;//0.52;    
   */

 //if(x[21]<HY1[pix].WP*0.5) x[21]=HY1[pix].WP*0.5;      //ϵ��0.5 ��2012-10-15��

 theta_o[1]=x[21]/HY1[pix].PR; 
 theta_o[2]=x[22]/HY2[pix].PR;
 theta_o[3]=x[23]/HY3[pix].PR;

gpuv_moisture(pix,depth3, B,kk, Psi,HY1,HY2,HY3, theta_o, &theta_n1,&theta_n2,&theta_n3);  //��������ˮ�ֵĴ�ֱ������

x[21]=theta_n1*HY1[pix].PR; 
x[22]=theta_n2*HY2[pix].PR;
x[23]=theta_n3*HY3[pix].PR;
 
/*

x[21]=__min(HY1[pix].PR,x[21]); 
x[22]=__min(HY2[pix].PR,x[22]);
x[23]=__min(HY3[pix].PR,x[23]);


 
x[21]=__max(0.5*HY1[pix].WP,x[21]); 
x[22]=__max(0.5*HY2[pix].WP,x[22]);
x[23]=__max(0.5*HY3[pix].WP,x[23]);
 */



if(x[21]>HY1[pix].PR) x[21]=HY1[pix].PR; 
if(x[22]>HY2[pix].PR) x[22]=HY2[pix].PR; 
if(x[23]>HY3[pix].PR) x[23]=HY3[pix].PR; 

 if( x[21]<0.5*HY1[pix].WP) x[21]=0.5*HY1[pix].WP; 
 if( x[22]<0.5*HY2[pix].WP) x[22]=0.5*HY2[pix].WP; 
 if( x[23]<0.5*HY3[pix].WP) x[23]=0.5*HY3[pix].WP; 


*soilwn1=x[21];
*soilwn2=x[22];
*soilwn3=x[23];


/*  outflow = outflow + new outflow  */

   // x[3] = x[3] + g[33]+aa+bb;          //�ܵľ����� ��Ҫ�Դ˱������л������õ�������ܾ���

	x[31]=x[31]+aa;                       //���¾���

	x[32]=x[32]+(g[1]+ g[3] + g[6])*bb+g[33];            //�ر���

    x[3] = x[3] + x[31]+x[32];//g[33]+aa+bb;     //�ܵľ����� ��Ҫ�Դ˱������л������õ�������ܾ���
	
	
	/*	trans = trans + new trans */

    x[4]=  x[4]+g[17]  +g[19];   // �ܵ�����
 
/*  evapo = evapo + evap_canopy + sublimatio_canopy + evap_soil + sublimation_snow */

   x[5] =x[5] + g[4]+g[5]+g[12] +g[13];    //������

/*  understory = trans_under + new trans_under */
    x[18] = x[18] + g[19];   

/********* carbon update **********/ 

    x[6] = x[6] + g[26]; 				/* acumulated NPP */
    x[24]= g[26]*1000.0;
//   x[7] = x[7] + (g[25]+g[30]+g[31]); 	/* acumulated total Rm */
//	x[14] = x[14] + g[25]; 				/* acumulated leaf Rm */
//	x[15] = x[15] + g[30]; 				/* acumulated stem Rm */
//	x[16] = x[16] + g[31]; 				/* acumulated root Rm */

   x[11]= x[11]+ g[24];				/* acumulated gpp */

/********* For this run only, uptake E **********/ 

	x[7] = x[7] +g[4]; 					/* E_can */
	x[14] = x[14] + g[5]; 				/* S_can */
	x[15] = x[15] + g[12]; 				/* E_soil */
	x[16] = x[16] + g[13]; 				/* S_soil */

    return;
}





