#include "hip/hip_runtime.h"
/*************************************************************************
  Program:     rs.c
  --------
  Description:
  ----------- 
	Output stomatal resistance/conductance.
***************************************************************************
  CCRS (EMS/Applications Division)
  Written by:   J. Liu   
		(Based on BIOME-BGC)   
  Last update:	May 1998
*****************************************************************************/
	

#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 


__device__ void gpurs(int long pix,float b[],float g[],float x[],float z[],float r[],float *rrr1,float *rrr2,float *rrr3,float TI,struct hy_c1 HY1[],struct hy_c2 HY2[], struct hy_c3 HY3[])
{  

/* declaration for temporary variables/functions */
	float ppfd;
 	float tavg;
	float tmin;
	float topt, tmax;
	float vpd,vpd_open,vpd_close;
 	//float psi,psi_open,psi_close;
	float m_tavg, m_psi, m_ppfd, m_vpd;//, m_tmin,  m_co2;
	float m_most;
 	float lai;
    
	float ppfd_coef;	
	float cs,cc,cbl;
	float i;	/* index for 0:sunlit leaves and 1:shaded leaves */
	float lai_under;
	  int short		  lc_p;
    float rr[4],mm[4];
    float m0;
    float min_psi;
    float aa1;

	/* assign variables that are used more than once */
  	tavg =      z[14];      //�����¶�
	tmin =      z[5];
	vpd =       0.1*z[16];		/* in kPa */
	ppfd_coef=  0.01;
	


	//topt=20+(90-z[45])*0.08;                          //Ϊγ�ȵľ���ֵ��2017-11-08��0.05��Ϊ0.08 
	
	topt=22.5+z[45]*0.25;                          //2017-11-17���޸ģ�Z[45]Ϊ����ƽ���¶� 
	if(topt<22.5) topt=22.5;


	tmax=36.0+z[45]*0.25;                           //ԭ��Ϊȫ��40.0�� 2017-11-17���޸ģ�Z[45]Ϊ����ƽ���¶�  
	if(tmax<36.0) tmax=36.0;                               



	lc_p=(short  int)z[23];

//========================================================================================================================================================================//
	//if(lc_p==16 || lc_p==17 || lc_p==18)    min_psi=0.2;
	//else                                    min_psi=0.05;

min_psi=0.01;




   	//psi =       1;
	lai =       z[10];

	//psi_open =  -1;			/* in -MPa */
	//psi_close = -8;
	//vpd_open =  7.5;		/* in mb */
	//vpd_close = 20.0;
	
	


   
/*
	switch(lc_p)
	{
	case 1:// conifer 
		lai_under=1.175*exp(-0.991*z[10]);
       	break;	
	case 2:  				// deciduous forest 
		lai_under=1.5;
       	break;
	case 5:  // mixe 
		lai_under=0.5*(1.5+1.175*exp(-0.991*z[10]));
		break;	
	default:
		lai_under=0.05;
  	}													
   */

lai_under=0.01;


 for(i=0;i<=1; i++)
 {      //������׵��ȵ�Ӱ�� 
        if (i==0) ppfd =0.5*4.55*z[36];   //������Ҷ����ͨ���ܶ�
        else      ppfd =0.5*4.55*z[37];   //������Ҷ����ͨ���ܶ�
    
    	m_ppfd = ppfd * ppfd_coef /(1.0 + (ppfd * ppfd_coef));

	/* use the tempcurve function to generate the daily average air 
	temperature conductance multiplier */

	if(tavg<0 || tavg>tmax)  m_tavg=0.0; //�¶ȶ����׵��ȵ�Ӱ��
		
	else
	{
		if(tavg<topt)   m_tavg=log(tavg+1)/log(topt+1);
		else    		m_tavg=cos(3.1415926*(tavg-topt)/(2*(tmax-topt)));
	}

	/* soil-leaf water potential multiplier */
	 if(x[21]<=HY1[pix].WP)  m0=(float) 0.0005;  //���������ˮ�����ڵ�ήϵ�����ܸ�
     else m0=(float) ((x[21]-HY1[pix].WP)/(HY1[pix].PR-HY1[pix].WP));
     
	 
	 g[41]=1.0/100.0*m0;     //��������迹   //2017��10��20����1/150.0 ��Ϊ1/100.0
	
//============================================================================================================================================/
if( lc_p==16 || lc_p==17 ||lc_p==18){

		//��һ��
		if(x[21]<=HY1[pix].WP)  mm[1]= min_psi;  //very dry__min(1.0,2.5*(x[21]-wilting)/(field-wilting))
		else if(x[21]>=HY1[pix].WP && x[21]<=HY1[pix].FC)
		{
			  mm[1]= (x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP)*2
			-(x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP)*(x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP);

		//	mm[1]=(x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP);

		}
		else mm[1]=1-0.25*(x[21]-HY1[pix].FC)/(HY1[pix].PR-HY1[pix].FC);  //2013-06-13��0.4 ��Ϊ0.2

		if(mm[1]<min_psi) mm[1]=min_psi;

		//�ڶ���
		if(x[22]<=HY2[pix].WP)  mm[2]= min_psi;  //very dry__min(1.0,2.5*(x[21]-wilting)/(field-wilting))
		else if(x[22]>=HY2[pix].WP && x[22]<=HY2[pix].FC) {
			
			mm[2]= (x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP)*2
			-(x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP)*(x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP);

			//mm[2]=(x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP);
		}
		else mm[2]=1-0.25*(x[22]-HY2[pix].FC)/(HY2[pix].PR-HY2[pix].FC);  //2013-06-13��0.4 ��Ϊ0.2

		if(mm[2]<min_psi) mm[2]=min_psi;



		//������
		if(x[23]<HY3[pix].WP)  mm[3]= min_psi;  //very dry__min(1.0,2.5*(x[21]-wilting)/(field-wilting))
		else if(x[23]>=HY3[pix].WP && x[23]<=HY3[pix].FC) {
			 mm[3]= (x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP)*2
			-(x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP)*(x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP);

			//mm[3]=(x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP);
		}
		else mm[3]=1-0.25*(x[23]-HY3[pix].FC)/(HY3[pix].PR-HY3[pix].FC);    //2013-06-13��0.4 ��Ϊ0.2


		if(mm[3]<min_psi) mm[3]=min_psi;


		//rr[0]=r[1]*mm[1]+r[2]*mm[2]+r[3]*mm[3];


		rr[1]=r[1];//*mm[1]/rr[0];
		rr[2]=r[2];//*mm[2]/rr[0];
		rr[3]=r[3];//*mm[3]/rr[0];

		m_psi=mm[1]*rr[1]+mm[2]*rr[2]+mm[3]*rr[3];

		//m_psi=rr[1]+rr[2]+rr[3];

		*rrr1=rr[1];

		*rrr2=rr[2];

		*rrr3=rr[3];

		if(m_psi<0)    m_psi=0;
		if(m_psi>1.0) m_psi=1.0;

}


else if( lc_p==11 ||lc_p==12|| lc_p==13 ||lc_p==14){

		//��һ��
		if(x[21]<=HY1[pix].WP)  mm[1]= min_psi;  //very dry__min(1.0,2.5*(x[21]-wilting)/(field-wilting))
		else if(x[21]>=HY1[pix].WP && x[21]<=HY1[pix].FC)
		{
			//  mm[1]= (x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP)*2
			//-(x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP)*(x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP);

			mm[1]=(x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP);

		}
		else mm[1]=1-0.25*(x[21]-HY1[pix].FC)/(HY1[pix].PR-HY1[pix].FC);  //2013-06-13��0.4 ��Ϊ0.2

		if(mm[1]<min_psi) mm[1]=min_psi;

		//�ڶ���
		if(x[22]<=HY2[pix].WP)  mm[2]= min_psi;  //very dry__min(1.0,2.5*(x[21]-wilting)/(field-wilting))
		else if(x[22]>=HY2[pix].WP && x[22]<=HY2[pix].FC) {
			
			//mm[2]= (x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP)*2
			//-(x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP)*(x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP);

			mm[2]=(x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP);
		}
		else mm[2]=1-0.25*(x[22]-HY2[pix].FC)/(HY2[pix].PR-HY2[pix].FC);  //2013-06-13��0.4 ��Ϊ0.2

		if(mm[2]<min_psi) mm[2]=min_psi;



		//������
		if(x[23]<HY3[pix].WP)  mm[3]= min_psi;  //very dry__min(1.0,2.5*(x[21]-wilting)/(field-wilting))
		else if(x[23]>=HY3[pix].WP && x[23]<=HY3[pix].FC) {
			// mm[3]= (x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP)*2
			//-(x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP)*(x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP);

			mm[3]=(x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP);
		}
		else mm[3]=1-0.25*(x[23]-HY3[pix].FC)/(HY3[pix].PR-HY3[pix].FC);    //2013-06-13��0.4 ��Ϊ0.2


		if(mm[3]<min_psi) mm[3]=min_psi;


		//rr[0]=r[1]*mm[1]+r[2]*mm[2]+r[3]*mm[3];


		rr[1]=r[1];//*mm[1]/rr[0];
		rr[2]=r[2];//*mm[2]/rr[0];
		rr[3]=r[3];//*mm[3]/rr[0];

		m_psi=mm[1]*rr[1]+mm[2]*rr[2]+mm[3]*rr[3];

		//m_psi=rr[1]+rr[2]+rr[3];

		*rrr1=rr[1];

		*rrr2=rr[2];

		*rrr3=rr[3];

		if(m_psi<0)    m_psi=0;
		if(m_psi>1.0) m_psi=1.0;

}
else{

 //��һ��
if(x[21]<=HY1[pix].WP)  mm[1]= min_psi;  //very dry__min(1.0,2.5*(x[21]-wilting)/(field-wilting))
else if(x[21]>=HY1[pix].WP && x[21]<=HY1[pix].FC)
{
	//  mm[1]= (x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP)*2
		  //  -(x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP)*(x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP);

 mm[1]=(x[21]-HY1[pix].WP)/(HY1[pix].FC-HY1[pix].WP);

}
else mm[1]=1-0.25*(x[21]-HY1[pix].FC)/(HY1[pix].PR-HY1[pix].FC);  //2013-06-13��0.4 ��Ϊ0.2
 
    if(mm[1]<min_psi) mm[1]=min_psi;

//�ڶ���
if(x[22]<=HY2[pix].WP)  mm[2]= min_psi;  //very dry__min(1.0,2.5*(x[21]-wilting)/(field-wilting))
else if(x[22]>=HY2[pix].WP && x[22]<=HY2[pix].FC) {
	// mm[2]= (x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP)*2
		   //-(x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP)*(x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP);

	mm[2]=(x[22]-HY2[pix].WP)/(HY2[pix].FC-HY2[pix].WP);
}
     else mm[2]=1-0.25*(x[22]-HY2[pix].FC)/(HY2[pix].PR-HY2[pix].FC);  //2013-06-13��0.4 ��Ϊ0.2
 
	if(mm[2]<min_psi) mm[2]=min_psi;



//������
if(x[23]<HY3[pix].WP)  mm[3]= min_psi;  //very dry__min(1.0,2.5*(x[21]-wilting)/(field-wilting))
else if(x[23]>=HY3[pix].WP && x[23]<=HY3[pix].FC) {
	 // mm[3]= (x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP)*2
		   //-(x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP)*(x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP);

 	mm[3]=(x[23]-HY3[pix].WP)/(HY3[pix].FC-HY3[pix].WP);
}
	else mm[3]=1-0.25*(x[23]-HY3[pix].FC)/(HY3[pix].PR-HY3[pix].FC);    //2013-06-13��0.4 ��Ϊ0.2
 
	
	    if(mm[3]<min_psi) mm[3]=min_psi;

 
    rr[0]=r[1]*mm[1]+r[2]*mm[2]+r[3]*mm[3];


rr[1]=r[1]*mm[1]/rr[0];
rr[2]=r[2]*mm[2]/rr[0];
rr[3]=r[3]*mm[3]/rr[0];
 
m_psi=mm[1]*rr[1]+mm[2]*rr[2]+mm[3]*rr[3];

//m_psi=rr[1]+rr[2]+rr[3];

 *rrr1=rr[1];
 
 *rrr2=rr[2];
 
 *rrr3=rr[3];

 if(m_psi<0)    m_psi=0;
  if(m_psi>1.0) m_psi=1.0;

	 }

g[80]=m_psi;

//m_psi=__max(0, m_psi);
//m_psi=__min(1, m_psi);
 

	/* CO2 multiplier */
//	m_co2 = 1.0;

	/* freezing night minimum temperature multiplier */
	//if (tmin > 0.0)        /* no effect */
	//	m_tmin = 1.0;
	//else
	//if (tmin < -8.0)       /* full tmin effect */
	//	m_tmin = 0.0;
	//else                   /* partial reduction (0.0 to -8.0 C) */
	//	m_tmin = 1.0 + (0.125 * tmin);
	
	// VPD�������ӣ�vapor pressure deficit multiplier, vpd in Pa 
	/*if(vpd<0.2)                       m_vpd=1.0;
    else if  (vpd>=0.2 && vpd<3.5 )   m_vpd=(3.5-vpd)/3.3;     //2012-10-14��2.8��Ϊ3.5
    else                              m_vpd=0.001;
*/

vpd_open=0.0+z[45]*0.01;
if(vpd_open<0.0) vpd_open=0.0;


vpd_close=3.0+z[45]*0.02;
if(vpd_close<3.0) vpd_close=3.0;

if(vpd<=vpd_open) m_vpd=1.0;
  else if  (vpd>vpd_open && vpd<=vpd_close )   m_vpd=(vpd_close-vpd)/(vpd_close-vpd_open);     //2012-10-14��2.8��Ϊ3.5
   else                                        m_vpd=0.001;
 


//if(vpd<0.93)                       m_vpd=1.0;
//else if  (vpd>=0.93 && vpd<4.0 )   m_vpd=pow((4.0-vpd)/(4.0-0.93),0.75)+0.0001;     //2017-05-13
 //else                              m_vpd=0.001;



	/* apply all multipliers to the maximum stomatal conductance */

//=====================================================================================================================================================================================/
	m_most= m_tavg* m_psi *  m_vpd;//;//*m_co2 * m_tmin ;
 
	//	m_most=1;
	cs =b[11]*m_ppfd*m_most;
	
	/* leaf boundary-layer conductance */
	//cbl=0.08;

       cbl=0.08;
	/* leaf cuticular conductance */
	cc=0.00005;

	/* final stomatal conductance for sunlit and shaded leaves  */

     aa1=cbl*(cc+cs)/(cbl+cc+cs);
     
	 
	// if(aa1<cs) cs=aa1;
    if (i==0) g[20] =cbl*(cc+cs)/(cbl+cc+cs);   //��Ҷ                   //2013��9��11�ոģ� ԭ������ѹ�Ͳ���
	else
	          g[21] =cbl*(cc+cs)/(cbl+cc+cs);   //��Ҷ

    }
	/* canopy conductance */
         g[22]=g[20]*z[32]+g[21]*z[33];

	/* canopy conductance for big leaf model */
        
		ppfd=0.5*4.55*z[20];
    
		m_ppfd = ppfd * ppfd_coef / (1.0 + (ppfd * ppfd_coef));
		
		g[22]=z[10]*b[11]*m_ppfd*m_most;

	/* understory conductance */
		ppfd=0.5*4.55*(z[9]-z[20]);
    
		m_ppfd = ppfd * ppfd_coef / (1.0 + (ppfd * ppfd_coef));
		g[19]=lai_under*b[39]*b[11]*m_ppfd*m_most;
		
    return;
} 
