#include "hip/hip_runtime.h"
/*************************************************************************
  Program:     doflux.c
  --------
  Description:
  ----------- 
	Calculate and update the fluxes in carbon and water cycles.
***************************************************************************
  CCRS (EMS/Applications Division)
  Written by:   J. Liu      
  Last update:	December 2000
*****************************************************************************/
	 
#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 


   __device__ void gpulw1(float FC1,float FC2, float FC3, float sw1, float  sw2, float sw3, float *lw1,float *lw2, float *lw3)
    {          
 
float m,Msat, Mopt,B1,B2,B3,B; 


B3=100.0;
//Layer1
m=-0.0008*FC1*FC1+0.0603*FC1-0.6769+0.5;

Mopt=-0.0062*FC1*FC1+1.0745*FC1+26.515;

Msat=(-0.00006*FC1*FC1+0.0124*FC1+0.0994);

B1=pow(sw1,m)-pow(Mopt,m);
B2=pow(Mopt,m)-pow(B3, m);
B=(B1/B2)*(B1/B2);

*lw1=0.75*pow(Msat,B)+0.25;

//Layer2
m=-0.0008*FC2*FC2+0.0603*FC2-0.6769+0.5;

Mopt=-0.0062*FC2*FC2+1.0745*FC2+26.515;
Msat=(-0.00006*FC2*FC2+0.0124*FC2+0.0994);

B1=pow(sw2,m)-pow(Mopt,m);
B2=pow(Mopt,m)-pow(B3, m);
B=(B1/B2)*(B1/B2);

*lw2=0.75*pow(Msat,B)+0.25;


//Layer3
m=-0.0008*FC3*FC3+0.0603*FC3-0.6769+0.5;

Mopt=-0.0062*FC3*FC3+1.0745*FC3+26.515;
Msat=(-0.00006*FC3*FC3+0.0124*FC3+0.0994);

B1=pow(sw3,m)-pow(Mopt,m);
B2=pow(Mopt,m)-pow(B3, m);
B=(B1/B2)*(B1/B2);

*lw3=0.75*pow(Msat,B)+0.25;

    return;
}





