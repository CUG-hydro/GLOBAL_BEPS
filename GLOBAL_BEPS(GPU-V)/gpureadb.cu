#include "hip/hip_runtime.h"
/*************************************************************************
  Program:     readb.c
  --------
  Description:
  -----------
 	Read biological parameters based on vegetation type.
***************************************************************************
  CCRS (EMS/Applications Division)
  Written by:   J. Liu      
  Modified by: X.F. Feng
  Last update:  Sep. 2003
*****************************************************************************/

#ifndef   MY_H_FILE      
#define   MY_H_FILE       
#include "gpubeps.cuh"
#endif 


  __device__  void gpureadb(float *b,  int short lc_p)      ///zfm��Ҫ��

{

	switch(lc_p)
	{

/* reference about max. canopy cond. from BIOME-BGC and ��ɳ�������� */ 
	case 4:  case 10:          //  evergreen needleleaf, temperature zone // evergreen needleleaf zhoulei 4.1 
		   
	      b[1]=20;// 8.2;		// specific leaf area m2 kg-1 C, from BIOME-BGC  
	      b[11]=0.004;			// b(11) max. canopy cond., h20  m s-1    //�ؼ�����    //2018-02-14 : 0.0040 to 0.0035
		 
		// b[12]=2.1;			// b(12) lwp at stomatal closure  -MPa  
		  b[12]=2.3;			// b(12) lwp at stomatal closure  -MPa , BIOME-BGC 
          b[16]=6.0e-04;		// b(16) max. mesophyll cond., CO2  m s-1      
          b[19]=0.0035/RTIMES;	// b(19) leaf resp co.  kg C-1 d-1 kg-1            /2017_11_06: 0,0025��Ϊ0.004
          b[20]=0.003/RTIMES;	// b(20) stem resp co.   kg C-1 d-1 kg-1           //2018-03-28: 0.0025 to 0.003
          b[21]=0.003/RTIMES;	// b(21) coarse root resp co.   kg C-1 d-1 kg-1    //2018-03-28: 0.0025 to 0.003
		  b[24]=0.0035/RTIMES;	// b(24) fine root resp co.   kg C-1 d-1 kg-1      //2018-03-28: 0.0025 to 0.003   2018-04-20: 0.003 to 0.004
		  
          b[25]=2.3;			// b(25) Q10: constant for exp. resp. for leaf	       
		  b[28]=2.0;			// b(28) Q10: constant for exp. resp. for stem	       
		  b[29]=2.0;			// b(29) Q10: constant for exp. resp. for roots	        
		  
		  b[39]=2.0;			// b(29) Ratio of gmax of overstory to understory	
          b[49]=45.0;            //Vcmax25  //2018-02-14: 50 to 45 
    	  b[50]=27.0;
       	break;


        case 5:                     // deciduous conifer    
		  b[1]=22.5;// 8.2;				
		  // b[1]= 10;              // specific leaf area m2 kg-1 C ,
		  // b[1]= 30;				// specific leaf area m2 kg-1 C , from BIIOME-BGC 
		 //  b[11]=0.0016;			// b(11) max. canopy cond., h20  m s-1         
          // b[11]=0.006;           // b(11) max. canopy cond., h20  m s-1 from BIIOME-BGC
		  b[11]=0.004;                                                                        // 20012_12_28,2018-02-14: 0.0040 to 0.0035
		  // b[12]=2.1;			// b(12) lwp at stomatal closure  -MPa  
		  b[12]=2.3;			// b(12) lwp at stomatal closure  -MPa , BIOME-BGC 
          b[16]=6.0e-04;		// b(16) max. mesophyll cond., CO2  m s-1      
       
		  b[19]=0.0030/RTIMES;	// b(19) leaf resp co.  kg C-1 d-1 kg-1        //2018-04-21:0.003 to 0.0035
          b[20]=0.0020/RTIMES;	// b(20) stem resp co.   kg C-1 d-1 kg-1        //2018-03-28: 0.0021 to 0.0025
          b[21]=0.0020/RTIMES;	// b(21) coarse root resp co.   kg C-1 d-1 kg-1 //2018-03-28: 0.0021 to 0.0025      
		  b[24]=0.0030/RTIMES;	// b(24) fine root resp co.   kg C-1 d-1 kg-1   //2018-03-28: 0.0033 to 0.003  2018-04-121: 0.0033 to 0.0035    
		  
		  b[25]=2.3;			// b(25) Q10: constant for exp. resp. for leaf	       
		  b[28]=2.0;			// b(28) Q10: constant for exp. resp. for stem	       
		  b[29]=2.0;			// b(29) Q10: constant for exp. resp. for roots	       
		  
		  b[39]=2.0;			// b(29) Ratio of gmax of overstory to understory	
          b[49]=45.0;             //Vcmax25  2018-02-14: 55 to 50    //2018-03-04: 50 to 45
		  b[50]=27.0;
       	break;


	case 1: case 7: case 8:                  /* evergreen broadleaf forest */ //zhoulei 4.1
           b[1]= 25.0;                       /* 1/2 (jane + biobgc) */         //2011��8��30��17.5 ��Ϊ25.0                    
		 
		  b[11]=0.004;  // b(11) max. canopy cond., h20  m s-1, BIOME-BGC     //���׵��ȣ�20171021��0.0065 ��Ϊ 0.005   //20171201:0.0055 //2018-03-01:0.0050 to 0.0045
		 
          //b[12]=2.1;		   /* b(12) lwp at stomatal closure  -MPa         */
          b[12]=3.9;			// b(12) lwp at stomatal closure  -MPa , BIOME-BGC 
          b[16]=8.0e-04;	   /* b(16) max. mesophyll cond., CO2  m s-1      */
        
		  b[19]=0.0012/RTIMES;  /* b(19) leaf resp co.  kg C-1 d-1 kg-1        *///2018-04-14:0.009 ��Ϊ0.0011 to 0.012    //2018-04-17: 0.012 to 0.1
          b[20]=0.0008/RTIMES;  /* b(20) stem resp co.   kg C-1 d-1 kg-1       *///20171120:0.008 ��Ϊ0.0008
          b[21]=0.0008/RTIMES;  /* b(21) coarse root resp co.   kg C-1 d-1 kg-1     *///20171120:0.008 ��Ϊ0.0008
		  b[24]=0.0012/RTIMES;  /* b(24) fine root resp co.   kg C-1 d-1 kg-1 */   //2018-04-14: 0.001 to 0.0012          //2018-04-17: 0.012 to 0.1          
		  
		  b[25]=2.0;			// b(25) Q10: constant for exp. resp. for leaf	     //2017-11-01��2.3��Ϊ2.0   
		  b[28]=1.9;			// b(28) Q10: constant for exp. resp. for stem	   //2017-11-01��2.1��Ϊ1.9
		  b[29]=1.9;			// b(29) Q10: constant for exp. resp. for roots	      //2017-11-01��2.13��Ϊ1.9 
		  
		  b[39]=0.5;		   /* b(29) Ratio of gmax of overstory to understory	*/
		
		  b[49]=30.0;          //Vcmax25   //2014-09-12 : 65 to 40     //2017-10-21��Ϊ37.5�� //2018-02-14: 35.0 to 32.5 
		
		   b[50]=27.0;          //2017-10-26��30.0 ��Ϊ25��0
		  
		  break;

	
	
	case 2: case 3:                        /* deciduous broadleaf forest */ 
		 // b[1]= 32.0;			   /* specific leaf area m2 kg-1 C, from BIOME-BGC */
		 //  b[1]= 26.6;			   /* specific leaf area m2 kg-1 C */	
		  b[1]=26.5; 
          
		  b[11]=0.005;                                //2012��12��28����0.005��Ϊ0.006 // 20013_06_28:.0.0065     20171119:0.0065��Ϊ0.005 
		         
          b[12]=2.1;		   /* b(12) lwp at stomatal closure  -MPa         */
          // b[12]=2.2;			// b(12) lwp at stomatal closure  -MPa , BIOME-BGC 
          b[16]=8.0e-04;	   /* b(16) max. mesophyll cond., CO2  m s-1      */
        
		  b[19]=0.0035/RTIMES;  /* b(19) leaf resp co.  kg C-1 d-1 kg-1        */  //zfm5-14  ԭ0.006 Խ�����Խ��
          b[20]=0.0025/RTIMES;  /* b(20) stem resp co.   kg C-1 d-1 kg-1       *///2017-11-06: 0.0012��Ϊ0.0015
          b[21]=0.0025/RTIMES;  /* b(21) coarse root resp co.   kg C-1 d-1 kg-1       *///2017-11-06: 0.0012��Ϊ0.0015
		  b[24]=0.0035/RTIMES;  /* b(24) fine root resp co.   kg C-1 d-1 kg-1 */
		
		  b[25]=2.3;			// b(25) Q10: constant for exp. resp. for leaf	       
		  b[28]=2.0;			// b(28) Q10: constant for exp. resp. for stem	       
		  b[29]=2.0;			// b(29) Q10: constant for exp. resp. for roots	       
		  b[39]=0.5;		   /* b(29) Ratio of gmax of overstory to understory	*/
		
		  b[49]=50.0;              //2014-09-12 58 to 62               //Vcmax25  //2018-03-04: 55 to 50
		  b[50]=30.0;  
		  break;
        //////////////////////////////////////////////////////////juw
	case 6: case 9: /* mixed */ 
		  //b[1]= 17;			/* specific leaf area m2 kg-1 C */	
			b[1]= 24.0;                                      
          //b[1]= 22;			/* specific leaf area m2 kg-1 C */
		  //b[11]=0.006;		/* b(11) max. canopy cond., h20  m s-1 
		   b[11]=0.0045;		//* b(11) max. canopy cond., h20  m s-1              2018-02-14: 0.0045 to 0.0040   
           
		  b[12]=2.3;		/* b(12) lwp at stomatal closure  -MPa         */
          b[16]=7.0e-04;	/* b(16) max. mesophyll cond., CO2  m s-1      */
         
		  b[19]=0.0035/RTIMES;  // b(19) leaf resp co.  kg C-1 d-1 kg-1               2009_5_14,��0.004 ��Ϊ0.0042.      2018-04-22 0.003 to 0.004
          b[20]=0.0018/RTIMES;  /* b(20) stem resp co.   kg C-1 d-1 kg-1      */      //2017-11-06: 0.0014��Ϊ0.0015     2018-04-22 0.0015 to 0.002
          b[21]=0.0018/RTIMES;  /* b(21) coarse root resp co.   kg C-1 d-1 kg-1       *///2017-11-06: 0.0014��Ϊ0.0015    2018-04-22 0.0015 to 0.002
		  b[24]=0.003/RTIMES;  /* b(24) fine root resp co.   kg C-1 d-1 kg-1       */  // 2018-04-18 0.003 to 0.0035      2018-04-22 0.003 to 0.004
		 
		  b[25]=2.3;			// b(25) Q10: constant for exp. resp. for leaf	       
		  b[28]=2.0;			// b(28) Q10: constant for exp. resp. for stem	       
		  b[29]=2.0;			// b(29) Q10: constant for exp. resp. for roots	       
		  b[39]=1.0;			/* b(29) Ratio of gmax of overstory to understory	*/
	     
		  b[49]=40.0;             //Vcmax25  2018-02-14: 45 to 40
		  b[50]=27.0;            
		  break;


/* Sep. 2003, form bio-bGC  zfm5.12*/
	case 11: case 12:   // 8:closed shrub;9:open shrub; 7: woody savanna;6:savannas; 11: permanent wetlands
        //  b[1]=12.0;				// specific leaf area m2 kg-1 C, from BIOME-BGC	
          b[1]=28.75;     //2011��7��27����15.0 ��Ϊ28.75
		  b[11]=0.0045;			                       // b(11) max. canopy cond., h20  m s-1, from BIOME-BGC  20171119:0.005��Ϊ0.0045 
		 // b[11]=0.0016;			// b(11) max. canopy cond., h20  m s-1         
		  b[12]=4.2;			// b(12) lwp at stomatal closure  -MPa , BIOME-BGC 
		 
		  b[19]=0.003/RTIMES;	// b(19) leaf resp co.  kg C-1 d-1 kg-1                                                      //2011��8��30����0.006��Ϊ0.005
          b[20]=0.0012/RTIMES;	// b(20) stem resp co.   kg C-1 d-1 kg-1      
          b[21]=0.0012/RTIMES;	// b(21) coarse root resp co.   kg C-1 d-1 kg-1       
		  b[24]=0.003/RTIMES;	// b(24) fine root resp co.   kg C-1 d-1 kg-1                                             //2011��8��30����0.003��Ϊ0.0025 
		 
		  b[25]=2.3;			// b(25) Q10: constant for exp. resp. for leaf	       
		  b[28]=2.1;			// b(28) Q10: constant for exp. resp. for stem	       
		  b[29]=2.1;			// b(29) Q10: constant for exp. resp. for roots	       
		  
		  b[39]=2.0;			// b(29) Ratio of gmax of overstory to understory
          b[49]=50.0;           //Vcmax25   //2018-03-04: 55 to 50
		   b[50]=27.0;
		  break; 
/* Sep. 2003, form bio-bGC  zfm5.12*/

	     case 14:   //  ============================ ================================================================ ��ľ
        //  b[1]=12.0;				// specific leaf area m2 kg-1 C, from BIOME-BGC	
          b[1]=28.5;                 //2009_5_14 ��Ϊ28.5�� ԭ��15.0  ����߹�����
		  b[11]=0.0045;			                           // b(11) max. canopy cond., h20  m s-1, from BIOME-BGC   20171119:0.005��Ϊ0.0035       
				  
		  // b[11]=0.0016;			// b(11) max. canopy cond., h20  m s-1         
		  b[12]=4.2;			// b(12) lwp at stomatal closure  -MPa , BIOME-BGC 
		  
		  b[19]=0.003/RTIMES;	// b(19) leaf resp co.  kg C-1 d-1 kg-1     //2011��8��30����0.006��Ϊ0.005   
          b[20]=0.0012/RTIMES;	// b(20) stem resp co.   kg C-1 d-1 kg-1      
          b[21]=0.0012/RTIMES;	// b(21) coarse root resp co.   kg C-1 d-1 kg-1       
		  b[24]=0.003/RTIMES;	// b(24) fine root resp co.   kg C-1 d-1 kg-1    //2011��8��30����0.003��Ϊ0.0025
		  
		  b[25]=2.3;			// b(25) Q10: constant for exp. resp. for leaf	       
		  b[28]=2.1;			// b(28) Q10: constant for exp. resp. for stem	       
		  b[29]=2.1;			// b(29) Q10: constant for exp. resp. for roots	       
		  b[39]=2.0;			// b(29) Ratio of gmax of overstory to understory 
         
		  b[49]=52.5;            //Vcmax25                               //2017-10-30��60��Ϊ50    /2017-10-30��60��Ϊ40  
		  b[50]=27.0;
		  break;

	     case 13: //  grasslands;
		   //b[1]= 10;
		   b[1]= 30;
		  //b[11]=0.019;		     // b(11) max. canopy cond., h20  m s-1  
		  b[11]=0.0045 ;              // //2018-02-14: 0.006 to 0.0045
		  b[12]=2.7;		/* b(12) lwp at stomatal closure  -MPa , from BIOME-BGC */
   		 
		  b[19]=0.004/RTIMES;	// b(19) leaf resp co.  kg C-1 d-1 kg-1     //2011��8��30����0.006��Ϊ0.005   
		  b[20]=0.003/RTIMES;	// b(20) stem resp co.   kg C-1 d-1 kg-1      
		  b[21]=0.002/RTIMES;	// b(21) coarse root resp co.   kg C-1 d-1 kg-1       
	      b[24]=0.004/RTIMES;	// b(19) leaf resp co.  kg C-1 d-1 kg-1     //2011��8��30����0.006��Ϊ0.005 

          b[25]=2.3;			// b(25) Q10: constant for exp. resp. for leaf	       
		  b[28]=2.1;			// b(28) Q10: constant for exp. resp. for stem	       
		  b[29]=2.1;			// b(29) Q10: constant for exp. resp. for roots	       
		  b[49]=50.0;            //Vcmax25  //2018-02-14: 55 to 50     //2018-04-15 :   50 to 55

         b[50]=30.0;
		  break;

         case 16: case 17:case 18:   // 16: cropland;
		   //b[1]= 10;
		   b[1]= 30;
		  //b[11]=0.019;		     // b(11) max. canopy cond., h20  m s-1  
		  b[11]=0.0055;                // form Jane   //5.14  ԭ0.010 ,0.008 //5.16   //2012��12��4����0.0060 ��Ϊ0.005
		  b[12]=2.7;	             /* b(12) lwp at stomatal closure  -MPa , from BIOME-BGC */
   		
		 
		  b[19]=0.004/RTIMES;	// b(19) leaf resp co.  kg C-1 d-1 kg-1     //2011��8��30����0.006��Ϊ0.005   
		  b[20]=0.003/RTIMES;	// b(20) stem resp co.   kg C-1 d-1 kg-1      
		  b[21]=0.0015/RTIMES;	// b(21) coarse root resp co.   kg C-1 d-1 kg-1       
	      b[24]=0.004/RTIMES;	// b(19) leaf resp co.  kg C-1 d-1 kg-1     //2011��8��30����0.006��Ϊ0.005 

      
		  
		  b[25]=2.4;			// b(25) Q10: constant for exp. resp. for leaf	         //2017-10-29��2.3��Ϊ2.4
		  b[28]=2.1;			// b(28) Q10: constant for exp. resp. for stem	         //2017-10-29: 2.1��Ϊ2.2 
		  b[29]=2.1;			// b(29) Q10: constant for exp. resp. for roots	       

		  b[49]=60.0;                 //Vcmax25  /Vcmax25  //2012��12��28:40.0;2013_02_28:50.0          //20130623��Ϊ60.0    //2014-09-12 60 to 65  20171127:60  
		   b[50]=27.0;
		  break;

	default:       //��Ҫ��15��17�࣬17��ķֲ���Χ�ȽϹ�


		  b[1]= 25;			    // specific leaf area m2 kg-1 C 	
       //   b[11]=0.001;		// b(11) max. canopy cond., h20  m s-1         
		  b[11]=0.0045;		    // b(11) max. canopy cond., h20  m s-1   
		   
          b[12]=2.7;		/* b(12) lwp at stomatal closure  -MPa , from BIOME-BGC */
   		  
		  b[19]=0.004/RTIMES;	// b(19) leaf resp co.  kg C-1 d-1 kg-1     //2011��8��30����0.006��Ϊ0.005   
		  b[20]=0.0015/RTIMES;	// b(20) stem resp co.   kg C-1 d-1 kg-1      
		  b[21]=0.0015/RTIMES;	// b(21) coarse root resp co.   kg C-1 d-1 kg-1       

          b[25]=2.3;			// b(25) Q10: constant for exp. resp. for leaf	       
		  b[28]=2.1;			// b(28) Q10: constant for exp. resp. for stem	       
		  b[29]=2.1;			// b(29) Q10: constant for exp. resp. for roots	       

          b[49]=52.5;            //Vcmax25  /Vcmax25  //2012��12��28:40.0;2013_02_28:50.0       //2018-02-14: 57.5 to 52.5
		   b[50]=27.0;

  	}													
}	
